#include "hip/hip_runtime.h"
#include "find.h"
#include "slice.h"
#include "table.h"
#include "star.h"
#include "basic.h"
//#include <sys/time.h>
#include <stdlib.h>

#include <stdio.h>
#include "basic-non.h"
#include "star_kernel.h"
#include "iostar.h"
#include "knapsack.h"

//#define rr
int warshall_test()
{
     Table *tab;
	 tab = new Table;
	 int eds=0;
     char fin[]="data/RG5000-4.dat";
     char fout[50];
     sprintf(fout,"res/Warshall/res_%i_w.dat",LENGTH1);
	 tab->Init(VER,VER); // matrix of adjacency
	 tab->readFromFileListAd_or(fin,& eds);
	 puts("file was read");
	 FILE *f_out;
	 double tt;
	 if((f_out=fopen(fout,"wt"))==NULL)return 0;
	 fprintf(f_out, "graph: |V|= %d |E|= %d \n", M,eds);
	 printf( "graph: |V|= %d |E|= %d \n", M,eds);
//	 struct timeval tv1,tv2,tv3;
	 	 	 hipEvent_t start, stop;
	 	     hipEventCreate(&start);
	 	     hipEventCreate(&stop);
	 	    float elapsedTime1, totalTime1;
/*
     gettimeofday(&tv1,NULL);
	 warshall(tab);
     gettimeofday(&tv2,NULL);
	 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
	 fprintf(f_out, "time of work warshall \t \t %f sec \n", tt);
	 printf("time of work warshall-C \t \t %f sec \n", tt);
	 puts("warshall done");
	 tab->writeToFile("res/Warshall/res_warshall-c.dat");
//	 puts("file res_warshall.dat was writen");
*/

	 tab->readFromFileListAd_or(fin,& eds);
	 puts("file was read");
//	 gettimeofday(&tv2,NULL);
	 hipEventRecord(start, 0);
	 warshall_o(tab);
	 hipEventRecord(stop, 0);
	 hipEventSynchronize(stop);
		      	     hipEventElapsedTime(&elapsedTime1, start, stop);// in 0.001 sec
		      	     tt = elapsedTime1/(1000);
//	 gettimeofday(&tv3,NULL); //
	 tab->writeToFile("res/Warshall/res_warshall-o.dat");
//     tt=0.000001*(tv3.tv_usec-tv2.tv_usec)+(tv3.tv_sec-tv2.tv_sec);
     fprintf(f_out, "time of work warshall_CUDA \t %f sec \n", tt);
     printf("time of work warshall_CUDA \t %f sec \n", tt);
     puts("warshal_o done");

     tab->readFromFileListAd_or(fin,& eds);
     puts("file was read");
//     	 gettimeofday(&tv2,NULL);
     	hipEventRecord(start, 0);
     	 warshall_c(tab);
//     	 gettimeofday(&tv3,NULL);
     	 hipEventRecord(stop, 0);
     	hipEventSynchronize(stop);
     	hipEventElapsedTime(&elapsedTime1, start, stop);// in 0.001 sec
     	tt = elapsedTime1/(1000);
     tab->writeToFile("res/Warshall/res_warshall-adapt.dat");
 //         tt=0.000001*(tv3.tv_usec-tv2.tv_usec)+(tv3.tv_sec-tv2.tv_sec);
          fprintf(f_out, "time of work warshall_CUD2  \t %f sec \n", tt);
          printf("time of work warshall_CUD2  \t %f sec \n", tt);
          puts("warshal_c done");
     fclose(f_out);
/*
	 	hipError_t err = hipGetLastError();

        Slice *root;
	 	root=new Slice;
	 	root->Init(M);
        root->SET();
	 	dim3 threads2D(threads1,1);
	 	dim3 blocks2D(blocks1,M);
	     hipEvent_t start, stop;
	  	printf("errors before associative %d\n",err);
	     hipEventCreate(&start);
	     hipEventCreate(&stop);
	     hipEventRecord(start, 0);
	     char s[20];
	     int i_num=0,i_num1=0;
//	     if((f_out=fopen("STEP_dat/STEP.dat","wt"))==NULL)return 0;
	     for(int i=M; i>0;i--)
	     {
//	    	 printf("numb %i",i);
	 //   	 X=left->col(i);//i);
	//    	 root->MASK(i);
	    	 i_num=root->NUMB();
//	    	 printf("step %i",i);
	//    	 i_num1=root->NUMB1();
  //           sprintf(s,"STEP_dat/step%04d",i);//ToString[i],".dat"}];
//	    	 root->print(s,0);
	    	 i_num1=root->STEP();
//	    	if(i<1000)
//	    		fprintf(f_out,"NUMB_naiv(%i)=%i   %i \n", i, i_num,i_num1);
//	    	 printf("NUMB_naiv(%i)=%i   %i \n", i, i_num1);
	     }
//	     fclose(f_out);
/*	     for( int i=20;i<70;i++)
//	    	 subtv_kernel<<<blocks1,threads1>>>(d_tab1,d_tab,h,d_x,d_res,d_z);
//	     subtc1_kernel<<<blocks1,threads1>>>(d_tab1, d_x, d_w, d_res,h,d_z);
	     { //tab->GetCol(root,i);
	       root->MASK(i);
	       root->print("MASK",0);
	     }*/
/*
//	    	 tmarge_kernel<<<blocks1,threads1>>>(d_tab,d_x,d_res);
//	    	 tmarge_kernel<<<blocks2D,threads2D>>>(d_tab,d_x,d_res);
//		  SUBTV(right,left,X,result);
//		  for( int i=0;i<1000;i++) MIN(right,X,Y,Z);
//    	 gettimeofday(&tv2,NULL);

	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
	     float elapsedTime1, totalTime1;
	     hipEventElapsedTime(&elapsedTime1, start, stop);

	     totalTime1 = elapsedTime1/(1000*M);

	     printf("associative time row= %f\n", totalTime1);

	err = hipGetLastError();
	printf("errors after associative %d\n",err);
//		 NV->writeToFile("res_DFS.dat");
//		 X->print("res_less_great",0);
//  	     root->print("res_subtc1_word",1);
//		 Y->print("res_MIN1-1000_slice",0);
*/
/*
//////////////////////////////////////////////
	          int blockSize;   // The launch configurator returned block size
			  int minGridSize; // The minimum grid size needed to achieve the
			                   // maximum occupancy for a full device launch
			  int gridSize;    // The actual grid size needed, based on input size

			  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
					  match_kernel, 0, 0);
			  printf("optimal minGS=%i BS=%i \n", minGridSize,blockSize);
//////////////////////////////////////////////
 *
 */
/*
		 clock_t t;
      t=clock();
      hipEventRecord(start, 0);
      for( int i=0;i<M;i++)
     	 //MIN(right,X,Y);
//     	 addv_kernel<<<blocks1,threads1>>>(d_tab1,d_tab,h,d_x,d_res,d_z);
      {     tab->GetRow_opt(root,i);
      tab->GetRow(root,i);}
	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
	     hipEventElapsedTime(&elapsedTime1, start, stop);

	     totalTime1 = elapsedTime1/(1000*M);

	     printf("associative time row_opt= %f\n", totalTime1);
*/

     return 1;
}

int DFS_test()
{

	 Table *left, *right, *code,*NV;
	 Slice *X, *Y,*root;
	 int i=1; // number of root;

	 left = new Table;
	 right=new Table;
	 code=new Table;
	 NV=new Table;
	 int eds=0;

	 left->Init(LENGTH1,M);
	 right->Init(LENGTH1,M);
	 code->Init(VER,M);
	 NV->Init(VER,M);

	 X=new Slice;
	 X->Init(LENGTH1);
	 X->SET();


	 Y=new Slice;
	 Y->Init(LENGTH1);


	 root=new Slice;
	 root->Init(M);

	 readFromFileListLR("graph10.dat",&eds,left,right);
	 char str[M+1];
	 for(int i=1; i<=VER;i++)
	    {
	    	long_to_binary1(i,str,M);
	    	*root=str;
	    	code->SetRow(root,i);
	    }
     puts("code end");
	 code->GetRow(root,i);
	 FILE *f_out;
	 double tt;
//	 struct timeval tv1,tv2;
	 if((f_out=fopen("res_info_DFS-100000.dat","wt"))==NULL)return 0;
		 fprintf(f_out, "graph: |V|= %d |E|= %d \n", VER,LENGTH1);

//	 gettimeofday(&tv1,NULL);
	 puts("before DFS");
	 DFS(left, right, code, root, NV, Y, X);
//	 gettimeofday(&tv2,NULL);
	 NV->writeToFile("res_DFS.dat");
	 X->print("res_DFS_nnum",0);
	 Y->print("res_DFS_Tree",0);
//     tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
     fprintf(f_out, "time of work DFS %f sec \n", tt);
     fclose(f_out);

     return 1;
}

int LibraryTest()
{ Table *left, *right,*result;
Slice *X, *Y, *Z,*root;

left = new Table;
right=new Table;
result= new Table;
int eds=0;

left->Init(LENGTH1,M);
right->Init(LENGTH1,M);
result->Init(LENGTH1,M);
puts("init was done");
X=new Slice;
X->Init(LENGTH1);
X->SET();

/*if (X->SOME()) puts("some");
else puts("zero");
if (X->SOME1()) puts("some1");
else puts("zero1");
X->CLR();
if (X->SOME()) puts("after X->CLR some");
else puts("zero");
if (X->SOME1()) puts("after X->CLR some1");
else puts("zero1");
*/

X->set(10,0);
Y=new Slice;
Y->Init(LENGTH1);
Z=new Slice;
Z->Init(LENGTH1);
root=new Slice;
root->Init(M);


char fin[]="data/test_graph10000.txt";
    char fout[50];
    sprintf(fout,"res/test_lib/res_%i_%i_star.dat",LENGTH1,H1);
	 FILE *f_out;
	 if((f_out=fopen(fout,"wt"))==NULL)return 0;
	 initIO();
	 readfromDimageL(fin,left,right);
left->GetRow(root,10);

//puts("data was read");
unsigned long long int *d_x,*d_z, *d_w;
	 LongPointer *d_tab,*d_tab1,*d_res;
     int Nl=left->size;
     int NN=X->NN,i_num,i_num1;
     int h=H1;
     int LOOPS=100;

     d_x=X->get_device_pointer();
	 d_w=root->get_device_pointer();
	 d_z=Y->get_device_pointer();
	 d_tab=left->get_device_pointer();
	 d_tab1=right->get_device_pointer();
	 d_res=result->get_device_pointer();
//	 root->print("res_wmerge_word-before",1);
//	 struct timeval tv1,tv2;
//	 double tt;
/*	 if((f_out=fopen("res_min_time-100000-16.dat","wt"))==NULL)return 0;
	 fprintf(f_out, "graph: |V|= %d |E|= %d ,blocks=%d, threads=%d \n", VER,LENGTH1,blocks1,threads1);
	  int h=M;
       printf("Length=%i NN=%i, blocks=%i, threads=%i \n",LENGTH1, NN1, blocks1, threads1);
//	  gettimeofday(&tv1,NULL);
//	  MAX(left,X,Y);
*/

//	 addc1_kernel<<<NN,1>>>(d_x,d_w,h,d_tab,d_z);
        X->SET();
	 	hipError_t err = hipGetLastError();
	 	printf("errors before associative %d\n",err);

	 	dim3 threads2D(threads1,1);
	 	dim3 blocks2D(blocks1,M);
	     hipEvent_t start, stop;
	     hipEventCreate(&start);
	     hipEventCreate(&stop);
//	     hipEventRecord(start, 0);
	     float elapsedTime1, totalTime1;
/*	     for(int i=M; i>0;i--)
	     {
	 //   	 X=left->col(i);//i);
	    	 i_num=X->NUMB();
//	    	 i_num1=X->NUMB1();
	    	 X->STEP();
	    	 fprintf(f_out,"NUMB_naiv(%i)=%i   %i \n", i, i_num,i_num1);
	     }
*/

	     hipEventRecord(start, 0);
	   	     for( int i=0;i<LOOPS;i++)
	   	    	 MATCH_CUDA(left,X,root,Z);
	   	     hipEventRecord(stop, 0);
	   	     hipEventSynchronize(stop);
	      	     hipEventElapsedTime(&elapsedTime1, start, stop);// in 0.001 sec
	      	     totalTime1 = elapsedTime1/(1000*LOOPS);
	      	     fprintf(f_out,"associative time MATCH= %f (%i)\n", totalTime1, NN1);

	     hipEventRecord(start, 0);
	     for( int i=0;i<LOOPS;i++)
	    	 MIN(right,X,Y,Z);
	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
   	     hipEventElapsedTime(&elapsedTime1, start, stop);// в милисекундах!
   	     totalTime1 = elapsedTime1/(1000*LOOPS);
   	     i_num=Z->FND();
   	     fprintf(f_out,"associative time MIN= %f (%i)\n", totalTime1,i_num);


	     hipEventRecord(start, 0);
//	     for( int i=0;i<1000;i++)
//                                             (LongPointer *d_t, LongPointer *d_r,int k, unsigned long long int *d_x, LongPointer *d_s,unsigned long long int *d_m)

	    	 subtv_kernel<<<blocks1,threads1>>>(d_tab1,           d_tab,               h,                         d_x,            d_res,                        d_z);
	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
   	     hipEventElapsedTime(&elapsedTime1, start, stop);// в милисекундах!
   	     totalTime1 = elapsedTime1/1000;
   	     fprintf(f_out,"associative time SUBTV= %f (%i)\n", totalTime1,NN1);

	     hipEventRecord(start, 0);
	     for( int i=0;i<LOOPS;i++)
	    	 tmerge_kernel<<<blocks1,threads1>>>(d_tab,d_x,d_res);
	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
   	     hipEventElapsedTime(&elapsedTime1, start, stop);// в милисекундах!
   	     totalTime1 = elapsedTime1/(1000*LOOPS);
   	     fprintf(f_out,"associative time TMARCH1D= %f (%i)\n", totalTime1,NN1);

	     hipEventRecord(start, 0);
	     for( int i=0;i<LOOPS;i++)
	    	 tmerge_kernel<<<blocks2D,H1>>>(d_tab,d_x,d_res);
	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
   	     hipEventElapsedTime(&elapsedTime1, start, stop);// в милисекундах!
   	     totalTime1 = elapsedTime1/(1000*LOOPS);//
   	     fprintf(f_out,"associative time TMARCH2D= %f (%i)\n", totalTime1,NN1);

//	    	 subtv_kernel<<<blocks1,threads1>>>(d_tab1,d_tab,h,d_x,d_res,d_z);
//	     subtc1_kernel<<<blocks1,threads1>>>(d_tab1, d_x, d_w, d_res,h,d_z);
//	     left->GetRow(root,i);
//	    	 tmarge_kernel<<<blocks1,threads1>>>(d_tab,d_x,d_res);
//	    	 tmarge_kernel<<<blocks2D,threads2D>>>(d_tab,d_x,d_res);
//		  SUBTV(right,left,X,result);
//		  for( int i=0;i<1000;i++) MIN(right,X,Y,Z);
 //    	 gettimeofday(&tv2,NULL);

//	     hipEventRecord(stop, 0);
//	     hipEventSynchronize(stop);
//	     float elapsedTime1, totalTime1;
//	     hipEventElapsedTime(&elapsedTime1, start, stop);
//	     totalTime1 = elapsedTime1/(1000*M);
//	     fprintf(f,"associative time NUMB= %f (%i)\n", totalTime1,i_num);
/*
 	err = hipGetLastError();
 	printf("errors after associative %d\n",err);
//		 NV->writeToFile("res_DFS.dat");
//		 X->print("res_less_great",0);
//  	     root->print("res_subtc1_word",1);
//		 Y->print("res_MIN1-1000_slice",0);
		 int j=Y->FND();
		 clock_t t;
         t=clock();
         */
 /*        hipEventRecord(start, 0);
         for(int i=M; i>0;i--)
       	     {
       	    	 X=left->col(i);//i);
       	    	 i_num=X->NUMB1();
       	    	printf("NUMB1(%i)=%i \n", i, i_num);
       	     }
    //     for( int i=0;i<1000;i++)
    //    	MIN(right,X,Y);
   //     	 addv_kernel<<<blocks1,threads1>>>(d_tab1,d_tab,h,d_x,d_res,d_z);
   //      left->GetRow_opt(root,i);
	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
	     hipEventElapsedTime(&elapsedTime1, start, stop);

	     totalTime1 = elapsedTime1/(1000*M);

	     printf("time NUMB_thrust= %f (%i)\n", totalTime1,i_num);
	     */
 /*
	     //       t=clock()-t;
 //        printf("+ time = %f\n", ((float)t)/(CLOCKS_PER_SEC*1000));
     	err = hipGetLastError();
      	printf("errors after associative %d\n",err);
 //        Y->print("res_MIN-1000_slice",0);
 //        root->print("res_min_1000_word",1);
 //          left->writeToFile("res_tmarge-1000_left.dat");
 //  	     right->writeToFile("res_min-1000_right.dat");
//		 result->writeToFile("res_addv-10_result.dat");
//	     tt=((tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec)*1000000)*0.01;
//	     fprintf(f_out, "time of work + %f usec \n", totalTime1);
	     fclose(f_out);
//	     printf("min = %i t=%.2lf usec \n",j,tt);

	     hipEventDestroy(start);
	     hipEventDestroy(stop);
 *
 */
	     maintest();

}

int triangles_test()
{
	 Table *tab;
	 tab = new Table;
	 int eds=0;

	 tab->Init(VER,VER); // matrix of adjacency
	 tab->readFromFileListAd_unor("graph_n13.dat",& eds);
	 puts("graph was read");
//	 tab->writeToFile("res_TM.dat");
	 printf("count of verteces=%i, count of edges=%i\n",VER ,eds);
	 FILE *f_out;
	 double tt;
//	 if((f_out=fopen("res_info_w.dat","wt"))==NULL)return 0;
//	 fprintf(f_out, "graph: |V|= %d |E|= %d \n", M,eds);
//     puts("file was read");
//	 struct timeval tv1,tv2,tv3;
//     gettimeofday(&tv1,NULL);
     CountTriangles(tab, &eds);
//     gettimeofday(&tv2,NULL);
//	 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
	 printf("time of work thriangles \t \t %f sec (%i)\n", tt,eds);
	 puts("countTriangles done");

//	 tab->writeToFile("res_warshall.dat");
	//    gettimeofday(&tv1,NULL);
	     CountTrianglesOPT(tab, &eds);
	//     gettimeofday(&tv2,NULL);
//		 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
		 printf("time of work optimizated thriangles \t \t %f sec (%i)\n", tt,eds);
		 puts("countTriangles done");

/*	 	hipError_t err = hipGetLastError();

	     hipEvent_t start, stop;
	     hipEventCreate(&start);
	     hipEventCreate(&stop);
	     hipEventRecord(start, 0);

	     hipEventRecord(stop, 0);
	     hipEventSynchronize(stop);
	     float elapsedTime1, totalTime1;
	     hipEventElapsedTime(&elapsedTime1, start, stop);

	     totalTime1 = elapsedTime1/(1000*1000);

	     printf("associative time row= %f\n", totalTime1);
*/
     return 1;
}
  void GRinput_test()
  { Table *tab;
	 tab = new Table;
	  initIO();
	 tab->Init(LENGTH1,M); // matrix of weights
	 readfromDimageW("USA-road-d.BAY.gr", tab);
	 writetoDimageW("wights.dat", tab);

  }

  void DecOut_test()
  {
	  Table *left, *right,*weight;
	  initIO();
	  left = new Table;
	  right=new Table;
	  weight=new Table;
	  int eds=0;

	  left->Init(LENGTH1,M);
	  right->Init(LENGTH1,M);
	  weight->Init(LENGTH1,M);

	  puts("init was done");

	  LongPointer *d_left;
	  d_left=left->get_device_pointer();
//	  readFromFileListLR("graph1000.dat",&eds,left,right);
	  readfromDimageL("USA-road-d.BAY.gr", left, right,weight);
//	  writeStrip("left_dec1.dat",d_left,0);
	  writetoDimageL("output.dat",left,right,weight);
	  printf("Infinite %i ",INFINITE);
  }



int main(void)
{
	InitArrays();

	knapsack_exp();
//	printf("Length=%i N=%i blocks=%i threads=%i \n", LENGTH1, NN1, blocks1,threads1);
}
