#include "hip/hip_runtime.h"
#include "param.h"
#include "find.h"
#include "table.h"
#include "iostar.h"
#include <stdio.h>
//#include <stdlib.h>
#include <sys/time.h>

extern char **tb;//[LENGTH1][M]![LENGTH1][LENGTH1+1];  //table transposed

int **tt;//[size_tt][LENGTH1]
int size_tt;
//VER for matrix of weight or adjacency
//2 for unweighed list of arcs
//3 for weighed list of arcs

void readfromDimageA(char *fn, Table *T )
// read matrix of adjacency from dimage format ('a' l r w)
{	int ras=max(100,M+1);
    char str[ras];

	FILE *f;
	int   n,i,k,l;
	int M2=T->size;
//	int eds=0;
	int length=T->length;

//	puts(" begin set by 0\n");
	for (l=0; l<M2;l++)
	{
		for(int i = 0;i < length;i++)
		{
			tb[i][l] ='0';
		}
	}

	if((f = fopen(fn,"rt")) == NULL)
	{
		puts("file error");
		return;
	}

	while(fgets(str,ras,f) != NULL)
{

  if(str[0]=='a')
  {
	n=atoi(str+1);
	k=n;
	l=1;
	while (k>0)
	{
		k=k/10;
		l++;
	}
	n--;
	i=atoi(str+l+2);
	i--;
	if ((i<M2) and (n<M2)){
		 tb[n][i]='1';

//		 puts(str);
//		 printf("<%i,%i> %c\n",n,i,str[l+3]);
//		 eds++;
	}
  }
  for(int i = 0;i < length;i++)
  {
  	tb[i][M2] = 0;
  }
}
fclose(f);

Slice *s;
s=new Slice;
s->Init(M2);
for(int i = 0;i < M2;i++)
{
//		printf("copying %d slice ",i);
for(int j = 0;j < length;j++)
	{
		str[j] = tb[i][j];
//			printf("%c",tb[j][i]);
	}
//	   	str[M] = 0;
//      printf("\n");

//		printf("slice %d %s \n",i,str);
	s = T->col(i+1);
	*s = str;
}
//    puts("end read from file\n");
}

__global__ void vect_to_strip(int j, int *d_tmp, LongPointer *d_tab)
{
__shared__ int ind_t;
           ind_t=j*H1+blockIdx.x; //номер обрабатываемого столбца полосы
//blockIdx.x - номер столбца в полосе, (H1-blockIdx.x) - позиция бита в слове вектора
//k=threadId.x - номер ull в полосе обрабатывает (k*64...(k+1)*64-1) элемента массива, каждая нить свой столбец.
// в последнем блоке нужно проверить выход за границы вектора

int pos=1<<(H1-blockIdx.x-1);
int ind0=threadIdx.x*SIZE_OF_LONG_INT;
int k=((ind0+SIZE_OF_LONG_INT)>LENGTH1)?(LENGTH1%SIZE_OF_LONG_INT):SIZE_OF_LONG_INT;// параметр цикла or (and)
unsigned long long int number=0;
unsigned long long int pos1=1;
	for (int i=0; i<k;i++)
	{
		if(d_tmp[ind0+i]&pos)
			number|=pos1;
		pos1=pos1<<1;
	}
	// в number собран threadIdx.x-й элемент ind_t-го столбца

	__shared__ LongPointer d_col;
	d_col=d_tab[ind_t];
	d_col[threadIdx.x]=number;
//    if ((blockIdx.x==0)||(blockIdx.x==(gridDim.x-1)))printf("<%i,%i> ",ind_t,threadIdx.x);
}

void readfromDimageW(char *fn, Table *T)
// read matrix of weights from dimage format ('a' l r w)
{
	int ras=100;
	    char str[ras];

		FILE *f;
		int   n,i,j,k,l;
		int M2=size_tt;
	//	int eds=0;
		int length=T->length;
		int *d_tt;

		puts(" begin set by 0\n");
		for (l=0; l<M2;l++)
		{
			for(int i = 0;i < length;i++)
			{
				tt[l][i]=INFINITE;
			}
		}
		puts(" end set by 0\n");
		if((f = fopen(fn,"rt")) == NULL)
		{
			puts("file error");
			return;
		}

		while(fgets(str,ras,f) != NULL)
	{

	  if(str[0]=='a')
	  {
		n=atoi(str+1);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}
		n--;

		i=atoi(str+l+2);
		k=i;
				while (k>0)
				{
					k=k/10;
					l++;
				}
		i--;
		j=atoi(str+l+3);
//		 printf("<%i,%i,%i> \n",n+1,i+1,j);
		if ((i<VER) && (n<VER))
		{
			d_tt=tt[n];
			 d_tt[i]=j;

	//		 puts(str);
	//		 printf("<%i,%i,%i> \n",n+1,i+1,j);
	//		 eds++;
		}
//		else printf("=======<%i,%i,%i> \n",n+1,i+1,j);
	  }
	}
	 fclose(f);
    puts("file was read");
// tt хранит десятичную матрицу весов на CPU, ее нужно преобразовать в бинарную на GPU.
//    действовать по столбцам
    int *d_tmp;
    LongPointer *d_tab;
    d_tab=T->get_device_pointer();

    hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);

    for (j=0; j<M2; j++)
    {
    	hipMemcpy(d_tmp,tt[j],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
 //   	printf("\n column %i\n",j);

    	vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
    }
    err = hipGetLastError();
    if (err!=0) printf("after vect_to_strip %d , %s \n",err,hipGetErrorString(err));
}

void readfromDimageC(char *fn, Table *T)
// read matrix of weights from dimage format ('a' l r w)
{
	int ras=100;
	    char str[ras];

		FILE *f;
		int   n,i,j,k,l;
		int M2=size_tt;
	//	int eds=0;
		int length=T->length;
		int *d_tt;

		puts(" begin set by 0\n");
		for (l=0; l<M2;l++)
		{
			for(int i = 0;i < length;i++)
			{
				tt[l][i]=INFINITE;
			}
		}
		puts(" end set by 0\n");
		if((f = fopen(fn,"rt")) == NULL)
		{
			puts("file error");
			return;
		}

		while(fgets(str,ras,f) != NULL)
	{

	  if(str[0]=='a')
	  {
		n=atoi(str+1);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}
		n--;

		i=atoi(str+l+2);
		k=i;
				while (k>0)
				{
					k=k/10;
					l++;
				}
		i--;
		j=atoi(str+l+3);
//		 printf("<%i,%i,%i> \n",n+1,i+1,j);
		if ((i<VER) && (n<VER))
		{
			d_tt=tt[i];
			 d_tt[n]=j;

	//		 puts(str);
	//		 printf("<%i,%i,%i> \n",n+1,i+1,j);
	//		 eds++;
		}
//		else printf("=======<%i,%i,%i> \n",n+1,i+1,j);
	  }
	}
	 fclose(f);
    puts("file was read");
// tt хранит десятичную матрицу весов на CPU, ее нужно преобразовать в бинарную на GPU.
//    действовать по столбцам
    int *d_tmp;
    LongPointer *d_tab;
    d_tab=T->get_device_pointer();

    hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);

    for (j=0; j<M2; j++)
    {
    	hipMemcpy(d_tmp,tt[j],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
 //   	printf("\n column %i\n",j);

    	vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
    }
    err = hipGetLastError();
    if (err!=0) printf("after vect_to_strip %d , %s \n",err,hipGetErrorString(err));
}

void readfromDimageL(char *fn, Table *L, Table *R)
// read list of unweighed arcs from dimage format ('a' l r w)
{
	int ras=100;
	    char str[ras];

		FILE *f;
		int   n,i,j,k,l;
		int M2=size_tt;
		int eds=0;
		int length=L->length;
		int *tt_left;
		int *tt_right;
         tt_left=tt[0];
		 tt_right=tt[1];
		 printf("tt->%p, tt_left->%p tt_right->%p \n", tt,tt_left,tt_right);
		puts(" begin set by 0\n");
			for(int i = 0;i < length;i++)
			{
				tt_left[i]=0;
				tt_right[i]=0;
			}
		puts(" ended set by 0\n");
		if((f = fopen(fn,"rt")) == NULL)
		{
			puts("file error");
			return;
		}

		while((fgets(str,ras,f) != NULL)&&(eds<LENGTH1))
	{

	  if(str[0]=='a')
	  {
		n=atoi(str+1);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}
//		n;

		i=atoi(str+l+2);
		k=i;
				while (k>0)
				{
					k=k/10;
					l++;
				}
//		i;
//		j=atoi(str+l+3);
//	 printf("<%i,%i> \n",n,i);
		if ((i<VER) and (n<VER)){
			 tt_left[eds]=n;
			 tt_right[eds]=i;
	//		 puts(str);
//			 printf("<%i,%i> \n",n,i);
			 eds++;
		}
	  }
	}
	 fclose(f);
    puts("file was read");

/*    if((f = fopen("tt.dat","wt")) == NULL)return;
    for(i=0; i<eds;i++)
    {
    	fprintf(f,"%i %i\n",tt_left[i],tt_right[i]);
    }
    fclose(f);*/
// tt хранит десятичный список вершин на CPU, ее нужно преобразовать в бинарную на GPU.
//    действовать по столбцам
    int *d_tmp;
    LongPointer *d_tab;

    j=0;
    hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
//Left
    hipMemcpy(d_tmp,tt[0],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
    puts("vect was copied");
    d_tab=L->get_device_pointer();
    vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
//Right
    hipMemcpy(d_tmp,tt[1],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
    d_tab=R->get_device_pointer();
    vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
}
void readfromDimageL(char *fn, Table *L, Table *R, Table *W)
// read list of weighted arcs from dimage format ('a' l r w)
{
	int ras=100;
	    char str[ras];

		FILE *f;
		int   n,i,j,k,l;
		int M2=size_tt;
		int eds=0;
		int length=L->length;
		int *tt_left;
		int *tt_right;
		int *tt_weight;
         tt_left=tt[0];
		 tt_right=tt[1];
		 tt_weight=tt[2];

	//	puts(" begin set by 0\n");
		for(int i = 0;i < length;i++)
					{
						tt_left[i]=0;
						tt_right[i]=0;
					}

		if((f = fopen(fn,"rt")) == NULL)
		{
			puts("file error");
			return;
		}

		while((fgets(str,ras,f) != NULL)&&(eds<LENGTH1))
	{

	  if(str[0]=='a')
	  {
		n=atoi(str+1);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}

		i=atoi(str+l+2);
		k=i;
				while (k>0)
				{
					k=k/10;
					l++;
				}
		j=atoi(str+l+3);
		if ((i<VER) and (n<VER)){
			 tt_left[eds]=n;
			 tt_right[eds]=i;
			 tt_weight[eds]=j;
	//		 puts(str);
//			 printf("<%i,%i> \n",n,i);
			 eds++;
		}
	  }
	}
	 fclose(f);
    puts("file was red");
// tt хранит десятичную матрицу весов на CPU, ее нужно преобразовать в бинарную на GPU.
//    действовать по столбцам
    int *d_tmp;
        LongPointer *d_tab;


        hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
        j=0;
//Left
    hipMemcpy(d_tmp,tt[0],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
    d_tab=L->get_device_pointer();
    vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
//Right
    hipMemcpy(d_tmp,tt[1],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
    d_tab=R->get_device_pointer();
    vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
//Weight
    hipMemcpy(d_tmp,tt[2],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
    d_tab=W->get_device_pointer();
    vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
}

__global__ void strip_to_vect(int j, int *d_tmp, LongPointer *d_tab)
{
__shared__ int ind_t;
           ind_t=j*H1+threadIdx.x; //номер обрабатываемого столбца полосы
//blockIdx.x - номер столбца в полосе, (H1-blockIdx.x) - позиция бита в слове вектора
//k=threadId.x - номер ull в полосе обрабатывает (k*64...(k+1)*64-1) элемента массива, каждая нить свой столбец.
// в последнем блоке нужно проверить выход за границы вектора
//printf("===================================__global__ strip_to_vect\n============================\n");
int pos=1<<(H1-threadIdx.x-1);
int ind0=blockIdx.x*SIZE_OF_LONG_INT;//number of elements
int k=(((blockIdx.x+1)*SIZE_OF_LONG_INT)>LENGTH1)?(LENGTH1%SIZE_OF_LONG_INT):SIZE_OF_LONG_INT;// параметр цикла or (and)
__shared__ int number;
   number=0;
  LongPointer d_col;
    d_col=d_tab[ind_t];
unsigned long long int num=d_col[blockIdx.x];
//printf("block=%i thread=%i pos=%i k=%i \n",blockIdx.x, threadIdx.x, pos,k);
unsigned long long int pos1=1;
	for (int i=0; i<k;i++)
	{
		if(num&pos1)
		{
			atomicOr(&number,pos);
//			printf("%i %i\n",ind0+k,threadIdx.x);
		}
		__syncthreads();
		pos1=pos1<<1;

		if (threadIdx.x==0)
		{
//			printf("block=%i thread=%i =====tmp[%i]=%i\n", ind0+i, number);
			d_tmp[ind0+i]=number;
			number=0;
		}

		__syncthreads();
	}
	// в number собран threadIdx.x-й элемент ind_t-го столбца

}
__global__ void strip_to_vect1(int j, int *d_tmp, LongPointer *d_tab)
{
__shared__ int ind_t;
           ind_t=(j+1)*H1-1; //номер последнего столбца обрабатываемой  полосы
//blockIdx.x - номер элемента в столбце, (H1-blockIdx.x) - позиция бита в слове вектора
//k=threadId.x - номер строки в элементе.
// в последнем блоке нужно проверить выход за границы вектора

int pos=1;
int ind0=blockIdx.x*SIZE_OF_LONG_INT+threadIdx.x;//number of row
//printf("====ind=%i \n",ind0);
   if(ind0<LENGTH1)
   {
	   int number=0;
	   LongPointer d_col;
	   unsigned long long int num;
	   unsigned long long int pos1=1;
	   	   	   	   pos1<<=threadIdx.x;
	   for(int i=0; i<H1;i++)
	   {
		   d_col=d_tab[ind_t];
		   num=d_col[blockIdx.x];
		   if(num&pos1)
		   {
			   number|=pos;
//			   printf("row=%i i=%i col_numb=%i pos=%i ===============",ind0,i,ind_t,pos);
		   }
		   pos<<=1;
		   ind_t--;
	   }
	   d_tmp[ind0]=number;
//	   printf("====ind=%i th=%i  number=%i\n",ind0,threadIdx.x,number);
   }
}
void writeStrip(char *fn, LongPointer *d_tab, int i)
{   FILE *f;
	int *d_tmp, d_t[LENGTH1];
	int inf=(1<<H1)-1;
	hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
//	 struct timeval tv1,tv2;
//	 double tt;
//	gettimeofday(&tv1,NULL);
	strip_to_vect<<<N1,H1>>>(i,d_tmp,d_tab);
//	gettimeofday(&tv2,NULL);
//			 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
//			 printf("time of work strip_to_vect \t \t %f sec \n", tt);

//	gettimeofday(&tv1,NULL);
//	strip_to_vect1<<<N1,SIZE_OF_LONG_INT>>>(i,d_tmp,d_tab);
//	gettimeofday(&tv2,NULL);
//				 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
//				 printf("time of work strip_to_vect1 \t \t %f sec \n", tt);

	hipMemcpy(d_t,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);
//	puts("bin to dec ended");
	if((f = fopen(fn,"wt")) == NULL)return;
	for(int i = 0;i<LENGTH1;i++)
			{
		        if(d_t[i]!=inf)fprintf(f,"%i \n",d_t[i]);
					else fprintf(f,"inf \n");
			}
//	puts("print all");
	fclose(f);
}
void printStrip( LongPointer *d_tab, int i)
{
	int *d_tmp, d_t[LENGTH1];
	int inf=(1<<H1)-1;
	hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
//	 struct timeval tv1,tv2;
//	 double tt;
//	gettimeofday(&tv1,NULL);
	strip_to_vect<<<N1,H1>>>(i,d_tmp,d_tab);
//	gettimeofday(&tv2,NULL);
//			 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
//			 printf("time of work strip_to_vect \t \t %f sec \n", tt);

//	gettimeofday(&tv1,NULL);
//	strip_to_vect1<<<N1,SIZE_OF_LONG_INT>>>(i,d_tmp,d_tab);
//	gettimeofday(&tv2,NULL);
//				 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
//				 printf("time of work strip_to_vect1 \t \t %f sec \n", tt);

	hipMemcpy(d_t,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);
//	puts("bin to dec ended");
//	printf("inf=%i \n", INFINITE);
	for(int i = 0;i<LENGTH1;i++)
			{
				if(d_t[i]!=inf)printf("%i ",d_t[i]);
						else printf("inf ");
			}
	printf("\n");
//	puts("print all");
}
void writetoDimageA(char *fn, Table *T )
// write matrix of adjacency to dimage format ('a' l r w)
{ FILE *f;
   int k,j=0;
   Slice *X;
   X=new Slice;
   X->Init(LENGTH1);
//   puts("WtDA 1");
  if((f = fopen(fn,"wt")) == NULL){puts("can not open file");return;}
  for(int i=1;i<=VER;i++)
  {
	  T->GetCol(X,i);
//	  X->print("X",1);
//	  printf("%i \t",i);
//	  puts("WtDA 2");
	  k=X->STEP();
	  while(k>0)
	  {   j++;
		  fprintf(f,"a %i %i\n",i,k);
		  k=X->STEP();
	  }
  }
  puts("WtDA 3");
  fprintf(f,"p sp %i %i\n",VER,j);
  fclose(f);
}

void writetoDimageW(char *fn, Table *T)
// write matrix of weights to dimage format ('a' l r w)
{
	FILE *f;
	int *d_tmp, d_t[LENGTH1];
	hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
	int * d_tt;
	LongPointer *d_tab;
	d_tab=T->get_device_pointer();
 printf("size_tt=%i\n",size_tt);
	for (int i=0; i<size_tt;i++)
	{
		d_tt=tt[i];
		strip_to_vect<<<N1,H1>>>(i,d_tmp,d_tab);
		hipMemcpy(d_tt,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);
	}

	if((f = fopen(fn,"wt")) == NULL)return;
	fprintf(f,"c \np  sp %i %i \nc graph containts %i nodes and %i arcs \n",VER, LENGTH1,VER, LENGTH1);
	for(int i = 0;i<VER;i++)
	{
		for(int j=0;j<LENGTH1;j++)
		{
			if (tt[j][i]!=INFINITE)
			{
				fprintf(f,"a %i %i %i\n",i+1,j+1,tt[j][i]);
			    printf(" %i\t", tt[j][i]);
			}
			else  printf(" --\t");
		}
	printf("\n");
	}
//	puts("print all");
fclose(f);

}

void writetoDimageL(char *fn, Table *L, Table *R)
// write list of unweighed arcs to dimage format ('a' l r w)
{
	  FILE *f;
		int *d_tmp, d_t[LENGTH1];
		int *d_left;
		d_left=tt[0];
		int *d_right;
		int i=0;
		d_right=tt[1];
		hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
/*		 struct timeval tv1,tv2;
		 double tt;
		gettimeofday(&tv1,NULL);*/
		LongPointer *d_tab;
		d_tab=L->get_device_pointer();
		strip_to_vect<<<N1,H1>>>(i,d_tmp,d_tab);
		hipMemcpy(d_left,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);

		d_tab=R->get_device_pointer();
		strip_to_vect<<<N1,SIZE_OF_LONG_INT>>>(i,d_tmp,d_tab);
		hipMemcpy(d_right,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);
	//	puts("bin to dec ended");
		if((f = fopen(fn,"wt")) == NULL)return;
		fprintf(f,"c \np  sp %i %i \nc graph containts %i nodes and %i arcs \n",VER, LENGTH1,VER, LENGTH1);
		for(i = 0;i<LENGTH1;i++)
				{
					fprintf(f,"a %i %i\n",d_left[i],d_right[i]);
				}
	//	puts("print all");
		fclose(f);
}

void writetoDimageL(char *fn, Table *L, Table *R, Table *W)
// write list of weighted arcs to dimage format ('a' l r w)
{
	  FILE *f;
		int *d_tmp, d_t[LENGTH1];
		int *d_left;
		d_left=tt[0];
		int *d_right;
		int i=0;
		d_right=tt[1];
		int *d_weight;
		d_weight=tt[2];
		hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);
/*		 struct timeval tv1,tv2;
		 double tt;
		gettimeofday(&tv1,NULL);*/
		LongPointer *d_tab;
		d_tab=L->get_device_pointer();
		strip_to_vect<<<N1,H1>>>(i,d_tmp,d_tab);
		hipMemcpy(d_left,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);

		d_tab=R->get_device_pointer();
		strip_to_vect<<<N1,SIZE_OF_LONG_INT>>>(i,d_tmp,d_tab);
		hipMemcpy(d_right,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);

		d_tab=W->get_device_pointer();
		strip_to_vect<<<N1,SIZE_OF_LONG_INT>>>(i,d_tmp,d_tab);
		hipMemcpy(d_weight,d_tmp,sizeof(int)*LENGTH1,hipMemcpyDeviceToHost);

	//	puts("bin to dec ended");
		if((f = fopen(fn,"wt")) == NULL)return;
		fprintf(f,"c \np  sp %i %i \nc graph containts %i nodes and %i arcs \n",VER, LENGTH1,VER, LENGTH1);
		for(i = 0;i<LENGTH1;i++)
				{
					fprintf(f,"a %i %i %i\n",d_left[i],d_right[i],d_weight[i]);
				}
	//	puts("print all");
		fclose(f);
}

void readfromDimageWC(char *fn, Table *T, Table *Cost)
{
	int ras=100;
	    char str[ras];

		FILE *f;
		int   n,i,j,k,l;
		int M2=size_tt;
	//	int eds=0;
		int length=T->length;
		int *d_tt;

		puts(" begin set by 0\n");
		for (l=0; l<M2;l++)
		{
			for(int i = 0;i < length;i++)
			{
				tt[l][i]=INFINITE;
			}
		}
		puts(" end set by 0\n");
		if((f = fopen(fn,"rt")) == NULL)
		{
			puts("file error");
			return;
		}

		while(fgets(str,ras,f) != NULL)
	{

	  if(str[0]=='a')
	  {
		n=atoi(str+1);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}
		n--;

		i=atoi(str+l+2);
		k=i;
				while (k>0)
				{
					k=k/10;
					l++;
				}
		i--;
		j=atoi(str+l+3);
//		 printf("<%i,%i,%i> \n",n+1,i+1,j);
		if ((i<VER) && (n<VER))
		{
			d_tt=tt[n];
			 d_tt[i]=j;

	//		 puts(str);
	//		 printf("<%i,%i,%i> \n",n+1,i+1,j);
	//		 eds++;
		}
//		else printf("=======<%i,%i,%i> \n",n+1,i+1,j);
	  }
	}
	 fclose(f);
    puts("file was read");
// tt хранит десятичную матрицу весов на CPU, ее нужно преобразовать в бинарную на GPU.
//    действовать по столбцам
    int *d_tmp;
    LongPointer *d_tab;
    d_tab=T->get_device_pointer();

    hipError_t err = hipMalloc(&d_tmp,sizeof(int)*LENGTH1);

    for (j=0; j<M2; j++)
    {   d_tt=tt[j];
    	hipMemcpy(d_tmp,tt[j],sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
    	printf("weight[%i][0]=%i \t",j,d_tt[0]);
 //   	printf("\n column %i\n",j);

    	vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
    }
    err = hipGetLastError();
    if (err!=0) printf("after vect_to_strip %d , %s \n",err,hipGetErrorString(err));
// для матрицы Cost нужна транспонированная матрица tt
puts("weight");
    int *tc, *ttc;
    tc=new int[VER];
    for (j=0; j<M2; j++)
        {
    	for(i=0; i<M2;i++)
    		{ ttc=tt[i];
    		  tc[i]=ttc[j];
    		 if(j==0) printf("cost[0][%i]=%i \t",i,tc[i]);
    		}
        	hipMemcpy(d_tmp,tc,sizeof(int)*LENGTH1,hipMemcpyHostToDevice);
     //   	printf("\n column %i\n",j);
 //           puts(" --------- ");
        	vect_to_strip<<<H1,NN1>>>(j,d_tmp,d_tab);
        }
}
void initIO()
{
//	if(matrix==1)
	{
		size_tt=VER;
	}
//	else if (weighted==0)
//	{ size_tt=2;}
//	     else size_tt=3;

	tt=new int*[size_tt];
	for(int i=0; i<size_tt;i++) tt[i]=new int[LENGTH1+1];
}
