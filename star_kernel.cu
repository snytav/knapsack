#include "hip/hip_runtime.h"
#include "star_kernel.h"
#include <stdio.h>
#include <stdlib.h>
#define col_bits 128
//#define c_th 100
        const int NN1=(LENGTH1-1)/64+1;
        int threads1=1;//(col_bits<NN1)?col_bits:NN1;
		int blocks1=NN1;//(NN1-1)/threads1+1;  //?15 одномоментно?
//#include "find.h"
//#define index  threadIdx.x + blockIdx.x*blockDim.x

//const int NN=(LENGTH1-1)/SIZE_OF_LONG_INT+1;

		// Fused the diagonal element root and dscal operation into
		// a single "cdiv" operation
		void launchMyKernel(int *array, int arrayCount)
		{
		  int blockSize;   // The launch configurator returned block size
		  int minGridSize; // The minimum grid size needed to achieve the
		                   // maximum occupancy for a full device launch
		  int gridSize;    // The actual grid size needed, based on input size

//		  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
//		                                      MyKernel, 0, 0);
		  // Round up according to array size
		  gridSize = (arrayCount + blockSize - 1) / blockSize;

//		  MyKernel<<< gridSize, blockSize >>>(array, arrayCount);

		  hipDeviceSynchronize();

		  // calculate theoretical occupancy
		  int maxActiveBlocks;
//		  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks,
//		                                                 MyKernel, blockSize,
//	                                                 0);

		  int device;
		  hipDeviceProp_t props;
		  hipGetDevice(&device);
		  hipGetDeviceProperties(&props, device);

		  float occupancy = (maxActiveBlocks * blockSize / props.warpSize) /
		                    (float)(props.maxThreadsPerMultiProcessor /
		                            props.warpSize);

		  printf("Launched blocks of size %d. Theoretical occupancy: %f\n",
		         blockSize, occupancy);
		}


		void fusedDscal()
		{
		    // The semibandwidth (column length) determines
		    // how many warps are required per column of the
		    // matrix.
			hipDeviceProp_t devProp;
			hipGetDeviceProperties ( &devProp, 0);

			int n=NN1;
		    const int warpSize = 32;
		    const int maxGridSize =14;//112; // this is 8 blocks per MP for a Telsa C2050

		    int warpCount = ((n -1)/ warpSize+1);// + (((n % warpSize) == 0) ? 0 : 1);
		    int warpPerBlock = max(1, min(4, warpCount));
		    // For the cdiv kernel, the block size is allowed to grow to
		    // four warps per block, and the block count becomes the warp count over four
		    // or the GPU "fill" whichever is smaller
		    int threadCount = warpSize * warpPerBlock;
		    int blockCount = min( maxGridSize, max(1, (warpCount-1)/warpPerBlock+1) );
//
            int warpInstruction=(NN1-1)/(threadCount*blockCount)+1;
//
		    dim3 BlockDim = dim3(threadCount, 1, 1);
		    dim3 GridDim  = dim3(blockCount, 1, 1);
        printf("calc: blocks=%i, threads=%i ,InsPerThread=%i\n",blockCount,threadCount,warpInstruction);

         printf ( "Compute capability : %d.%d\n", devProp.major, devProp.minor );
         printf ( "Name : %s\n", devProp.name );
         printf ( "Total Global Memory : %d\n", devProp.totalGlobalMem );
         printf ( "Shared memory per block: %d\n", devProp.sharedMemPerBlock );
         printf ( "Registers per block : %d\n", devProp.regsPerBlock );
         printf ( "Warp size : %d\n", devProp.warpSize );
         printf ( "Max threads per block : %d\n", devProp.maxThreadsPerBlock );
         printf ( "Total constant memory : %d\n", devProp.totalConstMem );
         printf("Multiprocessor count: %d\n", devProp.multiProcessorCount);

 //		void launchMyKernel(int *array, NN);

		}
//////////////////////////////////////////////////////////////////////////////////////////

__device__ void _and(unsigned long long int *d_v,unsigned long long int *d_v1)
		{

	if (index<NN1) d_v[index] &= d_v1[index];
		};

////////////////////////////////////////////////
__device__ void _or(unsigned long long int *d_v,unsigned long long int *d_v1)
		{

	if (index<NN1) d_v[index] |= d_v1[index];
		};

////////////////////////////////////////////////
__device__ void _xor(unsigned long long int *d_v,unsigned long long int *d_v1)
		{
	if (index<NN1) d_v[index] ^= d_v1[index];
		};

//////////////////////////////////////////////////

__device__ void _not(unsigned long long int *d_v)
		{
	if (index<NN1) d_v[index] = ~d_v[index];
		};

__device__ int _get_bit(unsigned long long int *d_v,int k)
{
	int num,sh;
	unsigned long long int p = 1;

	num = get_64bit_word(k,SIZE_OF_LONG_INT);
	sh =  position_in_64bit_word(k,SIZE_OF_LONG_INT);
	p = p << (sh-1);
#ifdef bbb
	printf("get_positio_bit n %d num %d sh %d shifted p %llu \n",k,num,d_v[num],p);
#endif
	return (d_v[num] & p ) && 1;
}

__device__ void _mask(unsigned long long int *d_v, int num)
{ unsigned long long int zero;
  int num_el=num>>6; // номер элемента, содержащий переход от 0 к 1;

	  if (index==num_el)
   {
	  zero=1>>(num % SIZE_OF_LONG_INT)-1;
   }
  else
  {
      zero=0;
      if (index>num_el)
      {
    	  zero=!zero;
      }
  }
   d_v[index]=zero;
}

__device__ LongPointer _col(LongPointer *d_tab,int i)
{
	return d_tab[i];
}

__device__ void _clr(unsigned long long int *d_v)
{
	if (index<NN1) d_v[index] = 0;
}

__device__ void _set(unsigned long long int *d_v)
{ unsigned long long int zero = 0;

	if (index<NN1) d_v[index]= ~zero;
}

__device__ void _assign(unsigned long long int *d_v,unsigned long long int *d_u)
{
	if (index<NN1)d_v[index]=d_u[index];
}

__device__ void _assign(unsigned long long int *d_v,unsigned long long int u)
{
	if (index<NN1) d_v[index]=u;
}

__device__ unsigned long long int _assign(unsigned long long int *d_u)
{
	if (index<NN1) return d_u[index];
	else return 0;
}


