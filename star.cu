#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "star.h"
//#include "table.cu"
#include "basic.h"
#include "star_kernel.h"
#include "iostar.h"

__global__ void Warshall_kernel(LongPointer *d_tab,int k,unsigned long long int *d_x)
{       unsigned long long int *d_v,*d_w,w;
        int s;
//        int sizel = M;
//        double d_blocks;
//       unsigned int blocks, threads = M < SIZE_OF_LONG_INT ? M: SIZE_OF_LONG_INT;

//        d_blocks = M;
//        d_blocks = d_blocks/(double)threads;
//       	blocks = (sizel > SIZE_OF_LONG_INT)? (int)ceil( d_blocks) : 1;
//        get_row<<<blocks,threads>>>(d_tab,d_x,k,M);
	    d_w=d_tab[k-1];//tab->GetCol(w,k);
	    w=d_w[blockIdx.x];
    	s=get_position_bit(d_x,threadIdx.x+1);
    	if(s==1)
    	{
    		d_v=d_tab[threadIdx.x];//tab->GetCol(w,k);
    		d_v[blockIdx.x]|=w;

    	}
}

void warshall_c(Table *tab)
{
	Slice *X;//,*w,*v;
	unsigned long long int *d_x;
	LongPointer *d_tab;
	int k,NN;
    unsigned int lgs=tab->length;
    unsigned int sz=tab->size;
    X = new Slice;
  //  w = new Slice;
   // v = new Slice;

    X->Init(lgs);
    NN=X->NN;
  //  w->Init(sz);
  //  v->Init(sz);
    d_tab=tab->get_device_pointer();
    for (k = 1;k <= M;k++)
    {
    	tab->GetRow(X,k); //*X = tab->col(k);
    	d_x=X->get_device_pointer();
    	Warshall_kernel<<<NN,sz>>>(d_tab,k,d_x);
    }
}

__global__ void Warshall_ogr_kernel(LongPointer *d_tab,int k,int *n,unsigned long long int *d_x)
{       unsigned long long int *d_v,*d_w,w;
        int s;
//        int sizel = M;
//        double d_blocks;
//       unsigned int blocks, threads = M < SIZE_OF_LONG_INT ? M: SIZE_OF_LONG_INT;

//        d_blocks = M;
//        d_blocks = d_blocks/(double)threads;
//       	blocks = (sizel > SIZE_OF_LONG_INT)? (int)ceil( d_blocks) : 1;
//        get_row<<<blocks,threads>>>(d_tab,d_x,k,M);
	    d_w=d_tab[k-1];//tab->GetCol(w,k);
	    w=d_w[blockIdx.x];
    	s=get_position_bit(d_x,threadIdx.x+1);
    	if(s==1)
    	{
    		d_v=d_tab[threadIdx.x];//tab->GetCol(w,k);
    		d_v[blockIdx.x]|=w;
    		(*n)++;

    	}
}
void warshall_c_ogr(Table *tab)
{
	Slice *X;//,*w,*v;
	unsigned long long int *d_x;
	LongPointer *d_tab;
	int k,NN,n2,n1,n;
    unsigned int lgs=tab->length;
    unsigned int sz=tab->size;
    X = new Slice;
  //  w = new Slice;
   // v = new Slice;

    X->Init(lgs);
    NN=X->NN;
  //  w->Init(sz);
  //  v->Init(sz);
    d_tab=tab->get_device_pointer();
    n2=0; n1=1;
    for (k = 1;k <= M;k++)
    {   if(n2==n1)
        {k=M+1;}
        else
        {
        n=0;
        tab->GetRow(X,k); //*X = tab->col(k);
    	d_x=X->get_device_pointer();
    	Warshall_ogr_kernel<<<NN,sz>>>(d_tab,k,&n,d_x);
    	printf("k=%i,n=%i \n",k,n);
    	n2=n1;
    	n1=n;
        }
    }
}
__global__ void WarshallDev(LongPointer *d_tab, int k, unsigned long long int *d_x)
{	 unsigned long long int *d_v,*d_w;
	        int s;

		    d_w=d_tab[k-1];//tab->GetCol(w,k);
		    s=_get_bit(d_x,threadIdx.x+1);
	    	//s=get_position_bit(d_x,threadIdx.x+1);
	    	if(s==1)
	    	{
	    		d_v=d_tab[threadIdx.x];//tab->GetCol(w,k);
	    		_or(d_v,d_w);

	    	}
}
void warshall_c2(Table *tab)
{
	Slice *X;//,*w,*v;
	unsigned long long int *d_x;
	LongPointer *d_tab;
	int k,NN;
    unsigned int lgs=tab->length;
    unsigned int sz=tab->size;
    X = new Slice;
  //  w = new Slice;
   // v = new Slice;

    X->Init(lgs);
    NN=X->NN;
  //  w->Init(sz);
  //  v->Init(sz);
    d_tab=tab->get_device_pointer();
    for (k = 1;k <= M;k++)
    {
    	tab->GetRow(X,k); //*X = tab->col(k);
    	d_x=X->get_device_pointer();
    	WarshallDev<<<NN,sz>>>(d_tab,k,d_x);
    }
}

void warshall(Table *tab)
{
	Slice *X,*w,*v;
	int i,k;
    unsigned int lgs=tab->length;
    unsigned int sz=tab->size;
    X = new Slice;
    w = new Slice;
    v = new Slice;

    X->Init(lgs);
    w->Init(sz);
    v->Init(sz);

    for (k = 1;k <= M;k++)
    {
    	tab->GetRow(X,k); //*X = tab->col(k);
    	tab->GetCol(w,k);//*w = tab->row(k);
//    	X->print("X",0);
//    	w->print("w",1);
    	i = X->STEP();
    	while(i>0)
    	{
    		tab->GetCol(v,i);//*v = tab->row(i);
    		v->OR(w);
    		tab->SetCol(v,i);//*tab->row(i)=v;
    		i = X->STEP();
     	}
    }
}

    void warshall_o(Table *tab)
    {
    	Slice *X,*w,*v;
    	int i,k,n2,n1,n;
        unsigned int lgs=tab->length;
        unsigned int sz=tab->size;
        X = new Slice;
        w = new Slice;
        v = new Slice;

        X->Init(lgs);
        w->Init(sz);
        v->Init(sz);


        n2=M*M;
//        n1=1;
        n=0;
        for (k = 1;k <= M;k++)
        {
        	if(n==n2)
        	{
//        		printf("k=%i, n=%i \n",k,n);
        		k=M+1;
        	}
        	else
        	{
        	n=0;
        	tab->GetRow(X,k); //*X = tab->col(k);
        	tab->GetCol(w,k);//*w = tab->row(k);
    //    	X->print("X",0);
    //    	w->print("w",1);
        	i = X->STEP();
        	while(i>0)
        	{
        		tab->col(i)->OR(w);
  /*      		tab->GetCol(v,i);//*v = tab->row(i);
        		v->OR(w);
        		tab->SetCol(v,i);//*tab->row(i)=v;
   */
        		i = X->STEP();
//       		n=n+v->NUMB1();
        	}
 //       	n2=n1;
 //       	n1=n;
    //    	printf("k=%i, n=%i \n",k,n);
            }
        }
    }


__global__ void Triangles_kernel(LongPointer *d_tab,int k,unsigned long long int *d_x)
{       unsigned long long int *d_v,*d_w,w;
   //     int s;
   //     int sizel = M;
//        double d_blocks;
//       unsigned int blocks, threads = M < SIZE_OF_LONG_INT ? M: SIZE_OF_LONG_INT;

//        d_blocks = M;
//        d_blocks = d_blocks/(double)threads;
//       	blocks = (sizel > SIZE_OF_LONG_INT)? (int)ceil( d_blocks) : 1;
//        get_row<<<blocks,threads>>>(d_tab,d_x,k,M);
	    d_w=d_tab[k-1];//tab->GetCol(w,k);
	  _and(d_x,d_w);
}
__global__ void add_count(int *d_count, int *d_numb)
{
//	if (d_numb[0]>0)printf("%i+ %i\n",d_count[0],d_numb[0]);
	*d_count+=*d_numb;
}

__device__ void triangles_copy(LongPointer *d_tab, unsigned long long int *d_and, int i, unsigned long long int *d_res)
{ unsigned long long int *d_col;
  d_col=_col(d_tab,i-1);//d_tab[i];
  _assign(d_res,d_col);
  _and(d_res,d_and);
}
__global__ void triangles_kernel(LongPointer *d_tab, unsigned long long int *d_and, int i, unsigned long long int *d_res)
{ unsigned long long int *d_col;

  d_col=_col(d_tab,i-1);//d_tab[i];
  _assign(d_res,d_col);
  _and(d_res,d_and);
}
void CountTrianglesOPT(Table *tab, int * count)
{ // FILE *f;
  //  if((f = fopen("count_tr.dat","wt")) == NULL) return;
//	(*count)=0;
//	puts("countTr in");
	Slice *X,*Y,*mask;//,*w,*v;
	unsigned long long int *d_x,*d_y,*d_mask;
	LongPointer *d_tab;
	int m,k,j,NN,*d_count_t,* d_numb_x;
    unsigned int lgs=tab->length;
    unsigned int sz=tab->size;
    X = new Slice;
    Y = new Slice;
    mask= new Slice;
   // v = new Slice;

    hipMalloc(&d_count_t,sizeof(int));
    hipMemset(d_count_t,0,sizeof(int));
    hipMalloc(&d_numb_x,sizeof(int));
    X->Init(lgs);
    Y->Init(lgs);
//    mask->Init(lgs);

    NN=X->NN;
  //  w->Init(sz);
  //  v->Init(sz);
    d_tab=tab->get_device_pointer();
    d_x=X->get_device_pointer();
    d_y=Y->get_device_pointer();
//    d_mask=mask->get_device_pointer();
//    puts("OPTIM::count init");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
 //   mask->SET();
    for (m=1; m<M;m++)
    {
 //   	triangles_kernel<<<blocks1,threads1>>>(d_tab,d_mask,m,d_x);
       tab->GetCol(X,m);
 ////   	mask->set(m,0);
//    	sprintf(s,"STEP_dat/mask%04d",m);
//    	mask->print(s,0);
 //   	count2=X->NUMB();
    	//if (count2>0)
//    	fprintf(f,"(%i) m=%i \n",count2,m);
    	j=X->STEP();
    	while(j>0)
    	{
//    	tab->GetCol(Y,j);
 //   	mask->MASK(j);
 //    	Y->AND(mask);
//    	mask->print("mask",0);
 //   	Y->AND(X);
  //  	triangles_kernel<<<blocks1,threads1>>>(d_tab,d_x,j,d_y); внесена в number_plus->copy_block_pluse
  //  	count1=Y->NUMB();//
//
    	number_plus(d_tab,d_x,j,d_y,NN,d_count_t,NN);
//    	count1=Y->NUMB();
//   	printf("m=%i,j=%i  \n",m,j);
 //   	add_count<<<1,1>>>(d_count_t,d_numb_x);
//    	(*count)+=count1;
    	//if (count1>0)
   // 	fprintf(f," count=%i: <%i,%i>:%i\n",*count,m,j,count1);
    	j=X->STEP();
    	}

    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime1, totalTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);

    totalTime1 = elapsedTime1/(1000);

    printf("associative time count of triangles= %f\n", totalTime1);
//    fclose(f);
    hipMemcpy(count,d_count_t,sizeof(int),hipMemcpyDeviceToHost);
}

void CountTriangles(Table *tab, int * count)
{  FILE *f;
    if((f = fopen("count_tr.dat","wt")) == NULL) return;
	(*count)=0;
//	puts("countTr in");
	Slice *X,*Y,*mask;//,*w,*v;
	unsigned long long int *d_x;
	LongPointer *d_tab;
	int m,k,j,NN,*d_count_t,* d_numb_x,count1,count2=0;
    unsigned int lgs=tab->length;
    unsigned int sz=tab->size;
    X = new Slice;
    Y = new Slice;
    mask= new Slice;
   // v = new Slice;

    hipMalloc(&d_count_t,sizeof(int));
    hipMemset(d_count_t,0,sizeof(int));
    hipMalloc(&d_numb_x,sizeof(int));
    X->Init(lgs);
    Y->Init(lgs);
    mask->Init(lgs);

    NN=X->NN;
  //  w->Init(sz);
  //  v->Init(sz);
    d_tab=tab->get_device_pointer();
    d_x=Y->get_device_pointer();
//    puts("count init");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (m=1; m<M;m++)
    {
    	tab->GetCol(X,m); //*X = tab->col(k);
 ////   	mask->MASK(m);
//    	if (m>(M-10))mask->print("mask",0);
  ////  	X->AND(mask);
 //   	count2=X->NUMB();
    	//if (count2>0)
//    	fprintf(f,"(%i) m=%i \n",count2,m);
    	j=X->STEP();
    	while(j>0)
    	{
    	tab->GetCol(Y,j);

 //   	mask->MASK(j);
 //    	Y->AND(mask);
//    	mask->print("mask",0);
    	Y->AND(X);
    	//Triangles_kernel<<<blocks1,threads1>>>(d_tab,j,d_x);
    	count1=Y->NUMB();//
  ////  	number(d_x,NN,d_numb_x,NN);

//    	printf("m=%i,j=%i  ",m,j);
//    	add_count<<<1,1>>>(d_count_t,d_numb_x);
    	(*count)+=count1;
    	if (count1>0)
    	fprintf(f," count=%i: <%i,%i>:%i\n",*count,m,j,count1);
    	j=X->STEP();
    	}

    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime1, totalTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);

    totalTime1 = elapsedTime1/(1000);

    printf("associative time count of triangles= %f\n", totalTime1);
    fclose(f);
//    hipMemcpy(count,d_count_t,sizeof(int),hipMemcpyDeviceToHost);
}


__global__ void arcs(LongPointer *d_r, LongPointer *d_l, unsigned long long int *d_z, unsigned long long int *d_n,unsigned long long int *d_y, unsigned long long int *d_x ,int size)
{		//	    puts("vertex has nomber");
 match(d_r,d_z,d_n,d_y,size);
//   Z->print("Z",0);
//    Y->print("input arcs",0);
//X->print("X was",0);
 d_y[blockIdx.x]=~d_y[blockIdx.x];//Y->NOT();
//	Y->print("not_Y",0);
  d_x[blockIdx.x]&=d_y[blockIdx.x];  //X->AND(Y);
//	puts("the arcs were deleted");
//	X->print("search",0);

 match(d_l,d_x,d_n,d_y,size);
}


void DFS(Table *left, Table *right, Table *code, Slice *root, Table *NV, Slice *T, Slice *X)
{
	Table *LIFO;
	int lif,ord;
	int i;
	Slice *Y,*Z,*U,*V,*w,*node;

	LIFO=new Table;
	LIFO->Init(VER,M); // for saving code of vertices
	Y=new Slice;
	Y->Init(LENGTH1);
	Z=new Slice;
	Z->Init(LENGTH1);
	U=new Slice;
	U->Init(VER);
	V=new Slice;
	V->Init(VER);
	w=new Slice;
	w->Init(M);
	node=new Slice;
	node->Init(M);

	LongPointer *d_r, *d_l;
	unsigned long long int *d_x, *d_y, *d_z, *d_n;
    	int size=left->size;
	 d_x=X->get_device_pointer();
//	 d_n=node->get_device_pointer();
	 d_z=Z->get_device_pointer();
	 d_y=Y->get_device_pointer();
	 d_r=right->get_device_pointer();
	 d_l=left->get_device_pointer();
	 int NN;
	 NN=X->NN;

	Z->SET();
	T->CLR();
	U->SET();
	ord=1;
	lif=1;
	*node=*root;
	do
	{
	 //   node->print("node",0);
		LIFO->SetRow(node,lif);//*LIFO->row(lif)=node;
		 d_n=node->get_device_pointer();
//	puts("node is getting to stek");
		if(ord>VER)exit(0);
/*	    code->GetRow(w,ord);//*w=code->row(ord);
//	    w->print("dfs-number",0);
	    MATCH_CUDA(code,U,node,V);
//	    U->print("U",0);
	    i=V->FND();
//	    V->print("V",0);
//	    printf("ord= %d, lif= %d node %d \t", ord, lif,i);
	 //   printf("node %i %i \t", i, ord);
	    NV->SetRow(w,i);//*NV->row(i)=w;*/
		NV->SetRow(node,ord);//
	    ord++;
/*//	    puts("vertex has nomber");
	    MATCH_CUDA(right,Z,node,Y);
	 //   Z->print("Z",0);
	//    Y->print("input arcs",0);
	   //X->print("X was",0);
		Y->NOT();
	//	Y->print("not_Y",0);
		X->AND(Y);
	//	puts("the arcs were deleted");
	//	X->print("search",0);

		MATCH_CUDA(left,X,node,Y);
	//	X->print("X after MATCH",0);
	//	zero=Y->ZERO();
	//	puts("output arcs");
	//	Y->print("Y_before_Zero",0);
	*/
        arcs<<<NN,1>>>(d_r,d_l,d_z,d_n,d_y,d_x,size);

	    i=Y->FND();
//		printf("FND befor while %i \n",i);
		while((i==0)&&(lif>1))
		{
			lif--;
	//		printf("stek up %i \n",lif);
			LIFO->GetRow(node,lif);//*node=LIFO->row(lif);
		//	node->print("node_up",0);
		//	X->print("X in stec up",0);
			MATCH_CUDA(left,X,node,Y);
//			zero=Y->ZERO();
			i=Y->FND();
	//		printf(" up ord= %d, lif= %d \n", ord, lif);
		}

	//	X->print("X before stek down",0);
    //	Y->print("Y_before_Some",0);
	//	i=Y->FND();
//		printf("FND %i\n ",i);
		if (i>0)
		{	Y->set(i,0);
//		 puts("stek down");
//			 printf("right(%i)\n",i);
			 T->set(i,1);
			 lif++;
			 right->GetRow(node,i); //*node=right->row(i);
		//	 node->print("node_down",0);
	//		 printf("down ord= %d, lif= %d \n", ord, lif);
		}
		//X->print("X before next step",0);
		//T->print("T befor next step",0);
	}
	while(lif>1);
//	printf("ord %i", ord-1);
}

/*
 Procedure DFS(left, right: table; code: table; root: word;
              Var NV: table; Var T, X: slice);
Var LIFO: table; {моделирует стек [N,size(code)]}
    lif,{хранит глубину стека}
    ord, {хранит текущий М-номер}
    i: integer;
    Y, Z: slice{left} ;
    U, V: slice{code} ;
    w, node: word;
Begin
    SET(Z); CLR(T); SET(U);
    ord:=1; lif:=1; node:=root;
    repeat
       ROW(lif,LIFO):=node // Текущая вершина заносится в стек.
{Нумерация текущей вершины.}
       w:=ROW(ord, code);// двоичный код номера
       MATCH(code, U, node, V); i:=FND(V);// позиция вершины
       ROW(i, NV):=w; ord:=ord+1;
{Убираются дуги, ведущие в вершину node.}
       MATCH(right, Z, node, Y);
       Y:=not Y;
       X:=X and Y;
{Поиск дуг, ведущих из вершины node в ненумерованные вершины.}
       MATCH(left, X, node, Y);
{Если таких дуг нет, и текущая вершина не равна root, то поднимаемся по стеку.}
       while ZERO(Y) and (lif>1) do
       begin
          lif:=lif-1;
          node:=ROW(lif, LIFO);
          MATCH(left,X,node,Y);
       end;
{Если есть дуга из текущей вершины в ненумерованную, то заносим ее в дерево
и голова этой дуги становится текущей вершиной. Увеличивается глубина стека}
       if SOME(Y) then
       begin
         i:=STEP(Y);
         T(i):= 1;
         lif:=lif+1;
         node:=ROW(i, right);
       end;
   until lif=1;
End;
 */

void dijkstra1(Table *T,int s,Table *D)
{   int k,h=H1;
	Table *R1;//,*R2;
	LongPointer *t_R1, *t_T,*t_D;//, *t_R2;
	R1= new Table;
	R1->Init(LENGTH1,h);
    t_R1=R1->get_device_pointer();

    t_T=T->get_device_pointer();
    t_D=D->get_device_pointer();

/*	R2= new Table;
	R2->Init(LENGTH1,h);
	t_R2=R2->get_device_pointer();
*/
	Slice *U,*X,*Z,*inf,*v;

	X=new Slice;
	X->Init(LENGTH1);

	Z=new Slice;
	Z->Init(LENGTH1);

	inf=new Slice;
	inf->Init(h);
//	printf("infinit length=%i, elements=%i\n", inf->length,inf->NN);
	inf->SET(); //0x7FFFFFFF

	v=new Slice;
	v->Init(h);

	U=new Slice;
	U->Init(LENGTH1);
	U->SET();
//U->print("Dejk_U1.dat",0);
	U->set(s,0);
//U->print("Dejk_U2.dat",0);
	k=s;
	WCOPY(inf,U,D,H1);
	D->writeToFile("D0.dat");
//	puts("t_D");
//	 printStrip(t_D,0);
/*
	puts("t_T");
	for(int i=0; i<VER; i++)
	{
	 printf("strip %i\n",i);
	 printStrip(t_T,i);
	}
puts("T before copy R3.dat");
//writetoDimageW("graph10Dijk.dat",T);
//T->writeToFile("R3.dat");
//printStrip(t_T,k);
/*/
U->print("Dejk_U.dat",0);
	while (U->SOME())
	{
//printf("vertex %i\n",k);
		TCOPY1(T,k,h,R1); //копирует полосу с номером k шириной h
		R1->writeToFile("R1-1000_bin.dat");
printf("t_R1 k=%i\n",k);
//printStrip(t_R1,0);
printStrip(t_T,k-1);
//		R1->writeToFile("R1.dat");
//printf("T after copy R2.dat and R4.dat ");
//		T->writeToFile("R2.dat");

//		 writetoDimageW("R4.dat",T);
		MATCH_CUDA(R1,U,inf,X);
X->print("X_MATCH.dat",0);
		X->XOR(U);
//		printf("length %i\t",X->length);
X->print("XxorU",0);
		if (U->SOME())
//		if (X->SOME())
		{
			D->GetRow(v,k);
			printf("row(D,%i) ",k);
			v->print("v",1);
			//ADDC(R1,v,X,R2);
			ADDC1(X,v,R1);
//puts("R1+v");
//printStrip(t_R1,0);
			SETMIN(R1,D,X,Z);
			TMERGE(R1,Z,D);
            Z->print("opt_D",0);
puts("###############################new D");
printStrip(t_D,0);
		}
//U->print("U_before_MIN",0);
		MIN(D,U,X);
		X->print("X_MIN.dat",1);
		k=X->FND();
		U->set(k,0);
		U->print("U", 0);
	}
//		writetoDimageW("graph10Dijk.dat",T);
//  		T->writeToFile("graph10Dijk_bin.dat");
}

void dijkstra2(Table *T,int s,Table *D)
{   int k,h=H1;
	Table *R1;//,*R2;
	LongPointer *t_R1, *t_T,*t_D;//, *t_R2;
hipError_t err = hipGetLastError();
printf("before init dijkstra2 %d , %s \n",err,hipGetErrorString(err));
	R1= new Table;
	R1->Init(LENGTH1,h);
    t_R1=R1->get_device_pointer();

    t_T=T->get_device_pointer();
    t_D=D->get_device_pointer();

/*	R2= new Table;
	R2->Init(LENGTH1,h);
	t_R2=R2->get_device_pointer();
*/
	Slice *U,*X,*Z,*P,*inf,*v;

	X=new Slice;
	X->Init(LENGTH1);

	Z=new Slice;
	Z->Init(LENGTH1);

	P=new Slice;
	P->Init(LENGTH1);

	inf=new Slice;
	inf->Init(h);
//	printf("infinit length=%i, elements=%i\n", inf->length,inf->NN);
	inf->SET(); //0x7FFFFFFF

	v=new Slice;
	v->Init(h);

	U=new Slice;
	U->Init(LENGTH1);
	U->SET();
//U->print("Dejk_U1.dat",0);
	U->set(s,0);
	P->set(s,1);
//U->print("Dejk_U2.dat",0);
	k=s;
	WCOPY(inf,U,D,H1);
//	D->writeToFile("D0.dat");
//	puts("t_D");
//	 printStrip(t_D,0);
/*
	puts("t_T");
	for(int i=0; i<VER; i++)
	{
	 printf("strip %i\n",i);
	 printStrip(t_T,i);
	}
puts("T before copy R3.dat");
//writetoDimageW("graph10Dijk.dat",T);
//T->writeToFile("R3.dat");
//printStrip(t_T,k);
/*/
//U->print("Dejk_U.dat",0);
	while (P->SOME())
	{
//printf("vertex %i\n",k);
		TCOPY1(T,k,h,R1); //копирует полосу с номером k шириной h
//		R1->writeToFile("R1-1000_bin.dat");
//printf("t_R1 k=%i\n",k);
//printStrip(t_R1,0);
//printStrip(t_T,k-1);
//		R1->writeToFile("R1.dat");
//printf("T after copy R2.dat and R4.dat ");
//		T->writeToFile("R2.dat");

//		 writetoDimageW("R4.dat",T);
		MATCH_CUDA(R1,U,inf,X);
//X->print("X_MATCH.dat",0);
		X->XOR(U);
//		printf("length %i\t",X->length);
//X->print("XxorU",0);
//		if (U->SOME())
		if (X->SOME())
		{
			D->GetRow(v,k);
//			printf("row(D,%i) ",k);
//			v->print("v",1);
			//ADDC(R1,v,X,R2);
			ADDC1(X,v,R1);
//puts("R1+v");
//printStrip(t_R1,0);
			SETMIN(R1,D,X,Z);
			TMERGE(R1,Z,D);
// Z->print("opt_D",0);
            P->OR(Z);
//puts("###############################new D");
//printStrip(t_D,0);
		}
//P->print("P_before_MIN",0);
		MIN(D,P,X);
//X->print("X_MIN.dat",1);
		k=X->FND();
//		U->set(k,0);
		P->set(k,0);

//err = hipGetLastError();
//printf("before init dijkstra2 %d , %s \n",err,hipGetErrorString(err));
//U->print("U", 0);
	}
//		writetoDimageW("graph10Dijk.dat",T);
//  		T->writeToFile("graph10Dijk_bin.dat");
}

__global__ void dijkstra_opt1(LongPointer *d_T,int k, int h, LongPointer *d_R1,unsigned long long int *d_U,unsigned long long int *d_inf,unsigned long long int *d_X)
{
//(T,k,h,R1,U,inf,X)
	// (LongPointer *d_t, int j, int h, LongPointer *d_f)
	tcopy1(d_T,k,h,d_R1);
	match(d_R1,d_U,d_inf,d_X,h);
	_xor(d_X,d_U);
}

__global__ void dijkstra_opt2(unsigned long long int *d_X,unsigned long long int *d_v,int h,LongPointer *d_R1,LongPointer *d_D,unsigned long long int *d_Z,unsigned long long int *d_P)
{
//	(X,v,R1,D,Z,P)
    			addc1(d_X,d_v,h,d_R1,d_Z);
				setmin(d_R1,d_D,d_X,d_Z);
				tmerge(d_R1,d_Z,d_D);
	            _or(d_P,d_Z);
				d_P[blockIdx.x]|=d_Z[blockIdx.x];
}

void dijkstra2_opt(Table *T,int s,Table *D)
{   int k,h=H1;
	Table *R1;//,*R2;
	LongPointer *t_R1, *t_T,*t_D;//, *t_R2;
	unsigned long long int *t_U,*t_X,*t_inf,*t_Z,*t_P,*t_v;
//hipError_t err = hipGetLastError();
//printf("before init dijkstra2_opt %d , %s \n",err,hipGetErrorString(err));

R1= new Table;
	R1->Init(LENGTH1,h);
    t_R1=R1->get_device_pointer();

    t_T=T->get_device_pointer();
    t_D=D->get_device_pointer();

	Slice *U,*X,*Z,*P,*inf,*v;

	X=new Slice;
	X->Init(LENGTH1);
	t_X=X->get_device_pointer();

	Z=new Slice;
	Z->Init(LENGTH1);
    t_Z=Z->get_device_pointer();

	P=new Slice;
	P->Init(LENGTH1);
	t_P=P->get_device_pointer();

	inf=new Slice;
	inf->Init(h);
	t_inf=inf->get_device_pointer();
//	printf("infinit length=%i, elements=%i\n", inf->length,inf->NN);
	inf->SET(); //0x7FFFFFFF

	v=new Slice;
	v->Init(h);
    t_v=v->get_device_pointer();

	U=new Slice;
	U->Init(LENGTH1);
	t_U=U->get_device_pointer();
	U->SET();
//U->print("Dejk_U1.dat",0);
	U->set(s,0);
	P->set(s,1);
//U->print("Dejk_U2.dat",0);
	k=s;
//err = hipGetLastError();
//printf("before dijkstra2_opt1 %d , %s \n",err,hipGetErrorString(err));
	WCOPY(inf,U,D,H1);
	while (P->SOME())
	{
/* ---dijkstra_opt1(T,k,h,R1,U,inf,X)
		TCOPY1(T,k,h,R1); //копирует полосу с номером k шириной h
		MATCH_CUDA(R1,U,inf,X);
		X->XOR(U);
*/
		dijkstra_opt1<<<NN1,1>>>(t_T,k,h,t_R1,t_U,t_inf,t_X);
//		puts("opt1 ");
//		X->print("X_2",0);
//err = hipGetLastError();
//printf("after dijkstra_opt1 %d , %s \n",err,hipGetErrorString(err));
//		if (U->SOME())
		if (X->SOME())
		{
			D->GetRow(v,k);

//v->print("v",0);
//            ADDC1(X,v,R1);
			dijkstra_opt2<<<NN1,1>>>(t_X,t_v,h,t_R1,t_D,t_Z,t_P);
//			puts("opt2 ");
//err = hipGetLastError();
//printf("after dijkstra_opt2 %d , %s \n",err,hipGetErrorString(err));
/*///	---dijkstra_opt2(X,v,R1,D,Z,P)
			ADDC1(X,v,R1);
			SETMIN(R1,D,X,Z);
			TMARGE(R1,Z,D);
            P->OR(Z);
*/
		}
//		err = hipGetLastError();
//		printf("before print %d , %s \n",err,hipGetErrorString(err));
//	P->print("P_opt",0);
//	err = hipGetLastError();
//	printf("after print %d , %s \n",err,hipGetErrorString(err));
		MIN(D,P,X);
//X->print("X_opt_MIN",0);
//err = hipGetLastError();
//printf("after MIN %d , %s \n",err,hipGetErrorString(err));
		k=X->FND();
//err = hipGetLastError();
//printf("after FND %d , %s \n",err,hipGetErrorString(err));
//		U->set(k,0);
//printf("k=%i\n",k);
		P->set(k,0);
//		P->print("P_af_set0",0);
	}
}
