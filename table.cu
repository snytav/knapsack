#include "hip/hip_runtime.h"
#include "find.h"
#include "table.h"
#include "param.h"
#include <stdio.h>
//#include "cuPrintf.cu"

 extern char **tb;//[LENGTH1][M]![LENGTH1][LENGTH1+1];  //table transposed

//#define ttt
int Table::Init(unsigned int lg,unsigned int sz)
{
	slice_device_pointer_table = (LongPointer *)malloc(sz*sizeof(LongPointer));
	hipMalloc(&d_slice_device_pointer_table,sz*sizeof(LongPointer));
    length=lg;
    size=sz;
//	puts("Table INIT");
	for(int i = 0;i < sz;i++)
	{
		table[i].Init(lg);
//		printf("%i: %p\n",i+1, table[i].get_device_pointer());
	}
//	puts("Table INIT-2");

	InitDevicePointerTable();
//	puts("Table INIT-3");

	return 0;
}

__global__ void printPointer(LongPointer *p)
{
	unsigned long long *u = p[threadIdx.x];

	printf("printPointer %u %p %llu\n",threadIdx.x,p[threadIdx.x],*u);
//	printf("%u %llu\n",threadIdx.x,p[threadIdx.x]);
}

void Table::InitDevicePointerTable()
{

	for(int i = 0;i < size;i++)
	{
#ifdef ttt
		printf("slice_device_pointer_table[i] *************** %d \n",i);
#endif
		slice_device_pointer_table[i] = table[i].get_device_pointer();
#ifdef ttt
		printf("slice_device_pointer_table[i] *************** %d \n",i);
		printf("slice_device_pointer_table[i] *************** %d %p\n",i,slice_device_pointer_table[i]);
#endif
	}
	hipMalloc(&d_slice_device_pointer_table,size*sizeof(unsigned long long int *));
	hipMemcpy(d_slice_device_pointer_table,slice_device_pointer_table,size*sizeof(unsigned long long int *),
			hipMemcpyHostToDevice);
#ifdef ttt
//	cudaPrintfInit();
//	printPointer<<<1,64>>>(d_slice_device_pointer_table);
//	    cudaPrintfDisplay(stdout, true);
//	    cudaPrintfEnd();
//	printPointer<<<1,64>>>(d_slice_device_pointer_table);
#endif
}

__global__ void get_row(LongPointer *p,unsigned long long *d_v,int i,unsigned int size)
{
//	char s[100];
	 __shared__ unsigned long long  int tmp[SIZE_OF_LONG_INT];
//	cuPrintf("get_row \n");
//	return;

	 unsigned int n1 = threadIdx.x + blockIdx.x*blockDim.x;
	 tmp[threadIdx.x] = 0;
	 if(n1 >= size ) return;
//	 return;
	 unsigned long long *d_rhs = p[n1];
#ifdef ttt
	 cuPrintf("get_row1.5 %u %p \n",n1,p[n1]);
//	 return;
	 cuPrintf("get_row2 %u %p \n",n1,d_rhs);
//	 return;
#endif
//	long_to_binary(*d_rhs,s,SIZE_OF_LONG_INT);
//	printf("get_row %s\n",s);
//	return;
	//assign_bit(d_v,threadIdx.x,n,OR);
	unsigned long long int n = get_position_bit(d_rhs,i);
//	return;
//	cuPrintf("i %d blockIdx.x %d threadIdx.x %d n %d \n",i,blockIdx.x,threadIdx.x,n);
//    return;
	int ni;

	ni = blockIdx.x;//get_64bit_word(threadIdx.x,SIZE_OF_LONG_INT);
	d_v[ni] = 0;

	int pos = position_in_64bit_word(threadIdx.x,SIZE_OF_LONG_INT);
	unsigned long long int u = n << pos;
#ifdef ttt
	cuPrintf("threadIdx.x %d ni%d pos %d n << pos %d\n",threadIdx.x,ni,pos,(int)u);
#endif
	tmp[threadIdx.x] = u;
#ifdef ttt
	cuPrintf("threadIdx.x %d %d getarray %d \n",threadIdx.x,(int)(tmp[threadIdx.x]),
			(int)(get_array(tmp,threadIdx.x,SIZE_OF_LONG_INT)));
#endif
    int M1=blockDim.x;//SIZE_OF_LONG_INT;
//	printf("before %d %luu \n",ni, d_v[ni]);
    d_v[ni] =  get_array(tmp,0,M1)
  	    	    	|  get_array(tmp,1,M1)
  	    	    	|  get_array(tmp,2,M1)
  	    	    	|  get_array(tmp,3,M1)
  	    	        |  get_array(tmp,4,M1)
  	    	        |  get_array(tmp,5,M1)
  	    	        |  get_array(tmp,6,M1)
  			        |  get_array(tmp,7,M1)
  	         		|  get_array(tmp,8,M1)
  			        |  get_array(tmp,9,M1)
  	                |  get_array(tmp,10,M1)
  	                |  get_array(tmp,11,M1)
  	                |  get_array(tmp,12,M1)
	                    |  get_array(tmp,13,M1)
	                    |  get_array(tmp,14,M1)
	                    |  get_array(tmp,15,M1)
                      |  get_array(tmp,16,M1)
                      |  get_array(tmp,17,M1)
                      |  get_array(tmp,18,M1)
          			|  get_array(tmp,19,M1)
			            |  get_array(tmp,20,M1)
		             	|  get_array(tmp,21,M1)
                      |  get_array(tmp,22,M1)
                      |  get_array(tmp,23,M1)
                      |  get_array(tmp,24,M1)
   	                |  get_array(tmp,25,M1)
	                    |  get_array(tmp,26,M1)
	                    |  get_array(tmp,27,M1)
                      |  get_array(tmp,28,M1)
                      |  get_array(tmp,29,M1)
                      |  get_array(tmp,30,M1)
	                    |  get_array(tmp,31,M1)
		                |  get_array(tmp,32,M1)
		                |  get_array(tmp,33,M1)
                      |  get_array(tmp,34,M1)
                      |  get_array(tmp,35,M1)
                      |  get_array(tmp,36,M1)
   	                |  get_array(tmp,37,M1)
	                    |  get_array(tmp,38,M1)
	                    |  get_array(tmp,39,M1)
                      |  get_array(tmp,40,M1)
                      |  get_array(tmp,41,M1)
                      |  get_array(tmp,42,M1)
			            |  get_array(tmp,43,M1)
	            		|  get_array(tmp,44,M1)
	             		|  get_array(tmp,45,M1)
                      |  get_array(tmp,46,M1)
                      |  get_array(tmp,47,M1)
                      |  get_array(tmp,48,M1)
    	                |  get_array(tmp,49,M1)
	                    |  get_array(tmp,50,M1)
	                    |  get_array(tmp,51,M1)
                      |  get_array(tmp,52,M1)
                      |  get_array(tmp,53,M1)
                      |  get_array(tmp,54,M1)
		                |  get_array(tmp,55,M1)
	         	        |  get_array(tmp,56,M1)
		                |  get_array(tmp,57,M1)
                      |  get_array(tmp,58,M1)
                      |  get_array(tmp,59,M1)
                      |  get_array(tmp,60,M1)
                      |  get_array(tmp,61,M1)
                      |  get_array(tmp,62,M1)
                      |  get_array(tmp,63,M1);

#ifdef ttt
   printf("%d %d: %d %d get_array %llu \n",blockIdx.x,blockDim.x, threadIdx.x,ni, get_array(tmp,threadIdx.x,M1));
#endif
	//assign_bit(d_v,threadIdx.x,n,OR);
//	long_to_binary(d_v[ni],s,M);
//	printf("get_row_res %s \n",s);

}
__global__ void get_row_opt(LongPointer *p,unsigned long long *d_v,int i,unsigned int size)
{
//	char s[100];
	 __shared__ unsigned long long  int tmp[SIZE_OF_LONG_INT];
//	cuPrintf("get_row \n");
//	return;
     unsigned long long int tmp_half[2];
	 unsigned int n1 = threadIdx.x + blockIdx.x*blockDim.x;
	 tmp[threadIdx.x] = 0;
	 if(n1 >= size ) return;
//	 return;
	 unsigned long long *d_rhs = p[n1];
#ifdef ttt
	 cuPrintf("get_row1.5 %u %p \n",n1,p[n1]);
//	 return;
	 cuPrintf("get_row2 %u %p \n",n1,d_rhs);
//	 return;
#endif
//	long_to_binary(*d_rhs,s,SIZE_OF_LONG_INT);
//	printf("get_row %s\n",s);
//	return;
	//assign_bit(d_v,threadIdx.x,n,OR);
	unsigned long long int n = get_position_bit(d_rhs,i);
//	return;
//	cuPrintf("i %d blockIdx.x %d threadIdx.x %d n %d \n",i,blockIdx.x,threadIdx.x,n);
//    return;
	int ni;

	ni = blockIdx.x;//get_64bit_word(threadIdx.x,SIZE_OF_LONG_INT);
	d_v[ni] = 0;

	int pos = position_in_64bit_word(threadIdx.x,SIZE_OF_LONG_INT);
	unsigned long long int u = n << pos;
#ifdef ttt
	cuPrintf("threadIdx.x %d ni%d pos %d n << pos %d\n",threadIdx.x,ni,pos,(int)u);
#endif
	tmp[threadIdx.x] = u;
#ifdef ttt
	cuPrintf("threadIdx.x %d %d getarray %d \n",threadIdx.x,(int)(tmp[threadIdx.x]),
			(int)(get_array(tmp,threadIdx.x,SIZE_OF_LONG_INT)));
#endif
    int M1=blockDim.x;//SIZE_OF_LONG_INT;
//	printf("before %d %luu \n",ni, d_v[ni]);
    if (threadIdx.x<32)
    		{tmp_half[0]=get_array(tmp,0,M1)
	    	|  get_array(tmp,1,M1)
	    	|  get_array(tmp,2,M1)
	    	|  get_array(tmp,3,M1)
	        |  get_array(tmp,4,M1)
	        |  get_array(tmp,5,M1)
	        |  get_array(tmp,6,M1)
	        |  get_array(tmp,7,M1)
   		|  get_array(tmp,8,M1)
	        |  get_array(tmp,9,M1)
          |  get_array(tmp,10,M1)
          |  get_array(tmp,11,M1)
          |  get_array(tmp,12,M1)
            |  get_array(tmp,13,M1)
            |  get_array(tmp,14,M1)
            |  get_array(tmp,15,M1)
          |  get_array(tmp,16,M1)
          |  get_array(tmp,17,M1)
          |  get_array(tmp,18,M1)
			|  get_array(tmp,19,M1)
            |  get_array(tmp,20,M1)
         	|  get_array(tmp,21,M1)
          |  get_array(tmp,22,M1)
          |  get_array(tmp,23,M1)
          |  get_array(tmp,24,M1)
           |  get_array(tmp,25,M1)
            |  get_array(tmp,26,M1)
            |  get_array(tmp,27,M1)
          |  get_array(tmp,28,M1)
          |  get_array(tmp,29,M1)
          |  get_array(tmp,30,M1)
            |  get_array(tmp,31,M1);
    		}
    else
    {
    	tmp_half[1]=get_array(tmp,32,M1)
				                |  get_array(tmp,33,M1)
		                      |  get_array(tmp,34,M1)
		                      |  get_array(tmp,35,M1)
		                      |  get_array(tmp,36,M1)
		   	                |  get_array(tmp,37,M1)
			                    |  get_array(tmp,38,M1)
			                    |  get_array(tmp,39,M1)
		                      |  get_array(tmp,40,M1)
		                      |  get_array(tmp,41,M1)
		                      |  get_array(tmp,42,M1)
					            |  get_array(tmp,43,M1)
			            		|  get_array(tmp,44,M1)
			             		|  get_array(tmp,45,M1)
		                      |  get_array(tmp,46,M1)
		                      |  get_array(tmp,47,M1)
		                      |  get_array(tmp,48,M1)
		    	                |  get_array(tmp,49,M1)
			                    |  get_array(tmp,50,M1)
			                    |  get_array(tmp,51,M1)
		                      |  get_array(tmp,52,M1)
		                      |  get_array(tmp,53,M1)
		                      |  get_array(tmp,54,M1)
				                |  get_array(tmp,55,M1)
			         	        |  get_array(tmp,56,M1)
				                |  get_array(tmp,57,M1)
		                      |  get_array(tmp,58,M1)
		                      |  get_array(tmp,59,M1)
		                      |  get_array(tmp,60,M1)
		                      |  get_array(tmp,61,M1)
		                      |  get_array(tmp,62,M1)
		                      |  get_array(tmp,63,M1);
    }
    d_v[ni] =  tmp_half[0]|tmp_half[1];

#ifdef ttt
   printf("%d %d: %d %d get_array %llu \n",blockIdx.x,blockDim.x, threadIdx.x,ni, get_array(tmp,threadIdx.x,M1));
#endif
	//assign_bit(d_v,threadIdx.x,n,OR);
//	long_to_binary(d_v[ni],s,M);
//	printf("get_row_res %s \n",s);

}

Slice *Table::row(int i)
{
	Slice *s;
	unsigned int M2;
	unsigned long long *d_v;
//	puts("row");
//	i--;
    M2=size;
	s = new Slice();
	s->Init(size);
//	printf("col: length %u, %u \n",s->length,s->NN);
    d_v = s->get_device_pointer();
#ifdef ttt
    printf("row2 %p,%p\n",s,d_v);
#endif
    int sizel = size; //size;
    double d_blocks;
   unsigned int blocks, threads = M2 < SIZE_OF_LONG_INT ? M2: SIZE_OF_LONG_INT;

    d_blocks = M2;
    d_blocks = d_blocks/(double)threads;
   	blocks = (sizel > SIZE_OF_LONG_INT)? (int)ceil( d_blocks) : 1;
//   	printf("col: blocks %u; threads %u \n",blocks,threads);
#ifdef ttt
   	for(int i = 0;i < length;i++)
   	{
   		printf("d_slice_device_pointer_table %d %p \n",i,slice_device_pointer_table[i]);
   	}
//#ifdef ttt
   	cudaPrintfInit();
   	printPointer<<<1,64>>>(d_slice_device_pointer_table);
    cudaPrintfDisplay(stdout, true);
       cudaPrintfEnd();

   	cudaPrintfInit();
#endif
 //   int num,sh;
//     	num = get_64bit_word(i,SIZE_OF_LONG_INT);
 //  		sh =  position_in_64bit_word(i,SIZE_OF_LONG_INT);
//   	printf("num %d %d, sh %d, %d \n",i, num,sh,i%64);

   	get_row<<<blocks,threads>>>(d_slice_device_pointer_table,d_v,i,length);
#ifdef ttt
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif
//#ifdef ttt
//    s->get_device_pointer()=d_v;
//     printf("col3 %p,%p\n",s,d_v);
//#endif
//    s->print("col-",0);
    return s;
}
void Table::GetRow(Slice *s,int i)
{
	unsigned int M2;
	unsigned long long *d_v;
//	puts("row");
//	i--;
    M2=size;
//	printf("col: length %u, %u \n",s->length,s->NN);
    d_v = s->get_device_pointer();
#ifdef ttt
    printf("row2 %p,%p\n",s,d_v);
#endif
    int sizel = size;
    double d_blocks;
   unsigned int blocks, threads = M2 < SIZE_OF_LONG_INT ? M2: SIZE_OF_LONG_INT;

    d_blocks = M2;
    d_blocks = d_blocks/(double)threads;
   	blocks = (sizel > SIZE_OF_LONG_INT)? (int)ceil( d_blocks) : 1;
 //  	printf("col: blocks %u; threads %u \n",blocks,threads);
#ifdef ttt
   	for(int i = 0;i < length;i++)
   	{
   		printf("d_slice_device_pointer_table %d %p \n",i,slice_device_pointer_table[i]);
   	}
//#ifdef ttt
   	cudaPrintfInit();
   	printPointer<<<1,64>>>(d_slice_device_pointer_table);
    cudaPrintfDisplay(stdout, true);
       cudaPrintfEnd();

   	cudaPrintfInit();
#endif
 //   int num,sh;
//     	num = get_64bit_word(i,SIZE_OF_LONG_INT);
 //  		sh =  position_in_64bit_word(i,SIZE_OF_LONG_INT);
//   	printf("num %d %d, sh %d, %d \n",i, num,sh,i%64);

   	get_row<<<blocks,threads>>>(d_slice_device_pointer_table,d_v,i,size);
#ifdef ttt
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif
//#ifdef ttt
//    s->get_device_pointer()=d_v;
//     printf("col3 %p,%p\n",s,d_v);
//#endif
//    s->print("col-",0);
//    return s;
}
void Table::GetRow_opt(Slice *s,int i)
{
	unsigned int M2;
	unsigned long long *d_v;
//	puts("row");
//	i--;
    M2=size;
//	printf("col: length %u, %u \n",s->length,s->NN);
    d_v = s->get_device_pointer();
#ifdef ttt
    printf("row2 %p,%p\n",s,d_v);
#endif
    int sizel = size;
    double d_blocks;
   unsigned int blocks, threads = M2 < SIZE_OF_LONG_INT ? M2: SIZE_OF_LONG_INT;

    d_blocks = M2;
    d_blocks = d_blocks/(double)threads;
   	blocks = (sizel > SIZE_OF_LONG_INT)? (int)ceil( d_blocks) : 1;
 //  	printf("col: blocks %u; threads %u \n",blocks,threads);
#ifdef ttt
   	for(int i = 0;i < length;i++)
   	{
   		printf("d_slice_device_pointer_table %d %p \n",i,slice_device_pointer_table[i]);
   	}
//#ifdef ttt
   	cudaPrintfInit();
   	printPointer<<<1,64>>>(d_slice_device_pointer_table);
    cudaPrintfDisplay(stdout, true);
       cudaPrintfEnd();

   	cudaPrintfInit();
#endif
 //   int num,sh;
//     	num = get_64bit_word(i,SIZE_OF_LONG_INT);
 //  		sh =  position_in_64bit_word(i,SIZE_OF_LONG_INT);
//   	printf("num %d %d, sh %d, %d \n",i, num,sh,i%64);

   	get_row_opt<<<blocks,threads>>>(d_slice_device_pointer_table,d_v,i,size);
#ifdef ttt
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif
//#ifdef ttt
//    s->get_device_pointer()=d_v;
//     printf("col3 %p,%p\n",s,d_v);
//#endif
//    s->print("col-",0);
//    return s;
}
void Table::SetCol(Slice *s,int i)
{   unsigned int NN=s->NN;
	unsigned long long int *d_v2,*d_v1;
	d_v2=slice_device_pointer_table[i-1];
	d_v1=s->get_device_pointer();
	set_kernel<<<NN,1>>>(d_v2,d_v1);
}
void Table::GetCol(Slice *s,int i)
{
		unsigned int NN=s->NN;
			unsigned long long int *d_v2,*d_v1;
			d_v2=slice_device_pointer_table[i-1];
			d_v1=s->get_device_pointer();
//			printf("GetRoW %u, row %p,s %p",NN,d_v2,d_v1);
			set_kernel<<<blocks1,threads1>>>(d_v1,d_v2);
}
void Table::readFromFile(char *fn)
{
//	char tb[M][LENGTH+1];  //table transposed
	char str[LENGTH1+1];
	FILE *f;
	int   n = 0;
    int M2=size;
	if((f = fopen(fn,"rt")) == NULL) return;

	while(fgets(str,2*M2,f) != NULL)
	{
		//puts(str);
//		printf("reading %d line ",n);
		for(int i = 0;i < M2;i++)
		{
			tb[n][i] = str[i];
//			printf("%c",tb[n][i]);
		}
//		printf("\n");
/*		for(int i = 0;i < length;i++)
		{
			tb[i][M2] = 0;
		}*/
		tb[n][M2] = 0;
		n++;
	}
	fclose(f);
//	printf("was read \n");
//	return;
	Slice *s;
	s=new Slice;
	s->Init(length);
	for(int i = 0;i < M2;i++)
	{

//		printf("copying %d slice ",i);
	for(int j = 0;j < length;j++)
		{
			str[j] = tb[j][i];
//			printf("%c",tb[j][i]);
		}
		str[LENGTH1] = 0;
//        printf("\n");

		printf("slice %d %s \n",i,str);
//		s = &(table[i]);
		*s = str;
		puts("tab");
		s->print("ss",0);
/*#ifdef ttt
		sprintf(fname,"before_slice%02d",i);
		s->print(fname,1);
		s->print(fname,0);


		unsigned long long int *d_v,*h_v;
		d_v = s->get_device_pointer();
        h_v = (unsigned long long *)malloc(N*sizeof(unsigned long long));
		hipError_t err = hipMemcpy(h_v,d_v,N*sizeof(unsigned long long),hipMemcpyDeviceToHost);
//        printf("string from slice ");
//
//		for(int j = 0;j < N;j++)
//		{
//            long_to_binary(h_v[j],str);
//            printf("%s",str);
//		}
//		printf("\n");
#endif */


//		sprintf(fname,"slice%02d",i);
//		s->print(fname,1);
//		s->print(fname,0);
	}
#ifdef tttt
    puts("end read from file\n");
#endif
}
void Table::readFromFileListAd_or(char *fn,int *eds)
{
	int ras = 100;// max(100, 2 * VER);
	char str[100];
	FILE *f;
	int   n,i,k,l;
	int M2=size;
	*eds=0;

//	puts(" begin set by 0\n");
    for (l=0; l<M2;l++)
    {
    for(int i = 0;i < length;i++)
	      {
	      	tb[i][l] ='0';
	      }
    }
	if((f = fopen(fn,"rt")) == NULL) return;
//	puts("set by 0\n");
	while(fgets(str,ras,f) != NULL)
	{
//		puts(str);
/*
		for(int i = 0;i < M;i++)
		{
			tb[n][i] = str[i];
//			printf("%c",tb[n][i]);
		}
//		printf("\n");
		for(int i = 0;i < LENGTH;i++)
		{
			tb[i][M] = 0;
		}
		n++;
*/

      if(str[0]!='#')
      {
		n=atoi(str);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}
		n--;
		i=atoi(str+l);
		i--;
//		printf("<%i,%i> \n",n,i);
		if ((i<M2) && (n<M2)){
			tb[n][i]='1';
			(*eds)++;
		}
      }
      for(int i = 0;i < length;i++)
      {
      	tb[i][M2] = 0;
      }
//      puts("-");
	}
	fclose(f);
//	puts("set by  list\n");
//	printf("\n");
//	return;
	Slice *s;
	s=new Slice;
	s->Init(size);
	for(int i = 0;i < size;i++)
	{
//		printf("copying %d slice ",i);
	for(int j = 0;j < length;j++)
		{
			str[j] = tb[i][j];
//			printf("%c",tb[j][i]);
		}
//		str[M] = 0;
//        printf("\n");

//		printf("slice %d %s \n",i,str);
		s = &(table[i]);
		*s = str;
//		puts(str);
/*#ifdef ttt
		sprintf(fname,"before_slice%02d",i);
		s->print(fname,1);
		s->print(fname,0);


		unsigned long long int *d_v,*h_v;
		d_v = s->get_device_pointer();
        h_v = (unsigned long long *)malloc(N*sizeof(unsigned long long));
		hipError_t err = hipMemcpy(h_v,d_v,N*sizeof(unsigned long long),hipMemcpyDeviceToHost);
#endif */

//		s->print("read",0);
	}
//    puts("end read from file\n");
}

void Table::readFromFileListAd_unor(char *fn,int *eds)
{

	char str[2*VER];
	FILE *f;
	int   n,i,k,l;
	int M2=size;
	*eds=0;

//	puts(" begin set by 0\n");
    for (l=0; l<M2;l++)
    {
    for(int i = 0;i < length;i++)
	      {
	      	tb[i][l] ='0';
	      }
    }
	if((f = fopen(fn,"rt")) == NULL)
	{
	puts("file error");
	return;
	}
	puts("set by 0\n");
	while(fgets(str,2*M2,f) != NULL)
	{
/*
		//puts(str);

		for(int i = 0;i < M;i++)
		{
			tb[n][i] = str[i];
//			printf("%c",tb[n][i]);
		}
//		printf("\n");
		for(int i = 0;i < LENGTH;i++)
		{
			tb[i][M] = 0;
		}
		n++;
*/

      if((str[0]!='#')&(str[0]!='%'))
      {
		n=atoi(str);
		k=n;
		l=1;
		while (k>0)
		{
			k=k/10;
			l++;
		}
		n--;
		i=atoi(str+l);
		i--;
		if ((i<M2) && (n<M2)){
			if (n<i) tb[n][i]='1';
			if (i<n) tb[i][n]='1';
			(*eds)++;
		}
      }
      for(int i = 0;i < length;i++)
      {
      	tb[i][M2] = 0;
      }
	}
	fclose(f);
//	puts("set by  list\n");
//	printf("\n");
//	return;
	Slice *s;
	s=new Slice;
	s->Init(size);
	for(int i = 0;i < size;i++)
	{
//		printf("copying %d slice ",i);
	for(int j = 0;j < length;j++)
		{
			str[j] = tb[i][j];
//			printf("%c",tb[j][i]);
		}
//		str[M] = 0;
//        printf("\n");

//		printf("slice %d %s \n",i,str);
		s = &(table[i]);
		*s = str;
//		puts(str);
/*#ifdef ttt
		sprintf(fname,"before_slice%02d",i);
		s->print(fname,1);
		s->print(fname,0);


		unsigned long long int *d_v,*h_v;
		d_v = s->get_device_pointer();
        h_v = (unsigned long long *)malloc(N*sizeof(unsigned long long));
		hipError_t err = hipMemcpy(h_v,d_v,N*sizeof(unsigned long long),hipMemcpyDeviceToHost);
#endif */

//		s->print("read",0);
	}
//    puts("end read from file\n");
}

void Table::writeToFile(char *fn)
{
//	char tb[M][LENGTH+1];  //table transposed
//	char str[NN1*64]; //LENGTH1+1 дает переполнение стека
	FILE *f;
	int   n = 0;

//	printf("write %d %d", length,M2);
	if((f = fopen(fn,"wt")) == NULL)return;
	Slice *s;
	 s=new Slice;
	 s->Init(size); //row
	for(n = 0;n < size;n++)
	{
		s = &(table[n]);
//		GetRow(s,n+1);
//   printf("n=%d",n);
//#ifdef tt
		s->convert_to_string(str);

//		s->print("after_slice",0);
//		s->print("after_slice",1);
//		puts(str);
//#endif
//		puts(str);
		for(int i = 0;i < length;i++)
		{
			tb[n][i] = str[i];
//			fprintf(f,"%c",tb[n][i]);
		}
//		fprintf(f,"\n");
	}
//    puts("table ended");
	for(n = 0;n < length;n++)
	{

		for(int i = 0;i < size;i++)
		{
			fprintf(f,"%c",tb[i][n]);
//			printf("<%i,%i>%c\t",i,n,tb[i][n]);
		}
		fprintf(f,"\n");
	}
	puts("print all");

	fclose(f);
	puts("file closed");
}

__global__ void set_row(LongPointer *p,int i,unsigned long long int *d_v, int size)
{
#ifdef QQ
	char s[100];
	long_to_binary(d_v[0],s);
	printf("s %s\n",s);
#endif
//	printf("qq\n");
//	return;
	int index=threadIdx.x + blockIdx.x*blockDim.x;//blockIdx.x
	if (index>size-1) return;
	unsigned long long *d_rhs = p[index];
	int n = get_position_bit(d_v,index+1);
//#ifdef ttt
//	printf("threadIdx.x %d %d n %d \n",blockIdx.x,i,n);
//#endif
//	long_to_binary(*d_rhs,s);

	assign_bit(d_rhs,i,n,SET);
}

void Table::SetRow(Slice *s,int i)
{
//	i--;

//	s->print("v_row_in",1);
	unsigned long long int *d_v = s->get_device_pointer();
	 hipError_t err;
#ifdef ttt
   unsigned long long int *h_v;
   char str[100];

   h_v = (unsigned long long int *)malloc(N1*sizeof(unsigned long long int));

   err = hipMemcpy(h_v,d_v,N1*sizeof(unsigned long long int),hipMemcpyDeviceToHost);

   long_to_binary(h_v[0],str,length);
   printf("entering SetRow err %d %s %s pointer %p,%p\n",err,hipGetErrorString(err),str,s,d_v);
#endif
    int size1 = size;
   unsigned int blocks, threads =size1< SIZE_OF_LONG_INT ? size1: SIZE_OF_LONG_INT;//1;



  	blocks =(size1 > SIZE_OF_LONG_INT)? (int)ceil( ((double)size1)/threads) : 1;//size1;
#ifdef ttt
    printf("SetRow %d %d i %d \n",blocks,threads,i);

    err = hipGetLastError();

    printf("eerr %d %s\n",err,hipGetErrorString(err));
#endif

    //slice addressing function again needs 1-based numbers
    set_row<<<blocks,threads>>>(d_slice_device_pointer_table,i,d_v,size1);
  //  printf("blocks %i, threads %i \n", blocks,threads);
    hipDeviceSynchronize();

    err = hipGetLastError();
#ifdef ttt
    printf("eerr %d %s\n",err,hipGetErrorString(err));
#endif
//   6 exit(0);
}

void readFromFileListLR(char *fn,int *eds, Table *left, Table *right)
{
	char str[50],sleft[65], sright[65];
	FILE *f;
	unsigned long long int   n,i;
	int k,j,num, edd;
	int M2=left->size;
	num=left->length;
	Slice *s;
	s =new Slice;
	s->Init(M);
	edd=1; //number of edges;

	//	puts(" begin set by 0\n");
/*
	for (l=0; l<M2;l++)
    {
    for(int i = 0;i < length;i++)
	      {
	      	tb[i][l] ='0';
	      }
    }
 */
	if((f = fopen(fn,"rt")) == NULL) return;
//	puts("set by 0\n");

	while((fgets(str,100,f) != NULL) && (edd<=num))
	{
      if(str[0]!='#')
      {
    	// puts(str);
		n=atoi(str);
		k=n;
		j=1;
		while (k>0)
		{
			k=k/10;
			j++;
		}
		i=atoi(str+j);
		if ((i<= VER) && (n<=VER))
		{
//			tb[i][n]='1';
			long_to_binary1(n,sleft,M2);
			long_to_binary1(i,sright,M2);
//			printf("left %d: %llu " ,edd, n);
//			puts(sleft);
//			printf("right %llu ",i);
//			puts(sright);

			*s=sleft;
			left->SetRow(s,edd);
			*s=sright;
			right->SetRow(s,edd);
			edd++;
			if(i==0)
				{
				printf("read %d %llu %llu\n",edd,n,i);
				exit(0);
				}
		}
      }
 /*     else
      {
    	  puts(str);
      }*/
	}
	*eds=edd;
	fclose(f);
//    puts("end read from file\n");

}
