#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/copy.h>
#include <algorithm>
#include <time.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

#include <thrust/detail/config.h>

//#include <thrust/system/cuda/detail/cuda_launch_config.h>
#include <thrust/tuple.h>

#include "param.h"

#define LOOPS 100
using namespace thrust::placeholders;

inline void find_min_max(thrust::device_vector<int> &dev_vec, int *min, int *max){
    thrust::pair<thrust::device_vector<int>::iterator,thrust::device_vector<int>::iterator> tuple;
    tuple = thrust::minmax_element(dev_vec.begin(),dev_vec.end());
    *min = *(tuple.first);
    *max = *tuple.second;
}
inline void find_min(thrust::device_vector<int> &dev_vec, int *min){
    thrust::device_vector<int>::iterator iter;
 //   thrust::detail::execution_policy_base<DerivedPolicy> &exec;

    iter = thrust::min_element(dev_vec.begin(),dev_vec.end());
    *min = *iter;

}
inline void find_min1(thrust::device_vector<int> &dev_vec, int *min){
    thrust::device_vector<int>::iterator iter;

    iter = thrust::min_element(thrust::cuda::par,dev_vec.begin(),dev_vec.end());
    *min = *iter;

}

class min_pos
{ int val;
public:
__host__ __device__ min_pos(int min){val=min;}
__host__ __device__ int operator()(int & c)const {return (c==val)?1:0;}
};

struct is_even
  {
    __host__ __device__
    bool operator()(const int x)
    {
      return (x % 2) == 0;
    }
};

struct equal_to
  {
    __host__ __device__
    bool operator()(const int x, const int y)
    {
      return (x == y);
    }
};
/*
class minus_pred
{ bool val;
public:
__host__ __device__ minus_pred(class Pred p){val=p();}
__host__ __device__ int operator()(int & c1,int & c2)const {return val?(c1-c2):0;}
};*/

template<class In, class Out, class Pred>
Out copy_if(In first, In last, Out res,Pred p)
{
 while (first!=last)
 {if (p(*first)) *res++=*first;
 ++first;}
 return res;
}
template<class In, class Out, class Pred>
void minus_if(In first1, In last1, In first2, Out res,Pred p)
{
 while (first1!=last1)
 {if (p(*first1)) *res++=*first1-*first2;
 ++first1;
 ++first2;}
 return res;
}

int maintest(){
    int minele;//, maxele;
    char fout[50];
         sprintf(fout,"res/test_lib/res_%i_thrust.dat",LENGTH1);
    FILE *f_out;
    if((f_out=fopen(fout,"wt"))==NULL)return 0;

    int N=LENGTH1;
    std::vector<int> a,b(N),res(N);
    for (int i=0; i<N; i++)
    {
      a.push_back(rand());
      b[i]=a[i];
    }
    int j=N/LOOPS;
    thrust::host_vector<int> h_a(N),h_b(N);
    thrust::copy(a.begin(), a.end(), h_a.begin());
    thrust::device_vector<int> d_a = h_a;
    thrust::device_vector<int> d_b=d_a,d_res(N);
    thrust::device_vector<int>::iterator iter;
 //   thrust::copy(d_a.begin(),d_a.begin(),d_b.begin());

    hipEvent_t start, stop;
//=========================================================================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i=0; i < LOOPS; i++){
  // ------MINUS_IF-------------------------
    	thrust::copy(d_a.begin(),d_a.end(),d_b.begin());
  //  	thrust::transform( d_a.begin(),d_a.end(),d_b.begin(),d_res.begin(),thrust::minus<int>());
    	 thrust::transform_if(d_a.begin(),d_a.end(),d_b.begin(),d_a.begin(), d_res.begin(),thrust::minus<int>(),_1%2==0);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime1, totalTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    totalTime1 = elapsedTime1/(1000*LOOPS);
    hipError_t err = hipGetLastError();
// 	printf("errors after thrust %d\n",err);
    fprintf(f_out,"thrust minus_if time = %f\n", totalTime1);

    //========================================================================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i=0; i < LOOPS; i++){
// ------MINIMUM_IF-------------------------
    	  thrust::device_vector<int> d_a = h_a;
        thrust::copy(h_a.begin(), h_a.end(), d_a.begin());

      find_min(d_a,&minele);
     thrust::device_vector<int> d_b(N);
      thrust::fill(d_b.begin(),d_b.end(),0);
//      thrust::copy(d_a.begin(),d_a.begin(),d_b.begin());
      thrust::transform(d_a.begin(),d_a.end(),d_b.begin(),min_pos(minele));// позиции минимальных элементов
//      thrust::transform_if(d_a.begin(),d_a.end(),d_b.begin(),d_a.begin(), d_res.begin(),min_pos(minele),_1%2==0);
//     thrust::transform(thrust::cuda::par, d_a.begin(),d_a.end(),d_b.begin(),d_res.begin(),thrust::minus<int>());
//       thrust::stable_sort(d_a.begin(),d_a.end(),thrust::greater<int>());
 //   	thrust::copy_if(d_a.begin(),d_a.end(),d_b.begin(),is_even());
 //       thrust::transform_if(d_a.begin(),d_a.end(),d_b.begin(),d_a.begin(), d_res.begin(),thrust::plus<int>(),_1%2==0);
      }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
 //   float elapsedTime1, totalTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    totalTime1 = elapsedTime1/(1000*LOOPS);
//    hipError_t
    err = hipGetLastError();
 //	printf("errors after thrust %d\n",err);

//    printf("thrust min element = %d\n", minele);
    fprintf(f_out,"thrust min time = %f\n", totalTime1);        //

    //=========================================================================================
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    for (int i=0; i < LOOPS; i++){
// ------MATCH-------------------------
    	  thrust::device_vector<int> d_a = h_a;
        thrust::copy(h_a.begin(), h_a.end(), d_a.begin());

//      find_min(d_a,&minele);
     thrust::device_vector<int> d_b(N);
      thrust::fill(d_b.begin(),d_b.end(),0);
//      thrust::copy(d_a.begin(),d_a.begin(),d_b.begin());
 //     iter = thrust::find_if(a.begin(), a.end(),[i](const int x){ return x == i; } );
      thrust::transform(d_a.begin(),d_a.end(),d_b.begin(),min_pos(i));

//     thrust::transform(thrust::cuda::par, d_a.begin(),d_a.end(),d_b.begin(),d_res.begin(),thrust::minus<int>());
//       thrust::stable_sort(d_a.begin(),d_a.end(),thrust::greater<int>());
 //   	thrust::copy_if(d_a.begin(),d_a.end(),d_b.begin(),is_even());
 //       thrust::transform_if(d_a.begin(),d_a.end(),d_b.begin(),d_a.begin(), d_res.begin(),thrust::plus<int>(),_1%2==0);
      }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
 //   float elapsedTime1, totalTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    totalTime1 = elapsedTime1/(1000*LOOPS);
//    hipError_t
    err = hipGetLastError();
 //	printf("errors after thrust %d\n",err);

//    printf("thrust min element = %d\n", minele);
    fprintf(f_out,"thrust match_if time = %f\n", totalTime1);        //

//=========================================================================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i=0; i < LOOPS; i++){
// ------COPY_IF-------------------------
    	  thrust::device_vector<int> d_a = h_a;
//        thrust::copy(h_a.begin(), h_a.end(), d_a.begin());
     thrust::device_vector<int> d_b(N);
    	thrust::copy_if(d_a.begin(),d_a.end(),d_b.begin(),is_even());
 //       thrust::transform_if(d_a.begin(),d_a.end(),d_b.begin(),d_a.begin(), d_res.begin(),thrust::plus<int>(),_1%2==0);
      }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
//    float elapsedTime1, totalTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    totalTime1 = elapsedTime1/(1000*LOOPS);
    //hipError_t
    err = hipGetLastError();
 //	printf("errors after thrust %d\n",err);

//    printf("thrust min element = %d\n", minele);
    fprintf(f_out,"thrust copy_if time = %f\n", totalTime1);        //

//    thrust::copy(d_b.begin(),d_b.end(),b.begin());
     clock_t t;

     std::vector<int>::iterator resultmax, resultmin;

 //=============================================================
     t = clock();
/*    std::sort(a.begin(), a.end());
    t = clock() - t;
    printf("STL sort time = %f\n", ((float)t)/(CLOCKS_PER_SEC));
*/

    for (int i = 0; i<LOOPS; i++){
 //     resultmax = std::max_element(a.begin(), a.end());
     resultmin = std::min_element(a.begin(), a.end());

 //   	std::transform(a.begin(),a.end(),b.begin(),res.begin(),std::minus<int>());
  //  	resultmax=std::lower_bound(a.begin(), a.end(),b[N-j*i]);
    //	std::copy(a.begin(),a.end(),b.begin());
 //   	minus_if(a.begin(),a.end(),b.begin(),res.begin(),is_even());
 //   	std::transform_if(a.begin(),a.end(),b.begin(),a.begin(), res.begin(),std::minus<int>(),_1%2==0);
     }
    t = clock() - t;
//    resultmax=resultmin;
 //   printf("STL sort= %d, max element = %d\n", b[j], *resultmax);
    fprintf(f_out,"STL min time = %f\n", ((float)t)/(CLOCKS_PER_SEC*LOOPS));
//====================================================================================
     t = clock();
     //copy_if(a.begin(),a.end(),b.begin(),is_even());
     for (int i = 0; i<LOOPS; i++) std::copy_if(a.begin(),a.end(),b.begin(),is_even());

     t = clock() - t;
     //    resultmax=resultmin;
      //   printf("STL sort= %d, max element = %d\n", b[j], *resultmax);
         fprintf(f_out,"STL copy_if time = %f\n", ((float)t)/(CLOCKS_PER_SEC*LOOPS));

          t = clock();
    	 std::transform(a.begin(),a.end(),b.begin(),res.begin(),std::minus<int>());
     t = clock() - t;
     fprintf(f_out,"STL minus time = %f\n", ((float)t)/(CLOCKS_PER_SEC));//*LOOPS));*/
 //    puts("d_b");
 //   thrust::copy(d_res.begin(), d_res.end(), std::ostream_iterator<int>(std::cout, "\n"));
//    thrust::copy(d_res.begin(),d_res.end(),res.begin());
//======================================================================================
     t = clock();
      //copy_if(a.begin(),a.end(),b.begin(),is_even());
      for (int i = 0; i<LOOPS; i++)
     	 std::find(a.begin(),a.end(),10);
     	 //transform(a.begin(),a.end(),b.begin(),res.begin(),std::minus<int>());
      t = clock() - t;
      fprintf(f_out,"STL match time = %f\n", ((float)t)/(CLOCKS_PER_SEC));//*LOOPS));*/
// for(int i=0;i<N;i++) printf("a,b[%i]=%i res=%i\n",i,b[i],res[i]);

//    thrust::copy(d_a.begin(), d_a.end(), std::ostream_iterator<int>(std::cout, "\n"));
  return 0;
}
