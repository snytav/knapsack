#include "hip/hip_runtime.h"
#include "knapsack.h"
#include <ctime>
#include <stdio.h>
#include <sys/time.h>
#include "basic.h"
//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"

#define WMAX 100
#define CMAX 1000
#define N_K  16

extern  const int NN1;


// для заданного числа n генерирует масив весов от 1 до WMAX и массив стоимости от 1 до CMAX
void problem_generate(int n, int *w, int *c)
{   srand( time(0));
	int i;
	for (i=0;i<n; i++)
	{
		w[i]=rand()%WMAX+1;
		c[i]=rand()%CMAX+1;
	}
}

// vjue
void branch_cut(int n,int *w, int W, Slice *T,Slice *B)
{   int i,k_tmp=0;
int w_t=W;
int w_b=W;

for (i=0;i<n;i++)
{
	if (w[i]<=w_b)
	{   B->set(i,1);
		w_b-=w[i];

	}
}

for (i=1;i<n+1;i++)
	{
	if(w[n-i]<=w_t)
			{
				w_t-=w[n-i];
				T->set(n-i,1);
			}
}
}

// подряд идущие, без пропусков
void branch_cut(int n, int *w, int W, int &k_t, int &k_b)
{   int i,k_tmp=0;
	int w_t=W;
	int w_b=W;
	k_t=n;
	k_b=0;
	for (i=0;i<n;i++)
	{
		if (w[i]<=w_b)
		{//   printf("%i",1);
			w_b-=w[i];
			k_b++;

		}else
		{
		//	printf("%i",0);
			if (k_tmp==0) k_tmp=k_b;
		}
	}
   // puts("");
	k_b=(k_tmp==0)?k_b:k_tmp+1;
	i=1; k_tmp=0;
//	puts("Mirrored");
	for (i=1;i<n+1;i++)
		{
		if(w[n-i]<=w_t)
				{ //printf("%i",1);
					w_t-=w[n-i];
					k_t--;
				}
		else
				{
					//printf("%i",0);
					if (k_tmp==0) k_tmp=k_t;
				}
	}
		k_t=(k_tmp==0)?k_t:k_tmp;
	//	printf("   %i:%i %i ",k_b,k_t ,w_t);
	//	 puts("");

}

__global__ void init_stable(LongPointer *d_T,int NN1)
{	unsigned long long int	*d_t,
	init_x[]={0xAAAAAAAAAAAAAAAA,0xCCCCCCCCCCCCCCCC,
		   0xF0F0F0F0F0F0F0F0,0xFF00FF00FF00FF00,
		   0xFFFF0000FFFF0000,0xFFFFFFFF00000000,
		   0xFFFFFFFFFFFFFFFF,0};
    unsigned long long int i,j=1,k;
    i=blockIdx.x;
    		{   d_t=d_T[M-i-1];
    			if (i<6)
    			{
    				for (k=0;k<NN1;k++) d_t[k]=init_x[i];

    			}
    			else
    			if (i<70)//там сдвиг уже не сработает, но длины таблицы <2:70
    			{
    				j=j<<(i-6);
    				for (k=0;k<NN1;k++)
    				   d_t[k]=((k&j)==j)?init_x[6]:init_x[7];
    			}
    		}
}
void initial_search_table(Table *T)
{   printf(" NN=%i sz=%i \n",NN1,M);

	//cudaPrintfInit ();
hipEvent_t start, stop;
float elapsedTime1;
	     hipEventCreate(&start);
	     hipEventCreate(&stop);
	     hipEventRecord(start, 0);
	init_stable<<<M,1>>>(T->get_device_pointer(),NN1);
	 hipEventRecord(stop, 0);
	 hipEventSynchronize(stop);
     hipEventElapsedTime(&elapsedTime1, start, stop);// in 0.001 sec

     printf("associative time init %f (%i)\n", elapsedTime1, NN1);
LongPointer *d_T=T->get_device_pointer();
	unsigned long long int i,k,j=1;
	unsigned long long int	*d_t;
//	hipMemcpy(hostT,d_T,sizeof(unsigned int int)*NN1*M,hipMemcpyDeviceToHost);

	unsigned  long long int tmp,hostT[M][NN1],

	 init_x[]={0xAAAAAAAAAAAAAAAA,0xCCCCCCCCCCCCCCCC,
			   0xF0F0F0F0F0F0F0F0,0xFF00FF00FF00FF00,
			   0xFFFF0000FFFF0000,0xFFFFFFFF00000000,
			   0xFFFFFFFFFFFFFFFF,0};
	double tt;
    struct timeval tv1,tv2;
    gettimeofday(&tv1,NULL);

	for(i=0;i<M;i++)
	{
//		d_t=(T->col(M-i))->get_device_pointer();
		if (i<6)
		{
			//hipMemset(d_t,(int)init_x[i],NN1*2*sizeof(unsigned int));
//			hipMemset(d_t,init_x[i],NN1*sizeof(unsigned long long int));
//			hipMemcpy(hostT,d_t,sizeof(unsigned int int)*NN1*M,hipMemcpyDeviceToHost);
			for (k=0;k<NN1;k++) hostT[M-1-i][k]=init_x[i];

		}
		else
		{  // j=1;
/*			if(i%2==0)
				hipMemset(d_t,init_x[6],NN1*sizeof(unsigned long long int));
		    else
		    	hipMemset(d_t,init_x[5],NN1*sizeof(unsigned long long int));
//			printf("%d: %lp %p\n", i, init_x[6], d_t);
 *
 */
//			printf("\n ==== %d ",i);
			for (k=0;k<NN1;k++)
			{
				hostT[M-1-i][k]=((k&j)==j)?	init_x[6]:init_x[7];
//				printf("%p ",hostT[i][k]);
			}
			j=j<<1;
		}

//		printf("%d: %p %p\n", i, init_x[i%8],hostT[i][0]);
	}
	 gettimeofday(&tv2,NULL);
	 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
	 printf("time of work seq %f sec \n", tt);
	//cudaPrintfDisplay (stdout, true);
	//cudaPrintfEnd ();

	hipError_t err = hipGetLastError();
	printf("after init search table %d , %s \n",err,hipGetErrorString(err));
}

void knapsack_exp()
{
	int w[M],c[M],W,k_b,k_t,i,j,W_t=0,W_b=0;
	problem_generate(M, w, c);
	W=rand()%(WMAX*M/2)+1;
	Slice *ST,*SB,*X,*w_max;

	ST=new Slice;
	ST->Init(M);
	SB=new Slice;
	SB->Init(M);
	X=new Slice;
	X->Init(LENGTH1);
	w_max=new Slice;
	w_max->Init(N_K);
	w_max->FromDigit(W);

	branch_cut(M, w,W,k_t,k_b);
	branch_cut(M, w,W, ST,SB);
	ST->print("st",0);
	SB->print("sb",0);

	unsigned long long int tmp,dig;
// print results branch_cut
	for (i=0; i<M;i++){
//	  tmp=c[i]<<(64-32);
//	  dig=__brevll(tmp); //переворот в правильную сторону для суммирования
		printf("<%i;%i> \n ",w[i],c[i]);
	}
	printf("\n W=%i \n",W);

/*	for (i=0; i<M;i++)
		if (i<k_t) printf("0");
		else {printf("1");
		W_t+=w[i];
		}
	printf("\n W=%i \n",W_t);

	for (i=0; i<M;i++)
		if (i<k_b) {
			printf("1");
			W_b+=w[i];
		}
		else printf("0");

	printf("\n W=%i \n",W_b);
*/
	Table *T;
	unsigned long long int hostT[M][NN1];
	T =new Table;
	T->Init(LENGTH1,M);
	puts("Init T");

	// инициализация таблицы перебора
	// появление в ней sb - условие окончания перебора
	initial_search_table(T);
//	T->writeToFile("log/init");


	/*
	Var T, WT, CT: Table;
	    Y,Z,Z1,Z2,Z3: Slice;
	    SN,ST,SB: Word(n)
	    v,u: Word(h);
	    */

/////////////////////////////////////////////////
	double tt;
    struct timeval tv1,tv2;
    gettimeofday(&tv1,NULL);
    char str[15];
	Table *WT, *CT;

	WT =new Table;
	WT->Init(LENGTH1,N_K);

	CT =new Table;
	CT->Init(LENGTH1,N_K);

	Slice  *Y,*Z,*Z1,*Z2,*Z3, *SN,*u,*v;

	Y = new Slice;
	Y->Init(LENGTH1);

	Z = new Slice;
	Z->Init(LENGTH1);

	Z1 = new Slice;
	Z1->Init(LENGTH1);

	Z2 = new Slice;
	Z2->Init(LENGTH1);

	Z3 = new Slice;
	Z3->Init(LENGTH1);

	SN = new Slice;
	SN->Init(M);

	v = new Slice;
	v->Init(N_K);

	u = new Slice;
	u->Init(N_K);

    T->GetRow(SN,LENGTH1);
    Z1->CLR();
    Z2->SET();Z2->set(1,0);
    Z3->SET();
    ADDC1(Z3,ST,T);
/*
    ST->print("log/ST",1);
    SN->print("log/SN",1);
    SB->print("log/SB",1);
*/
    printf("Начинаем перебор \n");
    j=0;
/*
 *
   sprintf(str, "log/TST%05d",i);
    T->writeToFile(str);
  */

    while (Z1->ZERO())
    {
/*
    	CLEAR(WT);
    	CLEAR(CT);
       puts("Clear WT and CT");
       for(i=1;i<M;i++)
    	{
          v->FromDigit(w[i+1]);
          u->FromDigit(c[i+1]);
          T->GetCol(Y,i);
          ADDC1(Y,v,WT);
          ADDC1(Y,u,CT);
    	}
       puts("summa done");
       LESS(WT,Z3,w_max,Z);
       puts("Less");
  //     MAX(CT,Z,X);
       puts("MAX");
       if (X->get(1)==0)// смена максимума
       {
          i=X->FND();
          T->GetRow(ST,i);
          T->SetRow(ST,1);
  //        row(1,CT):=row(i,CT);//необязательно, посчитается на следующем шаге
       }
       puts("max changes");
       MATCH(T,Z2,SB,Z1);
       puts("MATCH");
*/
       ADDC1(Z2,SN,T); // к первой строке не добавляется, там максимум
       puts("next step");
       j++;
       printf("%d ",j);
/*       sprintf(str, "log/TST%05d",i);
          T->writeToFile(str);
       T->GetRow(ST,LENGTH1);
       ST->print("log/tt",1);
       */
    }

     hipError_t err = hipGetLastError();
     printf("after init search table %d , %s \n",err,hipGetErrorString(err));

     gettimeofday(&tv2,NULL);
	 tt=0.000001*(tv2.tv_usec-tv1.tv_usec)+(tv2.tv_sec-tv1.tv_sec);
	 printf("time of all work seq %f sec \n", tt);

	sprintf(str, "res/NP/test%d.txt",j);
	T->writeToFile("res/NP/test.txt");
}
