#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "param.h"
#include "slice.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/copy.h>

unsigned long long int h_v[N1];// for print mast be copied from d_v


int Slice::Init(unsigned int k)
{

	hipError_t err1, err = hipGetLastError();

//	printf("before all error %d , %s \n",err,hipGetErrorString(err));
//	if (err!=0)exit(0);
	length=k;
	NN=(((k % SIZE_OF_LONG_INT) ==0)?(k/SIZE_OF_LONG_INT):(k/SIZE_OF_LONG_INT+1));
//    printf("slice.init %u ", NN);
#ifdef ssss
	int *d_i;
	printf("Slice init %d %s\n",err,hipGetErrorString(err));
	err = hipMalloc(&d_i,sizeof(int));

	d_first_non_zero =d_i;
	printf("Slice alloc error %d %s \n",err,hipGetErrorString(err));
#endif
//	err1 = hipGetLastError();
//	printf("before alloc error %d , %s \n",err1,hipGetErrorString(err1));
    err = hipMalloc(&d_v,NN*sizeof(unsigned long long int));
 //   printf("Slice alloc error %d , %s ,%p \n",err,hipGetErrorString(err),d_v);
#ifdef ssss
    printf("Slice alloc error %d %s \n",err,hipGetErrorString(err));
#endif
    hipMemset(d_v,0,NN*sizeof(unsigned long long int));

//	exit(0);

    return err;
}
//void Print();
//void set_from_host_array (unsigned long long int *f_h_v);
//void set_from_device_array (unsigned long long int *f_h_v);

__global__ void set_long_values(unsigned long long int *d_v,unsigned long long int num)
{
	//char s[100];
//	printf("set \n");//long %llu \n",num);
//	return;
//	long_to_binary(num,s);
	d_v[blockIdx.x] = num;
//	printf("set long %s \n",s);
}

//заполнить единичками,
void Slice::SET()
{
	 unsigned long long int zero = 0;
	 zero = ~zero;
#ifdef ss
	 char s[100];
	 long_to_binary(zero,s);
	 printf("SET %s \n",s);


	 hipError_t err = hipGetLastError();
	 printf("error before set_lon_values %d \n",err);
	 hipError_t err_c = hipMemcpy(h_v,d_v,sizeof(unsigned long long int),hipMemcpyDeviceToHost);
	 long_to_binary(h_v[0],s);
	 printf("h_v[0] %llu err %d %s\n",h_v[0],err_c,s);
	 print("q1",1);
#endif
     set_long_values<<<NN,1>>>(d_v,zero);
 //    printf("SET: %i->%llu \n",NN,zero);
#ifdef qq
     err_c = hipMemcpy(h_v,d_v,sizeof(unsigned long long int),hipMemcpyDeviceToHost);
	 long_to_binary(h_v[0],s);
	 printf("h_v[0] %llu err %d %s\n",h_v[0],err_c,s);


     print("q2",1);

	 err = hipGetLastError();
	 printf("error after set_lon_values %d \n",err);
#endif
}

//заполнить нулями,
void Slice::CLR()
{
	 unsigned long long int zero = 0;
    set_long_values<<<NN,1>>>(d_v,zero);
}
__global__ void set_mask_values(unsigned long long int *d_v, int num)
{ unsigned long long int zero=1;
  int num_el=num>>6; // номер элемента, содержащий переход от 0 к 1;
  int el=num % SIZE_OF_LONG_INT;
//  printf("%i in %i \n", num,num_el);
  if (blockIdx.x==num_el)
   {
	  zero=(el==0)?0:(zero<<(el-1))-1;
	  zero=~zero;
   }
  else
  {
      zero=0;
      if (blockIdx.x>num_el)
      {
    	  zero=~zero;
      }
  }
   d_v[blockIdx.x]=zero;
}
void Slice::MASK(int i)
{
	set_mask_values<<<NN,1>>>(d_v,i);
}

__global__ void get_kernel(unsigned long long int *d_v,unsigned char *d_num,bool get,int n)
{
		int num = get_position_bit(d_v,n);
		*d_num = (unsigned char)num;
}

__global__ void put(unsigned long long int *d_v,unsigned char d_num,int n)
{
	assign_bit(d_v,n,d_num,SET);
}

//доступ к i-ой компоненте слайса, как на чтение, так и на запись,
unsigned char Slice::get(int i)
{
//	hipError_t   err = hipGetLastError();
//    printf("begin get %d, %d , %s \n",i,err,hipGetErrorString(err));
    unsigned char n;
   static int flag=1;
   static unsigned char *d_n;
    if (flag==1)
    {
    hipMalloc(&d_n,sizeof(unsigned char));
    flag=0;
    }
	get_kernel<<<1,1>>>(d_v,d_n,1,i);

	hipMemcpy(&n,d_n,sizeof(unsigned char),hipMemcpyDeviceToHost);
//	err = hipGetLastError();
//	printf("end get %d, %d , %s \n",i,err,hipGetErrorString(err));
//	if(err!=0)exit(0);
//   printf("get_ %d : %us \n", i,n);
	return (unsigned char)n;
}

void Slice::set(int i,unsigned char n)
{

     put<<<1,1>>>(d_v,n,i);
}

// - выдает номер старшей единичке в слайсе Y
int Slice::FND()
{
	int h_first_non_zero;
	static 	int *d_first_non_zero;
    static int flag_malloc=1;


   if (flag_malloc==1)
  {
	hipMalloc(&d_first_non_zero,sizeof(int));
    flag_malloc=0;
   }
//    print_device_bit_row("FND",d_v,NN*SIZE_OF_LONG_INT,0,NN);

	first(d_v,NN,d_first_non_zero,NN);

	hipMemcpy(&h_first_non_zero,d_first_non_zero,sizeof(int),hipMemcpyDeviceToHost);
    if (h_first_non_zero>length)
    	h_first_non_zero=0;
//    printf("FND %i",h_first_non_zero);
	return h_first_non_zero;
}
int Slice::NUMB()
{

	int h_first_non_zero;
	static 	int *d_first_non_zero;
    static int flag_malloc=1;

    if (flag_malloc==1)
    {
	hipMalloc(&d_first_non_zero,sizeof(int));
	 flag_malloc=0;
    }
//    print_device_bit_row("NUMB",d_v,NN*SIZE_OF_LONG_INT,0,NN);

	number(d_v,NN,d_first_non_zero,NN);

	hipMemcpy(&h_first_non_zero,d_first_non_zero,sizeof(int),hipMemcpyDeviceToHost);
//    printf("NUMB=%i \n",h_first_non_zero);
	return h_first_non_zero;
}
__global__ void numb_thrust(int *dev_vec, LongPointer d_v)
{ int k;
  unsigned long long int zero=1;
  int tid=threadIdx.x+blockIdx.x*blockDim.x;
  if (tid<N1)
  {
	  dev_vec[tid]=__popcll(d_v[tid]);
//      printf("%i  ",dev_vec[tid]);
  }
//  else dev_vec[tid]=0;
  if (tid==(N1-1)) // in the last element need to zero the tail
  {
  	/*zero=(1<<(num % SIZE_OF_LONG_INT)-1)-1;
	  zero=~zero;*/
  	k=(LENGTH1%SIZE_OF_LONG_INT);
//    	printf("k=%i\n",k);
  	zero=(zero<<k)-1;
  	zero&=d_v[tid];
  	dev_vec[tid]=__popcll(zero);
  }
}
int Slice::NUMB1()
{
	 thrust::device_vector<int> d_a(N1);
	 int * dv_ptr = thrust::raw_pointer_cast(d_a.data());
	 numb_thrust<<<blocks1,threads1>>>(dv_ptr,d_v);
	int h_first_non_zero;
     h_first_non_zero=thrust::reduce(d_a.begin(),d_a.end());
 //   printf("NUMB1=%i \n",h_first_non_zero);
	return h_first_non_zero;
}
__global__ void fnd_thrust(int *dev_vec, LongPointer d_v)
{ int tid=threadIdx.x+blockIdx.x*blockDim.x;
  if (tid<N1)
  {   int fnd_tid= (__ffsll(d_v[tid])!=0)?(__ffsll(d_v[tid])+tid*SIZE_OF_LONG_INT):(SIZE_OF_LONG_INT*N1+1);
	  dev_vec[tid]=fnd_tid;
//      printf("%i  ",dev_vec[tid]);
  }
}
int Slice::FND1()
{
	 thrust::device_vector<int> d_a(N1);
	 int * dv_ptr = thrust::raw_pointer_cast(d_a.data());
	 fnd_thrust<<<blocks1,threads1>>>(dv_ptr,d_v);
	int h_first_non_zero;
     h_first_non_zero=*(thrust::min_element(d_a.begin(),d_a.end()));
 //   printf("NUMB1=%i \n",h_first_non_zero);
     if (h_first_non_zero>length)
         	h_first_non_zero=0;
	return h_first_non_zero;
}
// - то же самое, но эту единичку заменяет на ноль
int Slice::STEP()
{
//	print_device_bit_row("STEP",d_v,NN*SIZE_OF_LONG_INT,0,NN);
	int f = FND();
	if (f>0)set(f,0);
//	print_device_bit_row("S_res",d_v,NN*SIZE_OF_LONG_INT,0,NN);
//	printf("vertex %i ",f);
	return f;
}

// - конвертирует слайс в строку (используется крайне редко).
/*void Slice::CONVERT()
{
//	word_flag = 1;
}*/

//Побитовые X and Y, not X, X or Y, X xor Y
__global__ void and_long_values(unsigned long long int *d_v,unsigned long long int *d_v1)
{
	d_v[blockIdx.x] &= d_v1[blockIdx.x];
}

__global__ void or_long_values(unsigned long long int *d_v,unsigned long long int *d_v1)
{
#ifdef ssss
	unsigned long long old,old1;
	char s_old[100],s_old1[100],res[100];
	old  = d_v[blockIdx.x];
	old1 = d_v1[blockIdx.x];

	long_to_binary(old,s_old);
	long_to_binary(old1,s_old1);
#endif
	d_v[blockIdx.x] |= d_v1[blockIdx.x];
#ifdef ssss
	long_to_binary(d_v[blockIdx.x],res);
	printf("blockIdx.x %u old %llu %s %llu %s %llu %s\n",blockIdx.x,old,s_old,old1,s_old1,d_v[blockIdx.x],res);
#endif

}

__global__ void xor_long_values(unsigned long long int *d_v,unsigned long long int *d_v1)
{
	d_v[blockIdx.x] ^= d_v1[blockIdx.x];
}

__global__ void not_long_values(unsigned long long int *d_v)
{
	d_v[blockIdx.x]=~d_v[blockIdx.x];
}

Slice Slice::operator & (const Slice & b)
{
	and_long_values<<<NN,1>>>(d_v,b.d_v);

	return *this;
}
void Slice::AND(const Slice *b)
{
	and_long_values<<<NN,1>>>(d_v,b->d_v);
}
void Slice::OR(const Slice *b)
{
	or_long_values<<<NN,1>>>(d_v,b->d_v);
}
Slice Slice::operator | (const Slice & b)
{

	or_long_values<<<NN,1>>>(d_v,b.d_v);

	return *this;
}
void Slice::XOR(const Slice * b)
{
	xor_long_values<<<NN,1>>>(d_v,b->d_v);
}
Slice Slice::operator ^ (const Slice & b)
{

	xor_long_values<<<NN,1>>>(d_v,b.d_v);

	return *this;
}

Slice Slice::operator ~()
{
	not_long_values<<<NN,1>>>(d_v);
	return *this;
}
// - true, если X ненулевой.
void Slice::NOT()
{
	not_long_values<<<NN,1>>>(d_v);
}
bool Slice::SOME()
{
	int f = FND();
//  printf("SOME %d \n", f);
	return (f > 0);
}
bool Slice::SOME1()
{
	 int h_if_zero;
	static  int *d_if_zero;
    static int flag_malloc=1;

    if (flag_malloc==1)
    {
	hipMalloc(&d_if_zero,sizeof( int));
	  flag_malloc=0;
    }
//    print_device_bit_row("FND",d_v,NN*SIZE_OF_LONG_INT,0,NN);

	some(d_v,NN,d_if_zero,NN);

	hipMemcpy(&h_if_zero,d_if_zero,sizeof( int),hipMemcpyDeviceToHost);

   printf("SOME1 %d \n", h_if_zero);
	return (h_if_zero!=0);
}
bool Slice::ZERO()
{
	int f = FND();
//   printf("ZERO %d \n", f);
	return (f == 0);
}
unsigned long long int char_to_long(char *s)
{
	unsigned long long int u = 0,u1,t;
	double d;
//	char str[LENGTH1];
//    puts("char_to_long");
	int len = (LENGTH1 < SIZE_OF_LONG_INT ? LENGTH1: SIZE_OF_LONG_INT);

	for (int i = 0;i < len;i++)
	{
#ifdef ssss
		 printf("i %d\n",i);
#endif
		 d = pow(2.0,(double)i);
		 t = (unsigned long long int)ceil(d);
		 u1 = (s[i]-'0')*t;
         u += u1;
#ifdef ssss
         long_to_binary(u,str);
         printf("i %d d %40.25e t %25llu s[i] %c u1 %25llu u %25llu %s\n",i,d,t,s[i],u1,u,str);
#endif
	}
	return u;
}

Slice Slice::operator= (char *s)
{
	char num[SIZE_OF_LONG_INT+1];
#ifdef sssss
	puts(s);
#endif
	for(int i = 0;i < strlen(s);i += SIZE_OF_LONG_INT)
	{
	    strncpy(num,s+i,SIZE_OF_LONG_INT);
	    num[SIZE_OF_LONG_INT] = 0;
	    h_v[i/SIZE_OF_LONG_INT] = char_to_long(num);
#ifdef sssss
	    printf("i %d num %s %llu\n",i,num,h_v[i/SIZE_OF_LONG_INT]);
#endif
	}
	hipMemcpy(d_v,h_v,NN*sizeof(unsigned long long int),hipMemcpyHostToDevice);
//	puts("=s");
	return *this;
}



void Slice::assign(const Slice * s)
{
//	printf("before:= %llu %llu \n", d_v,s->d_v);
//	exit(0);
	if (NN!=s->NN)printf("sizes of slices are not equal %u %u %u %u \n",NN,s->NN,length,s->length);

    length=s->length;
    NN=s->NN;
	set_kernel<<<blocks1,threads1>>>(d_v,s->d_v);
//	printf("after:= %p %p \n", d_v,s->d_v);
}

Slice Slice::operator= (Slice *s)
{
//	printf("before:= %llu %llu \n", d_v,s->d_v);
//	exit(0);
	if (NN!=s->NN)printf("sizes of slices are not equal %u %u %u %u \n",NN,s->NN,length,s->length);

    length=s->length;
    NN=s->NN;
	set_kernel<<<NN,1>>>(d_v,s->d_v);
//	printf("after:= %p %p \n", d_v,s->d_v);
	return *this;
}

void Slice::convert_to_string(char *str)
{
    char s[SIZE_OF_LONG_INT+1];
//    printf("convert %d %d \n", NN, length);

    hipMemcpy(h_v,d_v,NN*sizeof(unsigned long long),hipMemcpyDeviceToHost);
//	printf("convert: %d %llu \n",0,h_v[0]);
   strcpy(str,"");
   for (int i = 0;i < NN;i++)
   {
       long_to_binary(h_v[i],s,length);
//#ifdef ssss
 //      puts(s);
//#endif
//       sprintf(s,"%s",s);
       strcat(str,s);
   }
   str[length] = 0;
//#ifdef ssss
//   puts(str);
//#endif
}

void __global__ digit_kernel(unsigned long long *w, unsigned long long *dig)
{
	dig[0]=__brevll(w[0]);
}

unsigned long long int Slice::ToDigit()
{ unsigned long long high,low,*d_dig1,d_dig,res=0;
	if (NN==1)
	{  hipMalloc(&d_dig1,sizeof(unsigned long long));
		digit_kernel<<<1,1>>>(d_v,d_dig1);
		hipMemcpy(&res,d_dig1,sizeof(unsigned long long),hipMemcpyDeviceToHost);
		res>>=(64-H1);
	}
/*	{
		hipMemcpy(&d_dig,d_v,sizeof(unsigned long long),hipMemcpyDeviceToHost);
		low=1;
		high=1<<(H1-1);
	for (int i=0; i<H1;i++)
	{   if (d_dig&low)
		res+=high;
		low<<=1;
		high>>=1;
	}
	}*/
	return res;
}
unsigned long long int Slice::FromDigit(unsigned long long dig)
{	 unsigned long long high,low,*d_dig1,d_dig,res=0;
		if (NN==1)
		{ dig<<=(64-H1);
			hipMalloc(&d_dig1,sizeof(unsigned long long));
			hipMemcpy(d_dig1,&dig,sizeof(unsigned long long),hipMemcpyHostToDevice);
			digit_kernel<<<1,1>>>(d_dig1,d_v);
		}
}




void __device__ trim_(unsigned long long int *d_v, unsigned long long int *d_v_in,int i, int h)
// предполагается и для матрицы, и для слайса
//threadth - для номера столбца
//block - для номера элемента по меньшему слайсу
/*{
	int i=10;
	int h=15;
	unsigned long long int d_v, d_v_in= 4842603519;
	int num_el=i>>6;//номер элемента в большем слайсе
   int num_bit_first= i % SIZE_OF_LONG_INT; // номер бита в элементе, который станет первым в маленьком слайсе
   int num_bit_last = (i+h) % SIZE_OF_LONG_INT;
   unsigned long long int teal, head =d_v_in<<num_bit_first;

  	   teal = d_v_in>>(SIZE_OF_LONG_INT-num_bit_first);
	   d_v=head | teal;

   printf("head=%ull teal=%ull \n",head,teal);
}*/

{  int num_el=(i-1)>>6;//номер первого элемента в большем слайсе
   int num_el1=(h-1)>>6; // номер последнего элемента в маленьком
   int num_el2=(i+h-1)>>6;// номер последнего элемента в большом слайсе
   int num_bit_first= i % SIZE_OF_LONG_INT -1; // номер бита в элементе, который станет первым в маленьком слайсе
   int num_bit_last = h % SIZE_OF_LONG_INT;
   char prb[65];
//   printf("num_els %i and %i (%i) bits from %i to %i \n",num_el,num_el1,num_el2, num_bit_first, num_bit_last);
   unsigned long long int teal, head =d_v_in[blockIdx.x+num_el]>>(num_bit_first);
//   long_to_binary(head,prb,64);
//   	   printf("\n head:");printf(prb);
   if (blockIdx.x +num_el< num_el2)
   {
	   teal = d_v_in[blockIdx.x+1+num_el]<<(SIZE_OF_LONG_INT-num_bit_first);
//	   long_to_binary(teal,prb,64);
//	   printf("\n teal:");printf(prb);
	   d_v[blockIdx.x]=head | teal;

//	  	   printf("\n elem:");printf(prb);
   }
   if (blockIdx.x==num_el1) // обрезать последние биты от num_bit_last
   {
	   teal=1;
	   teal=(num_bit_last==0)? ~0:((teal<<num_bit_last)-1);
//	   long_to_binary(teal,prb,64);
//	   printf("\n teal_up (%i):",num_bit_last);printf(prb);
	   d_v[blockIdx.x]&=teal;
   }
//   printf("ind=%i d_v_in=%llu head=%llu teal=%llu d_v=%llu\n",blockIdx.x+num_el, d_v_in[blockIdx.x+num_el],head,teal,d_v[blockIdx.x]);
}

void __global__ trim_slice_kernal(unsigned long long int *d_v, unsigned long long int *d_v_in,int i, int h)
{ trim_(d_v,d_v_in,i,h);
//   printf("in trim_kernal");
}

void Slice::trim(int i, int h, Slice *s)
{
//	puts("in Slice.trim \n");
//	unsigned long long int *d_v
	unsigned long long int *d_v_in;
	d_v_in= s->get_device_pointer();
	int hh,n=this->NN;
	if (s->length>i+h)
	{hh=h;}
	else
	{hh=s->length-i+1;}

	trim_slice_kernal<<<n,1>>>(d_v,d_v_in,i,hh);
}

void __device__ shiftup(unsigned long long int *d_v, unsigned long long int *d_v_in,int i)
{
	int num_el=i>>6;//номер элемента в большем слайсе
//int num_el1=(i+h)>>6;
int num_bit_first= i % SIZE_OF_LONG_INT ; // номер бита в элементе, который станет первым в маленьком слайсе
//int num_bit_last = h % SIZE_OF_LONG_INT;
//printf("num_els %i (%i) bits from %i  \n",blockIdx.x +num_el,gridDim.x,num_bit_first);
unsigned long long int teal, head =d_v_in[blockIdx.x+num_el]>>(num_bit_first);
if (blockIdx.x +num_el<gridDim.x)//?????????
{
	   teal = (blockIdx.x+1+num_el<gridDim.x)?d_v_in[blockIdx.x+1+num_el]<<(SIZE_OF_LONG_INT-num_bit_first):0;
	   d_v[blockIdx.x]=head | teal;
}
else // обрезать последние биты от num_bit_last
{
//	   teal=(1<<num_bit_last) -1;
	   d_v[blockIdx.x]=0;//head & teal;
}
}

void __device__ shiftdown(unsigned long long int *d_v, unsigned long long int *d_v_in,int i)
{
	int num_el=i>>6;//номер элемента в большем слайсе
//int num_el1=(i+h)>>6;
int num_bit_first= i % SIZE_OF_LONG_INT ; // номер бита в элементе, который станет первым в маленьком слайсе
//int num_bit_last = h % SIZE_OF_LONG_INT;
//printf("num_els %i  bits from %i  \n",blockIdx.x -num_el,num_bit_first);
unsigned long long int teal,head;
if (blockIdx.x >num_el)//?????????
{      head =d_v_in[blockIdx.x-num_el]<<(num_bit_first);
	   teal =d_v_in[blockIdx.x-1-num_el]>>(SIZE_OF_LONG_INT-num_bit_first);
	   d_v[blockIdx.x]=head | teal;
	   printf("num_els %i (%i)%llu head=%llu (%i)%llu teal=%llu  \n",blockIdx.x,blockIdx.x-num_el,d_v_in[blockIdx.x-num_el],head,blockIdx.x-1-num_el,d_v_in[blockIdx.x-1-num_el],teal);
}
else // обрезать последние биты от num_bit_last
{
//	   teal=(1<<num_bit_last) -1;
	   d_v[blockIdx.x]=(blockIdx.x==num_el)? (d_v_in[0]<<(num_bit_first)):0;//head & teal;
	   printf("num_els %i  0 %i \n",blockIdx.x,num_el );
}

}

void __global__ shiftdown_kernel(unsigned long long int *d_v, unsigned long long int *d_v_in,int i)
{
	shiftdown(d_v,d_v_in,i);
}

void __global__ shiftup_kernel(unsigned long long int *d_v, unsigned long long int *d_v_in,int i)
{
	shiftup(d_v,d_v_in,i);
}
void Slice::shift_up(int i,Slice *s)
{
	unsigned long long int *d_v_in;
	d_v_in= s->get_device_pointer();
	int n=NN;
	shiftup_kernel<<<n,1>>>(d_v,d_v_in,i);
}
void Slice::shift_down(int i,Slice *s)
{
	unsigned long long int *d_v_in;
	d_v_in= s->get_device_pointer();
	int n=NN;
	shiftdown_kernel<<<n,1>>>(d_v,d_v_in,i);
}
