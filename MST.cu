
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "MST.h"
#include "star_kernel.h"
#include "iostar.h"
#include "basic.h"

void MSTPaths(Table *left, Table *right, Table *weight, Table *code, Slice *S, Slice *T, Table *M1)
{ unsigned int i,k,l;

  Slice *S1;
  Slice *N12;
  Slice *N2;
  Slice *X,*Y,*Z;
  Slice *node,*node1;

  i=left->length;
  k=code->length;
  S1= new Slice;
  S1->Init(i);
  N12= new Slice;
  N12->Init(i);
  N2= new Slice;
  N2->Init(i);
  X=new Slice;
  X->Init(i);
  Z=new Slice;
  Z->Init(i);
  Y= new Slice;
  Y->Init(k);
  l=code->size;
  node1= new Slice;
  node1->Init(l);
  node=new Slice;
  node->Init(l);
 //-----------------------------------------------------------------
   N12->CLR(); N2->CLR(); Y->SET();
   T->CLR(); M1->SetCol(N12,1);
   code->GetRow(node,1);
      S1->assign(S); Z->assign(S);
      while  (Z->SOME())
      {
    	  MATCH(left,S1,node,X);
          N12->OR(X);
          MATCH(right,S1,node,X);
          N2->OR(X);
            X->assign(N12); X->AND(N2); X->NOT();
            S1->AND(X);
 // Positions of edges forming a cycle are deleted from the slice S.
            Z->assign(N12); Z->OR(N2); Z->AND(S1);
// Positions of candidates for including into T(S) are selected by ones in the slice Z.
            if (Z->SOME())
            {
                MIN(weight,Z,X); i=X->FND();
                T->set(i,1); S1->set(i,0);
// The edge from the i-th position is added to T(S).
                  if (N12->get(i)==1)
                  {
                      right->GetRow(node,i);
                      left->GetRow(node1,i);
                  }
                  else
                  {
                	  right->GetRow(node1,i);
                	  left->GetRow(node,i);
                  }
// A new vertex is written in the variable node. }
                  MATCH(code,Y,node,X); 	k=X->FND();
                  MATCH(code,Y,node1,X); 	l=X->FND();
                  M1->GetCol(X,l); X->set(i,1);
                  M1->SetCol(X,k);
            }
      }
}
