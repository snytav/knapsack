#include "hip/hip_runtime.h"

#include "find.h"
#include "param.h"
//#include "star_kernel.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>     /* strcat */

#define LEVELS 10
#define OPT_THREADS 128
#define OPT_REDUCE 16
//#define fff
//#define QQQ
        const int NN2=(LENGTH1-1)/64+1;
        int threads2=(OPT_THREADS<NN2)?OPT_THREADS:NN2;
		int blocks2=(NN2-1)/threads2+1;

//    unsigned long long  int h_v[N],
     unsigned long long int *h_new_v;
    LongPointer  d_vfrst[LEVELS],d_vnumb[LEVELS];
    char ** tb;

//    int i,sh,*d_res,res[N],r_step;


__host__ __device__  int position_in_64bit_word(int num,int div)
{
	int res = num%div,t;

	if(num == 0) return 0;

	t = (res > 0) ? (num%div) : div;
	//printf("shift num %d div %d res %d t %d\n ",num,div,res,t);

	return  t;
}

__host__ __device__  int get_64bit_word(int num,int div)
{
	int res = num%div,t;

	if(num == 0) return 0;

	t = (res > 0) ? num/div : num/div-1;
	//printf("get64 num %d div %d res %d t %d\n ",num,div,res,t);

	return  t;
}
__global__ void set_kernel(unsigned long long int *dst,unsigned long long int *src)
{
#ifdef ssss
	printf("set_kernel %u %llu %llu \n",blockIdx.x,dst[blockIdx.x],src[blockIdx.x]);
#endif
	int ind=threadIdx.x + blockIdx.x*blockDim.x;
    dst[ind] = src[ind];
// printf("Get_Col[%i] %llu %llu \n ",ind,dst[ind],src[ind]);

#ifdef ssss
	printf("set_kernel %u %llu %llu \n",blockIdx.x,dst[blockIdx.x],src[blockIdx.x]);
#endif
}
__host__ __device__ void or_given_bit_to_position(unsigned long long int *x,int bit,int pos)
{
	unsigned long long int one = bit;

	if(pos == 0) return;

	*x |= one << (pos-1);
}

__host__ __device__ void set_given_bit_to_position(unsigned long long int *x,int bit,int pos)
{
	unsigned long long int one = bit;

	if(pos == 0) return;

	*x = one << (pos-1);
}



__host__ __device__ void assign_given_bit_to_position(unsigned long long int *x,int bit,int pos,int op)
{
	unsigned long long int one = 1,bit_pos,zero = 0,res,p;

	if(pos == 0) return;
	p = *x;

	bit_pos = one << (pos-1);

/*	if(pos == 3)
		{
		char b[100],nb[100],bp[100],ress[100];
		int d;
			long_to_binary(p,b,64);
			long_to_binary(bit_pos,bp,64);
			long_to_binary(~bit_pos,nb,64);
			res = bit_pos;
			res |= zero;
			d = p;
			long_to_binary(res,ress,64);
			printf("bit %d bit_pos %s not-bit_pos %s x %d %llu %s res %s \n",bit,bp,nb,d,p,b,ress);
		}
*/
	*x =    (op == OR)  * ( *x | ((one=bit) << (pos-1)) )
		  + (op == SET) * ( (bit == 1) ? (*x | bit_pos) : (*x & (~bit_pos)) );

}

__host__ __device__ void set_bit_to_position(unsigned long long int *x,int pos)
{
	set_given_bit_to_position(x,1,pos);//,SET);
}


__host__ __device__ void long_to_binary(unsigned long long  int x,char *b, unsigned int leng)
{
    //static char b[500];
    int s,lz;
    char bit;
    //b[SIZE_OF_LONG_INT] = '\0';
   // printf("\n %25llu \n",x);
    unsigned long long int z;

    s =  SIZE_OF_LONG_INT-1;//leng-1;
    z = 1;
    z <<= s;
    for (; z > 0; z >>= 1)
    {
    //	printf("z %llu log %d\n",z,(int)(log(z)/log(2.0)));
       // strcat(b, ((x & z) == z) ? "1" : "0");
    	lz = (int)(log((double)z)/log(2.0));
    	bit  = (((x & z) == z) ? '1' : '0');
    	b[lz] = bit;
        //printf("%10llu %s \n",z,b);
    }
    b[s+1] = 0;
//    puts("long_to_binary");
//   puts(b);
  /*  for(int i = 0;i < s/2;i++)
    {
    	char tmp;

    	tmp = b[i];
    	b[i] = b[s - i];
    	b[s - i] = tmp;
    }*/

    int term = (leng <  SIZE_OF_LONG_INT) ? leng :SIZE_OF_LONG_INT;
    b[term] = 0;

  //  return b;
}
__host__ __device__ void long_to_binary1(unsigned long long  int x,char *b, unsigned int leng)
{
    //static char b[500];
    int s,lz;
    char bit;
    //b[SIZE_OF_LONG_INT] = '\0';
   // printf("\n %25llu \n",x);
    unsigned long long int z;

    s = leng-1;
    z = 1;
    z <<= s;
    lz=0;
    for (; z > 0; z >>= 1)
    {
    //	printf("z %llu log %d\n",z,(int)(log(z)/log(2.0)));
       // strcat(b, ((x & z) == z) ? "1" : "0");
 //   lz=(int)(log((double)z)/log(2.0));
    	bit  = (((x & z) == z) ? '1' : '0');
    	b[lz] = bit;
    	lz++;
        //printf("%10llu %s \n",z,b);
    }
    b[leng] = 0;

    int term = (leng <  SIZE_OF_LONG_INT) ? leng :SIZE_OF_LONG_INT;
    b[term] = 0;

  //  return b;
}

//редукция большого массива 64-разрядных целых к массиву размером в 64 раза меньше,
//где каждому целому числу изначального массива соответствует 1 бит, ненулевой если
//в соответсвующем элементе исходного массива был хотя бы один ненулевлй бит
__device__ unsigned long long int get_bit_position(unsigned long long  int x,int n)
{
	int pos, sh;               // n - позиция вектора x в большом векторе правой части
	unsigned long long p;
#ifdef bbbb
	unsigned long long p0;  // n_minor - номер 64-битной последовательности в маленьком векторе слева
	char str[500];
#endif

	                          // sh - position of 1 in the 64 bit sequence meaning that the corresponding
	                          // element of the long int array "x" has some non-zero bit

	     // позиция первого ненулевого бита в 64-разрядном целом числе
	     pos = __ffsll(x);


	     //printf("get_bit_position pos %d x %llu n %d \n",pos,x,n);

	     //элементы массива нумеруются с нуля, биты с единицы
	     sh = position_in_64bit_word(n+1,SIZE_OF_LONG_INT);
	     //флаг наличия в векторе x хотя бы одного ненулевого бита
	     set_given_bit_to_position(&p,pos && 1,sh);
	     //if(n >= 32)
#ifdef bbbb
     //    long_to_binary(p0,str0);
         long_to_binary(p,str);
	     printf("get_bit_position x %25llu n %3d sh %2d pos %3d pos && 1 %3d p0 %25llu p %25llu %s \n",x,n,sh,pos,pos && 1,p0,p,str);
#endif
	     //возвращаем часть элемента левого массива, сооотвествующую одному элементу правого массива
	     // (часть, потому что весь 64-разрядный элемент левого, укороченного массива должен содержать информацию о )
	     // 64-х соседних элементах правого массива
	     return p;
}

//записывает элемент "левого" массива, если нет выхода за границу, и если номер нити кратен 64
__device__ int write_bit(int thr_n,int n,int lhs_size,unsigned long long  int *x)
{
	int permit = thr_n%SIZE_OF_LONG_INT == 0;
	//printf("thrn %d write_bit n %d permit %d returnx %d \n",thr_n,n,permit,(n < lhs_size));
	return (permit ? ((n < lhs_size) ? x[n] : 0 ) :1);
}

//возвращает элемент массива, если нет выход за границу
__device__ unsigned long long int get_array(unsigned long long  int *x,int n,int size)
{
	 //   if(n >= 32) printf("n %d size %d (n < size) %d reurn %llu \n",n,size,(n < size),((n < size) ? x[n] : 0));
        return ((n < size) ? x[n] : 0);
}

//редукция "большого" массива x, до массива new_x, меньшего по размеру в 64 раза,
//где каждому целому числу изначального массива соответствует 1 бит, ненулевой если
//в соответсвующем элементе исходного массива был хотя бы один ненулевой бит

void __global__ find(unsigned long long  int *x,unsigned long long  int *new_x, unsigned int N)
{
	 unsigned int n = threadIdx.x + blockIdx.x*blockDim.x;
	 __shared__ unsigned long long  int tmp[SIZE_OF_LONG_INT];
//	 int pos,sh,p;
	 int NNN;

      NNN=blockDim.x;
	 tmp[threadIdx.x] = get_bit_position(get_array(x,n,N),n);


//     pos = __ffsll(x[n]);

     //printf("pos %d \n",pos);

  //   num[n] = pos;

    // return;
//   	 unsigned int n_minor;
//    n_minor = n/SIZE_OF_LONG_INT; // n_minor это позиция 64-битной последовательности в векторе результата, в левом массиве
//
//     sh = n%SIZE_OF_LONG_INT; //номер бита в отдельном элементе 64-битной послеждовательности
//     p =  (pos && 1) << sh;
     //printf("threadIdx.x %d n %d n_minor %d size %d pos %d sh %d p %d pf %d new_xb %llu pos %d\n",threadIdx.x,
//    		 n,n_minor,
//    		 SIZE_OF_LONG_INT,
//    		 pos,
//    		 sh,
//    		 p,get_bit_position(x[n],n),
//    		 new_x[n_minor],num[n]);

//TODO: change "n" for work with further parts of array, n= tthreadIdx.x*Size_long

     //n= threadIdx.x*SIZE_OF_LONG_INT;

//     if(threadIdx.x <= N/SIZE_OF_LONG_INT+1)
//     {
////         for(int i = 0; i < N;i++)
//         {
//        	 printf("array i %d %llu direct-x %llu \n",i,get_array(x,i,N),x[i]);
//         }
         // в каждый элемент нового, укороченного массива пишут 64 нити, каждая из которых обрабатывает 64
    	 // 64-разрядных числа, начиная с n
	 __syncthreads();
	 if (threadIdx.x==0)
      new_x[blockIdx.x] =  get_array(tmp,0,NNN)
    	    	    	|  get_array(tmp,1,NNN)
    	    	    	|  get_array(tmp,2,NNN)
    	    	    	|  get_array(tmp,3,NNN)
    	    	        |  get_array(tmp,4,NNN)
    	    	        |  get_array(tmp,5,NNN)
    	    	        |  get_array(tmp,6,NNN)
    			        |  get_array(tmp,7,NNN)
    	         		|  get_array(tmp,8,NNN)
    			        |  get_array(tmp,9,NNN)
    	                |  get_array(tmp,10,NNN)
    	                |  get_array(tmp,11,NNN)
    	                |  get_array(tmp,12,NNN)
	                    |  get_array(tmp,13,NNN)
	                    |  get_array(tmp,14,NNN)
	                    |  get_array(tmp,15,NNN)
                        |  get_array(tmp,16,NNN)
                        |  get_array(tmp,17,NNN)
                        |  get_array(tmp,18,NNN)
            			|  get_array(tmp,19,NNN)
			            |  get_array(tmp,20,NNN)
		             	|  get_array(tmp,21,NNN)
                        |  get_array(tmp,22,NNN)
                        |  get_array(tmp,23,NNN)
                        |  get_array(tmp,24,NNN)
     	                |  get_array(tmp,25,NNN)
	                    |  get_array(tmp,26,NNN)
	                    |  get_array(tmp,27,NNN)
                        |  get_array(tmp,28,NNN)
                        |  get_array(tmp,29,NNN)
                        |  get_array(tmp,30,NNN)
	                    |  get_array(tmp,31,NNN)
		                |  get_array(tmp,32,NNN)
		                |  get_array(tmp,33,NNN)
                        |  get_array(tmp,34,NNN)
                        |  get_array(tmp,35,NNN)
                        |  get_array(tmp,36,NNN)
     	                |  get_array(tmp,37,NNN)
	                    |  get_array(tmp,38,NNN)
	                    |  get_array(tmp,39,NNN)
                        |  get_array(tmp,40,NNN)
                        |  get_array(tmp,41,NNN)
                        |  get_array(tmp,42,NNN)
			            |  get_array(tmp,43,NNN)
	            		|  get_array(tmp,44,NNN)
	             		|  get_array(tmp,45,NNN)
                        |  get_array(tmp,46,NNN)
                        |  get_array(tmp,47,NNN)
                        |  get_array(tmp,48,NNN)
      	                |  get_array(tmp,49,NNN)
	                    |  get_array(tmp,50,NNN)
	                    |  get_array(tmp,51,NNN)
                        |  get_array(tmp,52,NNN)
                        |  get_array(tmp,53,NNN)
                        |  get_array(tmp,54,NNN)
		                |  get_array(tmp,55,NNN)
	         	        |  get_array(tmp,56,NNN)
		                |  get_array(tmp,57,NNN)
                        |  get_array(tmp,58,NNN)
                        |  get_array(tmp,59,NNN)
                        |  get_array(tmp,60,NNN)
                        |  get_array(tmp,61,NNN)
                        |  get_array(tmp,62,NNN)
                        |  get_array(tmp,63,NNN);

//         		           get_bit_position(get_array(x,n+1,N),n+1) |
//		                   get_bit_position(get_array(x,n+2,N),n+2) |
//		                   get_bit_position(get_array(x,n+3,N),n+3) |
//		                   get_bit_position(get_array(x,n+4,N),n+4) |
//		                   get_bit_position(get_array(x,n+5,N),n+5) |
//		                   get_bit_position(get_array(x,n+6,N),n+6) |
//		                   get_bit_position(get_array(x,n+7,N),n+7) |
//		                   get_bit_position(get_array(x,n+8,N),n+8) |
//		                   get_bit_position(get_array(x,n+9,N),n+9) |
//		                   get_bit_position(get_array(x,n+10,N),n+10) |
//		                   get_bit_position(get_array(x,n+11,N),n+11) |
//		                   get_bit_position(get_array(x,n+12,N),n+12) |
//		                   get_bit_position(get_array(x,n+13,N),n+13) |
//		                   get_bit_position(get_array(x,n+14,N),n+14) |
//		                   get_bit_position(get_array(x,n+15,N),n+15) |
//		                   get_bit_position(get_array(x,n+16,N),n+16) |
//		                   get_bit_position(get_array(x,n+17,N),n+17) |
//		                   get_bit_position(get_array(x,n+18,N),n+18) |
//		                   get_bit_position(get_array(x,n+19,N),n+19) |
//		                   get_bit_position(get_array(x,n+20,N),n+20) |
//		                   get_bit_position(get_array(x,n+21,N),n+21) |
//		                   get_bit_position(get_array(x,n+22,N),n+22) |
//		                   get_bit_position(get_array(x,n+23,N),n+23) |
//		                   get_bit_position(get_array(x,n+24,N),n+24) |
//		                   get_bit_position(get_array(x,n+25,N),n+25) |
//		                   get_bit_position(get_array(x,n+26,N),n+26) |
//		                   get_bit_position(get_array(x,n+27,N),n+27) |
//		                   get_bit_position(get_array(x,n+28,N),n+28) |
//		                   get_bit_position(get_array(x,n+29,N),n+29) |
//		                   get_bit_position(get_array(x,n+30,N),n+30) |
//		                   get_bit_position(get_array(x,n+31,N),n+31) |
//		                   get_bit_position(get_array(x,n+32,N),n+32) |
//		                   get_bit_position(get_array(x,n+33,N),n+33) |
//		                   get_bit_position(get_array(x,n+34,N),n+34) |
//		                   get_bit_position(get_array(x,n+35,N),n+35) |
//		                   get_bit_position(get_array(x,n+36,N),n+36) |
//		                   get_bit_position(get_array(x,n+37,N),n+37) |
//		                   get_bit_position(get_array(x,n+38,N),n+38) |
//		                   get_bit_position(get_array(x,n+39,N),n+39) |
//		                   get_bit_position(get_array(x,n+40,N),n+40) |
//		                   get_bit_position(get_array(x,n+41,N),n+41) |
//		                   get_bit_position(get_array(x,n+42,N),n+42) |
//		                   get_bit_position(get_array(x,n+43,N),n+43) |
//		                   get_bit_position(get_array(x,n+44,N),n+44) |
//		                   get_bit_position(get_array(x,n+45,N),n+45) |
//		                   get_bit_position(get_array(x,n+46,N),n+46) |
//		                   get_bit_position(get_array(x,n+47,N),n+47) |
//		                   get_bit_position(get_array(x,n+48,N),n+48) |
//		                   get_bit_position(get_array(x,n+49,N),n+49) |
//		                   get_bit_position(get_array(x,n+50,N),n+50) |
//		                   get_bit_position(get_array(x,n+51,N),n+51) |
//		                   get_bit_position(get_array(x,n+52,N),n+52) |
//		                   get_bit_position(get_array(x,n+53,N),n+53) |
//		                   get_bit_position(get_array(x,n+54,N),n+54) |
//		                   get_bit_position(get_array(x,n+55,N),n+55) |
//		                   get_bit_position(get_array(x,n+56,N),n+56) |
//		                   get_bit_position(get_array(x,n+57,N),n+57) |
//		                   get_bit_position(get_array(x,n+58,N),n+58) |
//		                   get_bit_position(get_array(x,n+59,N),n+59) |
//		                   get_bit_position(get_array(x,n+60,N),n+60) |
//		                   get_bit_position(get_array(x,n+61,N),n+61) |
//		                   get_bit_position(get_array(x,n+62,N),n+62) |
//		                   get_bit_position(get_array(x,n+63,N),n+63) |
//		                   get_bit_position(get_array(x,n+64,N),n+64) |
//                           0
//    	                  ); // |
		                   // get_bit_position(x[n+4],n+4);

//     if(threadIdx.x==0)    printf("new_xa %llu n_minor %d \n",new_x[blockIdx.x],blockIdx.x);
     //}
}

__host__ __device__ int get_position_bit(unsigned long long int *h,int n)
{
	int num,sh;
	unsigned long long int p = 1;

	num = get_64bit_word(n,SIZE_OF_LONG_INT);
	sh =  position_in_64bit_word(n,SIZE_OF_LONG_INT);


	set_bit_to_position(&p,sh);
//	p = p << sh;
#ifdef bbb
	printf("get_positio_bit n %d num %d sh %d shifted p %llu \n",n,num,h[num],p);
#endif

	return (h[num] & p ) && 1;
}

__global__ void copy_block(unsigned long long int *dv,unsigned long long int *dv0)
{	    __syncthreads();
	int tid=threadIdx.x+ blockIdx.x*blockDim.x;
    if (tid<NN2)dv[tid]=dv0[tid];
}
__global__ void copy_block1(unsigned long long int *dv,unsigned long long int *dv0)
{   int k,tid=threadIdx.x + blockIdx.x*blockDim.x;
    unsigned long long int zero=1;
    if(tid<NN2)  dv[tid]=dv0[tid];
//    else dv[tid]=0;
    if (tid==(NN2-1)) // in the last element need to zero the tail
    {
    	/*zero=(1<<(num % SIZE_OF_LONG_INT)-1)-1;
	  zero=~zero;*/
    	k=(LENGTH1%SIZE_OF_LONG_INT);
    	zero=(zero<<k)-1;
//    	printf("k=%i  %llu \n",k, zero);
    	if (k!=0)
    		dv[tid]=dv0[tid]&zero;
    	else dv[tid]=dv0[tid];
    }


//	printf("numb_%i %i\n",tid,__popcll(dv0[tid]));
}
void reduce_array(unsigned long long  int *d_v1,unsigned long long  int*d_v,unsigned int size,unsigned int level, unsigned int N)
{
//	char s1[1000],s2[1000];
//	unsigned long long int h_new_v[N],h_v[N];
	hipError_t err1;//,err0;

	hipError_t err = hipGetLastError();
//	printf("errors at enter reduce_array %d\n",err);

	unsigned int blocks, threads = (size < SIZE_OF_LONG_INT) ? size : SIZE_OF_LONG_INT;

//	err0 = hipMemcpy(h_v,d_v,sizeof(unsigned long long  int)*size,hipMemcpyDeviceToHost);
//	printf("size %d err %d %s  %p\n",size,err0,hipGetErrorString(err0),d_v);

//		printf("size1 %d \n",size);

	blocks = (int)ceil( ((double)size)/threads);
//    printf("reduce_array#####  size %d blocks %d threads %d \n",size,blocks, threads);

    find<<<blocks,threads>>>(d_v,d_v1,size);

    hipDeviceSynchronize();

    err1 = hipGetLastError();

    if(err1 != hipSuccess)
    	{
#ifdef frst
    	printf("kernel error %d %s size %d\n",err1,hipGetErrorString(err1),size);
#endif
    		exit(0);
    	}
#ifdef frst
	err0 = hipMemcpy(h_new_v,d_v1,sizeof(unsigned long long  int)*size,hipMemcpyDeviceToHost);
	if(err0 != hipSuccess)
	{

		printf("D2H error0 %d %s\n",err0,hipGetErrorString(err0));
		exit(0);
	}
	err1 = hipMemcpy(h_v,d_v,sizeof(unsigned long long  int)*size,hipMemcpyDeviceToHost);
	//printf("h_new0 %ul\n",h_new_v[0]);
	//err = hipMemcpy(res,d_res,sizeof(int)*size,hipMemcpyDeviceToHost);
        //printf("h_new0 %ul\n",h_new_v[0]);
	//err1 = hipMemcpy(h_v,d_v,sizeof(unsigned long long  int)*size,hipMemcpyDeviceToHost);


	printf("D2H error %d %s\n",err1,hipGetErrorString(err1));
	FILE *f_res;
    char fname[100];

	sprintf(fname,"result%02d.dat",level);
	if( (f_res =fopen(fname,"wt")) == NULL ) return 0;
	for(int i = 0;i < size;i++)
	{
		   long_to_binary(h_v[i],s1);
		   long_to_binary(h_new_v[i],s2);
		   //printf("i %3d %s,%25llu res %d\n",i,s2,h_new_v[i],res[i]);
	       fprintf(f_res,"i %3d %s,%25llu result_vector %d init %s \n",i,s2,h_new_v[i],get_position_bit(h_new_v,i),s1);
	      // printf("i %3d %s,%25llu res %d\n",i,s2,h_new_v[i],res[i]);
	}
	fclose(f_res);
#endif
}

__global__ void first_non_zero(unsigned long long int *d_v,int *n,int size,int *d_first_non_zero)
{


	    if(*n == -1)
	    {
	    	*d_first_non_zero = 0;
	    	return;
	    }
//TODO: 1. needed to define position in the whole initial bit sequence, not only in one array element
//	    2. 0th bit of the second array element must be somehow 65  !!!!
//solution: make a "kosher" % function
//and a "kosher" set-to-position function
	    int nz = ((size == 1) ? __ffsll(d_v[0]) : (__ffsll(d_v[*n-1]) + (*n-1)*size) );
	    (*d_first_non_zero) = nz;
	    printf("first_non_zero n %d size %d nz %d  ffsll %d to-add %d\n",*n,size,nz,__ffsll(d_v[*n-1]),(*n-1)*size);
}

__global__ void first_backward(LongPointer *d_v,int *d_first_non_zero,int level)
{
	int f[LEVELS];
	unsigned long long int *dvl,u;
	char lprt[100];

//	printf("inverse level %d \n",level);


	f[level+1] = 1;
	while(level >= 0)
	{
		dvl = d_v[level];
		int index1 = f[level+1]-1;// + (f[level+1]-1)*SIZE_OF_LONG_INT;
        u = dvl[index1];
#ifdef fff
        long_to_binary(u,lprt,LENGTH1);
		printf("element number %d at level %d %llu %s (numbers in array from 0, positions in bit sequence from 1)\n",
				index,level+1,u,lprt);
#endif
	    f[level] = __ffsll(u) + index1*SIZE_OF_LONG_INT;
#ifdef fff
        printf("level %d u %llu %s f[level] %d\n",level,u,lprt,f[level]);
#endif
//        if(level == 0)return;
		//printf("level %d f %d f[+1] %d\n",level,f[level],f[level+1]);
		level--;
	}
	*d_first_non_zero = f[0];// + (f[1]-1)*SIZE_OF_LONG_INT;
	if (*d_first_non_zero>LENGTH1) *d_first_non_zero=0;
#ifdef ffff
	printf("d_first_non_zero %d  pointer= %p\n",*d_first_non_zero,d_first_non_zero);
#endif
}

int first(unsigned long long int *dv0,int size,int *d_first_non_zero, unsigned int N)
{
	    static int frst=1;
	    static LongPointer *dev_d_v;
	    int big_n = size,level = 0,n=1;


	    hipError_t err = hipGetLastError(),err_m,err_c;
#ifdef QQQ
	    char str[100];
	    print_device_bit_row("first0",dv0,big_n*SIZE_OF_LONG_INT,0,N);
#endif
	//    hipMemcpy(d_v[0],dv0,N*sizeof(unsigned long long  int),hipMemcpyDeviceToDevice); //must be!!!
	    copy_block<<<1,N>>>(d_vfrst[0],dv0);
	    hipDeviceSynchronize();
#ifdef QQQ
        print_device_bit_column("first1",dv0,big_n*SIZE_OF_LONG_INT,N);
	    	printf("errors at enter first %d\n",err);
	    	printf("START n %3d big_n %3d level %d \n ",n,big_n,level);
#endif
	    for(big_n = size; big_n > 1; big_n  = (int)ceil((double)big_n/(double)SIZE_OF_LONG_INT))
	    {
	    	n = (int)ceil((double)big_n/(double)SIZE_OF_LONG_INT);
#ifdef QQQ
	    	printf("n %3d big_n %3d level %d \n ",n,big_n,level);
	    	hipError_t err = hipGetLastError();
	    	printf("errors before reduce %d\n",err);

	    	sprintf(str,"level%02d",level);
	    	print_device_bit_column(str,dv[level],big_n*SIZE_OF_LONG_INT,N);
#endif
	        reduce_array(d_vfrst[level+1],d_vfrst[level],big_n,level,N);
#ifdef QQQ
	    	sprintf(str,"level%02d_result",level);
	    	print_device_bit_column(str,dv[level+1],big_n,N);
	        err = hipGetLastError();
	       	    	printf("errors at after reduce %d\n",err);
#endif
	        level++;

	    }
// printf("FND: level=%i \t",level);
	    if (frst==1)
	    {
	    	err_m = hipMalloc(&dev_d_v,sizeof(LongPointer)*LEVELS);
	    	err_c = hipMemcpy(dev_d_v,d_vfrst,sizeof(LongPointer)*LEVELS,hipMemcpyHostToDevice);
	    	frst=0;
	    }

#ifdef ffff
        printf("malloc %d copy %d\n",err_m,err_c);
#endif
	    err = hipGetLastError();
//	   	printf("errors at before inverse %d %s\n",err,hipGetErrorString(err));
	   	    	       	    	//TODO: make a device copy of the d_v array and set it as 1st parameter of first_backward
//        puts("INVERSE");
	    first_backward<<<1,1>>>(dev_d_v,d_first_non_zero,level);
	    hipDeviceSynchronize();
	    err = hipGetLastError();
//	    	       	    	printf("errors at after inverse %d %s\n",err,hipGetErrorString(err));
//	    while(level >= 0)
//	    {
//	    	int h_first_non_zero;
//	    	hipMemcpy(&h_first_non_zero,d_first_non_zero,sizeof(int),hipMemcpyDeviceToHost);
//	    	printf("n %3d level %d first non-zero %5d \n",n,level,h_first_non_zero);
//	        first_non_zero<<<1,1>>>(d_v[level],d_first_non_zero,n,d_first_non_zero);
//
//
//	        hipMemcpy(&h_first_non_zero,d_first_non_zero,sizeof(int),hipMemcpyDeviceToHost);
//	        printf("n %3d level %d first non-zero %5d \n",n,level,h_first_non_zero);
//
//	        n *= SIZE_OF_LONG_INT;
//	        level--;
//	    }

	return 0;
}
__global__ void some_backward(LongPointer d_v,int *d_first_non_zero)
{
		*d_first_non_zero= (d_v[0]!=0)?1:0;
}
int some(unsigned long long int *dv0,int size,int *d_first_non_zero,unsigned int N)
{
	    static int frst=1;
	    static LongPointer *dev_d_v;
	    int big_n = size,level = 0;//,n=1;
	    hipError_t err = hipGetLastError();
	    copy_block<<<1,N>>>(d_vfrst[0],dv0);

	    for(big_n = size; big_n > 1; big_n  = (int)ceil((double)big_n/(double)SIZE_OF_LONG_INT))
	    {
//	    	n = (int)ceil((double)big_n/(double)SIZE_OF_LONG_INT);
	        reduce_array(d_vfrst[level+1],d_vfrst[level],big_n,level,N);
	        level++;
	    }

	    if (frst==1)
	    {
	    	hipMalloc(&dev_d_v,sizeof(LongPointer)*LEVELS);
	    	hipMemcpy(dev_d_v,d_vfrst,sizeof(LongPointer)*LEVELS,hipMemcpyHostToDevice);
	    	frst=0;
	    }

	    err = hipGetLastError();
	    some_backward<<<1,1>>>(d_vfrst[level],d_first_non_zero);
	return 0;
}


const unsigned long long int m[6]={0x5555555555555555,
				  0x3333333333333333,
                0x0f0f0f0f0f0f0f0f,
				  0x00ff00ff00ff00ff,
	    		  0x0000ffff0000ffff,
				  0x00000000ffffffff};
unsigned long long int *d_m;

__device__ void numb_shift1(LongPointer d_v,unsigned long long int *m ,int red_numb,int N)
{ int i;
  int index=threadIdx.x + blockIdx.x*blockDim.x;
  unsigned long long int b1,b2;
   if (index<N)
   {  if(d_v[index]>0)
	   for(int j=0; j<3; j++)
	   {
	   i=1<<red_numb;
	   b1=d_v[index]&m[red_numb];
	   b2=(d_v[index]>>i)&m[red_numb];
	   d_v[index]=b1+b2;
	   red_numb++;}
   }
}

__device__ void numb_reduce(LongPointer dv, LongPointer dv1,int red_numb, int N)
{   //blockDim.x=min(OPT_THREADS,2^(2^red_num -1))
	__shared__ unsigned long long int cache[OPT_THREADS];
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	int cacheIndex=threadIdx.x;
	int red_count=(red_numb==0)?OPT_REDUCE:min(OPT_THREADS,blockDim.x);
//	printf("red_numb=%i, red_count=%i\n", red_numb, red_count);
	int i=OPT_THREADS;

	if (N>1)
	{
	cache[cacheIndex]=get_array(dv,tid,N);
	while (red_count <= i) i >>= 1;// минимальная степень двойки>blockDim.x
//	if(tid==0) printf("\n red_count=%i, i=%i \n",red_count,i);
	__syncthreads();
	while (i!=0)
	{
//		if (cacheIndex<i)//
		if (cacheIndex%red_count<i)
		{
			cache[cacheIndex]+=get_array(cache,cacheIndex+i,blockDim.x);
//		printf(" %i+%i:%llu  ",cacheIndex,cacheIndex+i,get_array(cache,cacheIndex+i,blockDim.x));
		}
		__syncthreads();
		i/=2;
	}
//	if (cacheIndex==0)//
	if (cacheIndex%red_count==0)
		{
	dv1[tid/red_count]=cache[cacheIndex];
//		dv1[blockIdx.x]=cache[0];
//		printf("red_count=%i   dv1[%i:%i]=%llu   ",red_count,tid/red_count,cacheIndex,cache[cacheIndex]);
		};
	}
	else dv1[tid]=dv[tid];
}
__global__ void numb_shift(LongPointer d_v,LongPointer d_v1,unsigned long long int *d_m ,int red_numb,int N)
{  int i=3*red_numb;
    if(i<6)	numb_shift1(d_v,d_m,i,N);//3 шага
    __syncthreads();
	numb_reduce(d_v,d_v1,i,N); // свертка на 16:128 или копирование
}

__global__ void numb_backward(LongPointer d_v,int *d_first_non_zero)
{
		*d_first_non_zero=d_v[0];
//		if (d_v[0]>0)printf("numb: %i\n",*d_first_non_zero);
}
int number(unsigned long long int *dv0,int size, int *d_numb,unsigned int N)
{
    static int frst=1;
    static LongPointer *dev_d_v;
    int //big_n = size,
    		level = 0;//,n=1;
 //   int i;// , red_count;
    hipError_t err = hipGetLastError();
    err=hipMemset(d_numb,0,sizeof(int));
 //   printf("hipMemset %d , %s \n",err,hipGetErrorString(err));
    unsigned int blocks, threads = (size < OPT_THREADS) ? size : OPT_THREADS;
    blocks = (int)ceil( ((double)size)/threads);
    copy_block1<<<blocks,threads>>>(d_vnumb[0],dv0);

//    for(level=0; level < 2;level++)
//    {
 //   	    threads = (size < OPT_THREADS) ? size : OPT_THREADS;
 //   	    blocks = (int)ceil( ((double)size)/threads);
 //       printf("size %d blocks %d threads %d \n",size,blocks, threads);
       numb_shift<<<blocks,threads>>>(d_vnumb[level],d_vnumb[level+1],d_m,level,size);//3 шага, свертка не более, чем на 16
       if (size>1) size=((size-1)/OPT_REDUCE)+1;
       level++;

                threads = (size < OPT_THREADS) ? size : OPT_THREADS;
          	    blocks = (int)ceil( ((double)size)/threads);
//              printf("size %d blocks %d threads %d \n",size,blocks, threads);
             numb_shift<<<blocks,threads>>>(d_vnumb[level],d_vnumb[level+1],d_m,level,size);//3 шага, свертка не более чем на 128
             if (size>1) size=((size-1)/OPT_THREADS)+1;
             level++;
//       printf("\n=========================== level=%i, red_count=%i, size=%i \n",level, red_count, big_n);
 //       printf("level=%i, size=%i, red_count=%i \n",level,big_n, red_count);
//    }

//Если необходимо еще сворачивать (N>1048576)
    while (size>1)
    { threads = (size < OPT_THREADS) ? size : OPT_THREADS;
      blocks = (int)ceil( ((double)size)/threads);
//    printf("size %d blocks %d threads %d \n",size,blocks, threads);
      numb_shift<<<blocks,threads>>>(d_vnumb[level],d_vnumb[level+1],d_m,level,size);//только свертка
      size=((size-1)/OPT_THREADS)+1;
      level++;
    }

    if (frst==1)
    {
    	hipMalloc(&dev_d_v,sizeof(LongPointer)*LEVELS);
    	hipMemcpy(dev_d_v,d_vfrst,sizeof(LongPointer)*LEVELS,hipMemcpyHostToDevice);
    	frst=0;
    }

    err = hipGetLastError();
    numb_backward<<<1,1>>>(d_vnumb[level],d_numb);
return 0;
}
__global__ void copy_block_plus(unsigned long long int *dv,unsigned long long int *dv0, LongPointer *d_tab, unsigned long long int *d_and, int j)
		{   int k,tid=threadIdx.x + blockIdx.x*blockDim.x;
		    unsigned long long int zero=1;

		    unsigned long long int *d_col;

		      d_col=d_tab[j-1];//d_tab[i];
		 //     _assign(d_res,d_col);
		 //     _and(d_res,d_and);
		      if(tid<NN2)  dv0[tid]=d_col[tid]&d_and[tid];

		    if(tid<NN2)  dv[tid]=dv0[tid];
		//    else dv[tid]=0;
		    if (tid==(NN2-1)) // in the last element need to zero the tail
		    {
		    	/*zero=(1<<(num % SIZE_OF_LONG_INT)-1)-1;
			  zero=~zero;*/
		    	k=(LENGTH1%SIZE_OF_LONG_INT);
		//    	printf("k=%i\n",k);
		    	zero=(zero<<k)-1;
		    	if (k!=0) dv0[tid]&=zero;
		    	dv[tid]=dv0[tid];
		    }
		}
__global__ void numb_backward_plus(LongPointer d_v,int *d_first_non_zero)
{
		*d_first_non_zero+=d_v[0];
//		if (d_v[0]>0)printf("numb:+%i= %i\n",d_v[0],*d_first_non_zero);
}
int number_plus(LongPointer *d_tab, unsigned long long int *d_and, int j,unsigned long long int *dv0, int size, int *d_numb,unsigned int N)
{
//    static int frst=1;
//    static LongPointer *dev_d_v;
    int big_n = size,level = 0;//,n=1;
    int i, red_count;
    hipError_t err = hipGetLastError();
////    err=hipMemset(d_numb,0,sizeof(int));
 //   printf("hipMemset %d , %s \n",err,hipGetErrorString(err));
    unsigned int blocks, threads = (size < OPT_THREADS) ? size : OPT_THREADS;
    blocks = (int)ceil( ((double)size)/threads);
    copy_block_plus<<<blocks,threads>>>(d_vnumb[0],dv0, d_tab, d_and,j);

//    for(level=0; level < 2;level++)
//    {
 //   	    threads = (size < OPT_THREADS) ? size : OPT_THREADS;
 //   	    blocks = (int)ceil( ((double)size)/threads);
 //       printf("size %d blocks %d threads %d \n",size,blocks, threads);
       numb_shift<<<blocks,threads>>>(d_vnumb[level],d_vnumb[level+1],d_m,level,size);//3 шага, свертка не более, чем на 16
       if (size>1) size=((size-1)/OPT_REDUCE)+1;
       level++;

                threads = (size < OPT_THREADS) ? size : OPT_THREADS;
          	    blocks = (int)ceil( ((double)size)/threads);
//              printf("size %d blocks %d threads %d \n",size,blocks, threads);
             numb_shift<<<blocks,threads>>>(d_vnumb[level],d_vnumb[level+1],d_m,level,size);//3 шага, свертка не более чем на 128
             if (size>1) size=((size-1)/OPT_THREADS)+1;
             level++;
//       printf("\n=========================== level=%i, red_count=%i, size=%i \n",level, red_count, big_n);
 //       printf("level=%i, size=%i, red_count=%i \n",level,big_n, red_count);
//    }

//Если необходимо еще сворачивать (N>1048576)
    while (size>1)
    { threads = (size < OPT_THREADS) ? size : OPT_THREADS;
      blocks = (int)ceil( ((double)size)/threads);
//    printf("size %d blocks %d threads %d \n",size,blocks, threads);
      numb_shift<<<blocks,threads>>>(d_vnumb[level],d_vnumb[level+1],d_m,level,size);//только свертка
      size=((size-1)/OPT_THREADS)+1;
      level++;
    }

/*    if (frst==1)
    {
    	hipMalloc(&dev_d_v,sizeof(LongPointer)*LEVELS);
    	hipMemcpy(dev_d_v,d_vfrst,sizeof(LongPointer)*LEVELS,hipMemcpyHostToDevice);
    	frst=0;
    }
*/
    err = hipGetLastError();
    numb_backward_plus<<<1,1>>>(d_vnumb[level],d_numb);
return 0;
}

__host__ __device__ void assign_bit(unsigned long long int *h_v,int nz,int bit,int op)
{
   int ni;

   ni = get_64bit_word(nz,SIZE_OF_LONG_INT);
   int pos = position_in_64bit_word(nz,SIZE_OF_LONG_INT);

   assign_given_bit_to_position(&h_v[ni],bit,pos,op);
   //set_bit_to_position(&h_v[ni],pos);
 }

__host__ __device__ void set_bit(unsigned long long int *h_v,int nz)
{
   int ni;

   ni = get_64bit_word(nz,SIZE_OF_LONG_INT);
   int pos = position_in_64bit_word(nz,SIZE_OF_LONG_INT);

   set_bit_to_position(&h_v[ni],pos);
 }

void print_host_bit_column(char *label,unsigned long long *h_v,int length)
{
     FILE *f_ini;
     char s[1000];

     sprintf(s,"%s_bit.dat",label);
     if((f_ini =fopen(s,"wt"))== NULL) return;

     for(int i = 1;i <= length;i++)
     {
             fprintf(f_ini,"%10d %d \n",i,get_position_bit(h_v,i));
     }
     fclose(f_ini);
}

void print_device_bit_column(char *label,unsigned long long *d_v,int length,unsigned int N)
{
//	 static unsigned long long *h_v;
//	 static int flag_malloc=1;

/*	 if(flag_malloc==1)
	 {
	 h_v = (unsigned long long *)malloc(N*sizeof(unsigned long long));
	 flag_malloc=0;
	 }*/
     hipMemcpy(h_new_v,d_v,N*sizeof(unsigned long long),hipMemcpyDeviceToHost);

     print_host_bit_column(label,h_new_v,length);

//     free(h_v);
}

 void print_device_bit_row(char *label,unsigned long long *d_v,int length,int row_flag,unsigned int N)
{
//	 unsigned long long *h_v;
//	static int flag_malloc=1;
	char s[N1][65];
	char bit_row[N1*65+1];
	FILE *f_ini;
	char fname[1000];

//	sprintf(fname,"%s.dat",label);
//	if((f_ini =fopen(fname,"wt"))== NULL) return;
//	fprintf(f_ini,"QQQQQQQ \n");
//	fclose(f_ini);



 //    if (flag_malloc==1)
 //    {
//	 h_v = (unsigned long long *)malloc(N*sizeof(unsigned long long));
//     flag_malloc=0;
 //    }
     hipError_t err = hipMemcpy(h_new_v,d_v,N*sizeof(unsigned long long),hipMemcpyDeviceToHost);
     strcpy(bit_row,"");

     for (int i = 0;i < N;i++)
     {

         long_to_binary(h_new_v[i],s[i],64);
//        puts(s[i]);
//         printf("%s %llu\n",s[i],h_v[i]);
//         fprintf(f_ini,"%s",s);

         strcat(bit_row,s[i]);
     }

 	 sprintf(fname,"%s.dat",label);
     bit_row[length] = 0;
     if(row_flag == 1)
     {
     	if((f_ini =fopen(fname,"wt"))== NULL) return;
        fprintf(f_ini,"%s\n",bit_row);

        fclose(f_ini);
     }
     else
     {
    	 if((f_ini =fopen(fname,"wt"))== NULL) return;
    	 for(int i = 0;i < length;i++)
    	 {
    		 fprintf(f_ini,"%5d %3c\n",i+1,bit_row[i]);
    	 }
    	 fclose(f_ini);
     }
     puts(label);
     puts(bit_row);
//     fclose(f_ini);
    // free(h_v);
}

void InitArrays()
{
	h_new_v=(unsigned long long  int *)malloc(sizeof(unsigned long long  int)*N1);
	tb=new char*[M];
    for(int i = 0;i < LEVELS;i++)
    {
	    unsigned long long *tmp;
        hipError_t err = hipMalloc(&tmp,sizeof(unsigned long long  int)*N1);
#ifdef ffff
        printf("hipMalloc %d err %d %s %p \n",i,err,hipGetErrorString(err),tmp);
#endif
        d_vfrst[i] = tmp;
        hipMemset(d_vfrst[i],0,sizeof(unsigned long long  int)*N1);
    }

    for(int i = 0;i < LEVELS;i++)
    {
	    unsigned long long *tmp;
        hipError_t err = hipMalloc(&tmp,sizeof(unsigned long long  int)*N1);
#ifdef ffff
        printf("hipMalloc %d err %d %s %p \n",i,err,hipGetErrorString(err),tmp);
#endif
        d_vnumb[i] = tmp;
        hipMemset(d_vnumb[i],0,sizeof(unsigned long long  int)*N1);
    }

	for(int i=0; i<M;i++) tb[i]=new char[LENGTH1+1];

//копируется константный массив для вычисления numb
	 hipMalloc(&d_m,sizeof(unsigned long long  int)*6);
	 hipError_t err_m= hipMemcpy(d_m,m,6*sizeof(unsigned long long  int),hipMemcpyHostToDevice);
	 printf("hipMalloc err %d %s\n",err_m,hipGetErrorString(err_m));
//    printf("m=[%llu,%llu,%llu,%llu,%llu,%llu] \n",m[0],m[1],m[2],m[3],m[4],m[5]);

}

//int main(void)
//{
//    unsigned long long  int h_v[N],h_new_v[N];
//    LongPointer  d_v[LEVELS];
//    int i,sh,*d_res,res[N],r_step;
//    char s1[1000],s2[1000],lprt[500];
//    FILE *f_ini,*f_res;
//    unsigned int blocks, threads = SIZE_OF_LONG_INT;
//    int *d_first_non_zero;
//    unsigned long long  int one = 1;
//
//    hipMalloc(&d_first_non_zero,sizeof(int));
//
//    blocks = (int)ceil( ((double)N)/((double)SIZE_OF_LONG_INT));
//
//    srandom(time(NULL));
//    r_step = random()%10 +3;
//
//    printf("random step %d\n",r_step);
//
//    if((f_ini =fopen("init.dat","wt"))== NULL) return 0;
//
//    set_bit(h_v,POS_NON_ZERO);
//
//    for(i = 0;i < N;i++)
//    {
//    	    h_v[i] = 0;
//    }
//
//
//    print_host_bit_column("init",h_v,LENGTH);
//    set_bit(h_v,POS_NON_ZERO);
//    print_host_bit_column("init1",h_v,LENGTH);
//
//    for(i = 0;i < N;i++)
//        {
//        //	    h_v[i] = 0;
//        	  	long_to_binary(h_v[i],lprt);
//                fprintf(f_ini,"init %3d %25llu %s \n",i,h_v[i],lprt);
//        }
//    fclose(f_ini);
//   // exit(0);
//
//    for(int i = 0;i < LEVELS;i++)
//    {
//    	unsigned long long *tmp;
//        hipError_t err = hipMalloc(&tmp,sizeof(unsigned long long  int)*N);
//        printf("hipMalloc %d err %d %s %p \n",i,err,hipGetErrorString(err),tmp);
//        d_v[i] = tmp;
//        hipMemset(d_v[i],0,sizeof(unsigned long long  int)*N);
//    }
//
////    hipMalloc(&d_v,sizeof(unsigned long long  int)*N);
////
////    hipMalloc(&d_res,sizeof(int)*N);
////
////    hipMalloc(&d_v2,sizeof(unsigned long long  int)*N);
////    hipMemset(d_v2,0,sizeof(unsigned long long  int)*N);
////
////    hipMalloc(&d_v3,sizeof(unsigned long long  int)*N);
////    hipMemset(d_v3,0,sizeof(unsigned long long  int)*N);
//
//    hipError_t err1 = hipMemcpy(d_v[0],h_v,sizeof(unsigned long long  int)*N,hipMemcpyHostToDevice);
//    	printf("errors at after copy %d\n",err1);
//    //printf("H2D error %d %s\n",err1,hipGetErrorString(err1));
//
////    hipMemcpy(h_v1,d_v,sizeof(unsigned long long  int)*N,hipMemcpyHostToDevice);
////    err1 = hipGetLastError();
////        printf("debug copy error %d %s\n",err1,hipGetErrorString(err1));
////TODO:
//
////
//
//
//    //  6. then up to 64x64
//    //  7. second stage
//    //  8. reverse
//    //  9. profile
//
//    //один блок,число нитей N/SIZE_OF_LONG_INT+1
//    //пишет только одна из 64
//
//    first(d_v,N,d_first_non_zero);
//
//    return 0;
//
////    int big_n = N,level = 0,n;
////    for(big_n = N; big_n > 1; big_n  = (int)ceil((double)big_n/(double)SIZE_OF_LONG_INT))
////    {
////    	n = (int)ceil((double)big_n/(double)SIZE_OF_LONG_INT);
////    	printf("n %3d big_n %3d level %d \n ",n,big_n,level);
////        reduce_array(d_v[level+1],d_v[level],big_n,level);
////
////        level++;
////
////    }
////
////
////    return 0;
//
////
////   // find<<<blocks,threads>>>(d_v,d_v1,d_res);
////
////    err1 = hipGetLastError();
////    //printf("kernel error %d %s\n",err1,hipGetErrorString(err1));
////
////	//hipError_t err,err0 = hipMemcpy(h_new_v,d_v1,sizeof(unsigned long long  int)*N,hipMemcpyDeviceToHost);
////	//printf("h_new0 %ul\n",h_new_v[0]);
//////	err = hipMemcpy(res,d_res,sizeof(int)*N,hipMemcpyDeviceToHost);
////        //printf("h_new0 %ul\n",h_new_v[0]);
////
////
////	//printf("D2H error %d %s\n",err,hipGetErrorString(err));
////	//printf("D2H error0 %d %s\n",err0,hipGetErrorString(err0));
////
////	if( (f_res =fopen("result.dat","wt")) == NULL ) return 0;
////	for(i = 0;i < N;i++)
////	{
////		   long_to_binary(h_v[i],s1);
////		   long_to_binary(h_new_v[i],s2);
////		   //printf("i %3d %s,%25llu res %d\n",i,s2,h_new_v[i],res[i]);
////	       fprintf(f_res,"i %3d %s,%25llu result_vector %d init %s \n",i,s2,h_new_v[i],get_position_bit(h_new_v,i),s1);
////	      // printf("i %3d %s,%25llu res %d\n",i,s2,h_new_v[i],res[i]);
////	}
////	fclose(f_res);
//
//    return 0;
//
//}
