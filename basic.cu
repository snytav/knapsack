#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
//#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "basic.h"
#include "star_kernel.h"
//#include "cuPrintf.cu"


void MATCH(Table *tab, Slice *X, Slice *w, Slice *Z)
{

	unsigned int j,k;
	int s;
	k=tab->size;
//printf("\n MATCH %i \n", k);
	j=X->length;
 Slice *Y;
 Y=new Slice;
 Y->Init(j);

 Z->assign(X);
 for(int i=1;i<=k;i++)
 {
 tab->GetCol(Y,i);
  s= w->get(i);
  if (s==1) Z->AND(Y);
  else
  {
	  Y->NOT();
	  Z->AND(Y);
  }
 }
}

__device__ void match(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_w,unsigned long long int *d_z,int size)
{	 unsigned long long int *d_col=new unsigned long long int[gridDim.x];
	 int s;
//	 int index=blockIdx.x; //threadIdx.x + blockIdx.x*blockDim.x;
//	 cuPrintf("march_kernel \n");
	 _assign(d_z,d_x);

	 for(int i=0;i<size;i++)
	  {
//	  d_d=(unsigned long long *)(d_tab[i]);
//		 d_d=d_tab[i];
//	  d_col=d_d[blockIdx.x];
	  _assign(d_col,_col(d_tab,i));//d_tab[i];
	  s=get_position_bit(d_w,i+1);// s= d_w->get(i);
//	  printf("i =%d blockIdx.x= %d d_z[]= %ull d_col %ull \n",i,blockIdx.x,d_w[0],d_col);
 	   if (s==1)
	   {
	 	  //Z->AND(Y);
	 	 _and(d_z,d_col);
	    }
	   else
	   {
	 	  _not(d_col);
//	 	 printf("i =%d blockIdx.x= %d not d_col %ull \n",i,blockIdx.x,d_col);
	 	  //Y->NOT();
	 	//  Z->AND(Y);
	 	 _and(d_z,d_col);
	   }
  // printf("i =%d blockIdx.x= %d s=%d res= %ull \n",i,blockIdx.x,s,d_z[blockIdx.x]);
	  }
	 delete [] d_col;
	 }

 __global__ void match_kernel(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_w,unsigned long long int *d_z,int size)
 {
    match(d_tab,d_x,d_w,d_z,size);
	 }

 void MATCH_CUDA(Table *tab, Slice *X, Slice *w, Slice *Z)
{
	 unsigned long long int *d_x,*d_z, *d_w;
	 LongPointer *d_tab;
     int Nl=tab->size;
     int NN=X->NN;

     d_x=X->get_device_pointer();
	 d_w=w->get_device_pointer();
	 d_z=Z->get_device_pointer();
	 d_tab=tab->get_device_pointer();

	 match_kernel<<<NN,1>>>(d_tab,d_x,d_w,d_z,Nl);
 }
 __device__ void gel(LongPointer *d_tab, unsigned long long int * d_w,unsigned long long int *d_x,unsigned long long int *d_y,int size)
 {     unsigned long long int x,y,z,b,*d_col;
       int s;

	    x=0;//_clr(d_x);
	    y=0;//_clr(d_y);
        z=~0;//_set(z);
       for(int i=0;i<size;i++)
       {
    	 d_col=_col(d_tab,i);
    	 b=_assign(d_col);
         s=get_position_bit(d_w,i+1);
		 if (s==1)
		 {
//	(* In the slice Y we accumulate position of those i-th rows for which row(i,T)<w. *)
		   b=(~b) &z;//not(b); _and(b,z);
		   y|=b;     //_or(d_y,b);
//	printf("%i:l=%llu \n",blockIdx.x,b);
		 }
		 else{
			// _and(b,z);
			x|=b&z;// _or(d_x,b);
//	printf("%i:g=%llu \n",blockIdx.x,b);
//	(* In the slice X we accumulate position of those i-th rows for which row(i,T)>w. *)
		 }
		 //_not(b);
		 z&=~b;//_and(z,b);
//	(* Positions of the selected rows are deleting from the slice Z. *)
       }
       _assign(d_x,x);
       _assign(d_y,y);
 }
__global__ void gel_kernel(LongPointer *d_tab, unsigned long long int * d_w,unsigned long long int *d_x,unsigned long long int *d_y,int size)
{
    gel(d_tab, d_w, d_x,d_y,size);
}
 void GEL(Table *T, Slice *w, Slice *X,Slice *Y){
 unsigned long long int *d_x,*d_y, *d_w;
	 	 LongPointer *d_tab;
	      int NN=X->NN;
	      int size=T->size;
	      d_x=X->get_device_pointer();
	 	 d_w=w->get_device_pointer();
	 	 d_y=Y->get_device_pointer();
	 	 d_tab=T->get_device_pointer();
	 	gel_kernel<<<NN,1>>>(d_tab,d_w,d_x,d_y,size);
 }


void LESS(Table *T, Slice *X, Slice *v,Slice *Y)
{ unsigned long long int *d_x,*d_y, *d_v;
 LongPointer *d_tab;
 int NN=X->NN;
 int size=T->size;
 d_x=X->get_device_pointer();
 d_v=v->get_device_pointer();
 d_y=Y->get_device_pointer();
 d_tab=T->get_device_pointer();
  less_kernel<<<NN,1>>>(d_tab,d_x,d_v,d_y,size);
};

__global__ void less_kernel(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_v,unsigned long long int *d_y, int size)
{
   less(d_tab,d_x,d_v,d_y,size);
};

__device__ void less(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_v,unsigned long long int *d_y, int size)
{
	unsigned long long int b,y,y1, *d_col;
	int s;

   y1=_assign(d_x);
   y=0;
   for(int i=0;i<size;i++)
   {
      d_col=_col(d_tab,i);
	  b=_assign(d_col);
	  s=get_position_bit(d_v,i+1);
	  if (s==1)
	  {
	 	b=y1&(~b);
	 	y|=b;
	  }
	  y1&=~b;
    }
   _assign(d_y,y);
};

//procedure Great(T: Table; X: slice; v: word; Var Y: slice);
//Var
//  B, C: Slice;
//  n, i: integer;
//begin
//     B:=X;
//     Clr(Y);
//     For i:=1 to n do
//     begin
//      C:=col(i, T);
//      if v(i)=0 then
//      begin
 //        C:=C and B;
 //        Y:=C xor Y;
 //        C:=not C;
 //     end;
 //     B:=C and B;
 //    end;
//end;
void GREAT(Table *T, Slice *X, Slice *v,Slice *Y)
{ unsigned long long int *d_x,*d_y, *d_v;
 LongPointer *d_tab;
 int NN=X->NN;
 int size=T->size;
 d_x=X->get_device_pointer();
 d_v=v->get_device_pointer();
 d_y=Y->get_device_pointer();
 d_tab=T->get_device_pointer();
  less_kernel<<<NN,1>>>(d_tab,d_x,d_v,d_y, size);
};

__global__ void great_kernel(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_v,unsigned long long int *d_y, int size)
{
   great(d_tab,d_x,d_v,d_y, size);
};

__device__ void great(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_v,unsigned long long int *d_y,int size)
{
	unsigned long long int b,c,y, *d_col;
	int s;

   b=_assign(d_x);
   y=0;//_clr(d_y);
   for(int i=0;i<size;i++)
   {
      d_col=_col(d_tab,i);
	  c=_assign(d_col);
	  s=get_position_bit(d_v,i+1);
	  if (s==0)
	  {
	 	c&=b;//_and(c,b);
	 	y^=c;//_xor(d_y,c);
	 	c=~c;//_not(c);
	  }
	  b&=c;//_and(b,c);

    }
   _assign(d_y,y);
};
__global__ void min_part(LongPointer *d_tab,unsigned long long *d_y,int i,unsigned long long int *d_z)//<<<NN,1>>>
{
	unsigned long long int *d_col;
      d_col=_col(d_tab,i);
	  _assign(d_y,d_col);
	  _not(d_y);
	  _and(d_y,d_z);
};

__global__ void min_part1(LongPointer *d_tab,unsigned long long *d_y,int i,unsigned long long int *d_z,int *d_first_non_zero,int size)//<<<NN,1>>>
{
	unsigned long long int *d_col;
//	printf("MIN_part1     i=%i first=%i \n",i,*d_first_non_zero);
	 if ((i>0)&(d_first_non_zero[0]>0))
		 d_z[blockIdx.x]=d_y[blockIdx.x];//_assign(d_z,d_y);
	 if (i<size)
	 {
      d_col=_col(d_tab,i);
	 // _assign(d_y,d_col);
      d_y[blockIdx.x]=~d_col[blockIdx.x];
	 // _not(d_y);
	 // _and(d_y,d_z);
      d_y[blockIdx.x]&=d_z[blockIdx.x];
	 }
};

void MIN(Table *T, Slice *X, Slice*Z)
{unsigned long long int *d_z,*d_y;
int *d_first_non_zero;
LongPointer *d_t;
int size=T->size;
// hipError_t err = hipGetLastError();
// printf("before MIN %d , %s \n",err,hipGetErrorString(err));
 Slice *Y;
 Y=new Slice;
 Y->Init(LENGTH1);

 d_y=Y->get_device_pointer();
 d_z=Z->get_device_pointer();
 d_t=T->get_device_pointer();
 Z->assign(X);
// err = hipGetLastError();
//  printf("after init %d , %s \n",err,hipGetErrorString(err));
 hipMalloc(&d_first_non_zero,sizeof(int));

 int i=0;
// err = hipGetLastError();
//  printf("after hipMalloc %d , %s \n",err,hipGetErrorString(err));


 // Z->print("MIN:Z",0);
 
 //(LongPointer * d_tab, unsigned long long* d_y, int i, unsigned long long int* d_z, int* d_first_non_zero, int size)//<<<NN,1>>>

 min_part1<<<NN1,1>>>(d_t,d_y,i,d_z,d_first_non_zero,size);
// Y->print("MIN:Y",0);
// Z->print("MIN:Z",0);
 for(i=1;i<=size;i++)
{
	 first(d_y,NN1,d_first_non_zero,NN1);

     //min_part1<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero);
	 min_part1<<<NN1,1>>>(d_t,d_y,i,d_z,d_first_non_zero,size);
//err = hipGetLastError();
//printf("%i MIN %d , %s \n",i,err,hipGetErrorString(err));
//	 Y->print("MIN:Y",0);
 //    Z->print("MIN:Z",0);
 }
};

void MIN(Table *T, Slice *X, Slice*Z, Slice *Y)
{unsigned long long int *d_z,*d_y;
  int *d_first_non_zero;
  int size=T->size;
LongPointer *d_t;
 d_y=Y->get_device_pointer();
 d_z=Z->get_device_pointer();
 d_t=T->get_device_pointer();
 Z->assign(X);
 hipMalloc(&d_first_non_zero,sizeof(int));
 int i=0;
 min_part1<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero,size);
 for(i=1;i<size;i++)
{
	 first(d_y,NN1,d_first_non_zero,NN1);
     min_part1<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero,size);
 }
};

void MIN_1(Table *T, Slice *X, Slice*Z, Slice *Y)
{unsigned long long int *d_z,*d_y;
  int *d_first_non_zero;
  int size=T->size;
LongPointer *d_t;
 d_y=Y->get_device_pointer();
 d_z=Z->get_device_pointer();
 d_t=T->get_device_pointer();
 Z->assign(X);
 hipMalloc(&d_first_non_zero,sizeof(int));
 int i=0;
 min_part1<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero);
 for(i=1;i<size;i++)
{
	 some(d_y,NN1,d_first_non_zero,NN1);
     min_part1<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero);
 }
};
__global__ void max_part(LongPointer *d_tab,unsigned long long *d_y,int i,unsigned long long int *d_z)//<<<NN,1>>>
{
	unsigned long long int *d_col;
//	 if (k>0) _assign(d_z,d_y);
      d_col=_col(d_tab,i);
	  _assign(d_y,d_col);
	  _and(d_y,d_z);
};
__global__ void max_part(LongPointer *d_tab,unsigned long long *d_y,int i,unsigned long long int *d_z,int *d_first_non_zero)//<<<NN,1>>>
{
	unsigned long long int *d_col;
//	 if (k>0) _assign(d_z,d_y);
	 if ((i>0)&(d_first_non_zero>0)) _assign(d_z,d_y);
      d_col=_col(d_tab,i);
	  _assign(d_y,d_col);
	  _and(d_y,d_z);
};
void MAX(Table *T, Slice *X, Slice*Z)
{unsigned long long int *d_z,*d_y;
int *d_first_non_zero;
LongPointer *d_t;
int NN=X->NN;
int size=T->size;
 Slice *Y;
 Y=new Slice;
 Y->Init(LENGTH1);

 d_y=Y->get_device_pointer();
 d_z=Z->get_device_pointer();
 d_t=T->get_device_pointer();
 Z->assign(X);
 hipMalloc(&d_first_non_zero,sizeof(int));

 hipError_t err = hipGetLastError();
 printf("before MAX %d , %s \n",err,hipGetErrorString(err));

 int i=0;
  max_part<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero);
  err = hipGetLastError();
  printf("first MAX %d , %s \n",err,hipGetErrorString(err));
  for(i=1;i<size;i++)
 {
 	 first(d_y,NN1,d_first_non_zero,NN1);
 	 puts("first");
      max_part<<<blocks1,threads1>>>(d_t,d_y,i,d_z,d_first_non_zero);
      printf("%d ",i);
  }
  err = hipGetLastError();
  printf("after MAX %d , %s \n",err,hipGetErrorString(err));
};

//__device__ void min(LongPointer *d_tab,unsigned long long int *d_x,unsigned long long int *d_z)// нет из-за синхронизации перед some()

void SETMIN(Table *T, Table *F, Slice *X, Slice *Z)
{unsigned long long int *d_x,*d_z;
LongPointer *d_t, *d_f;
int NN=X->NN;
int size=T->size;
d_x=X->get_device_pointer();
d_z=Z->get_device_pointer();
d_t=T->get_device_pointer();
d_f=F->get_device_pointer();
 setmin_kernel<<<NN,1>>>(d_t,d_f,d_x,d_z, size);

};
__global__ void setmin_kernel(LongPointer *d_t, LongPointer *d_f,unsigned long long int *d_x,unsigned long long int *d_z, int size)
{
   setmin(d_t,d_f,d_x,d_z,size);
};
__device__ void setmin(LongPointer *d_t, LongPointer *d_f,unsigned long long int *d_x,unsigned long long int *d_z, int size )
{    unsigned long long int *col_t,*col_f,m,b,y,x,z;
     x=d_x[blockIdx.x];
     z=0;
	 for (int i=0;i<size;i++)
	 {
		 col_t=_col(d_t,i);
		 col_f=_col(d_f,i);
		 b=col_t[blockIdx.x];
		 y=col_f[blockIdx.x];
		 m=b^y;
		 m&=x;
		 b=y&(~b);
		 b&=x;
		 z|=b;
		 x&=~m;
	 }
	 d_x[blockIdx.x]=x;
	 d_z[blockIdx.x]=z;
};

void SETMAX(Table *T, Table *F, Slice *X, Slice *Z)
{unsigned long long int *d_x,*d_z;
LongPointer *d_t, *d_f;
int NN=X->NN;
int size=T->size;
d_x=X->get_device_pointer();
d_z=Z->get_device_pointer();
d_t=T->get_device_pointer();
d_f=F->get_device_pointer();
 setmax_kernel<<<NN,1>>>(d_t,d_f,d_x,d_z,size);

};
__global__ void setmax_kernel(LongPointer *d_t, LongPointer *d_f,unsigned long long int *d_x,unsigned long long int *d_z,int size)
{
   setmax(d_t,d_f,d_x,d_z,size);
};
__device__ void setmax(LongPointer *d_t, LongPointer *d_f,unsigned long long int *d_x,unsigned long long int *d_z,int size )
{    unsigned long long int *col_t,*col_f,m,b,y,x,z;
     x=_assign(d_x);
     z=0;
	 for (int i=0;i<size;i++)
	 { col_t=_col(d_t,i);
	 col_f=_col(d_f,i);
	 b=_assign(col_t);
	 y=_assign(col_f);
	 m=b^y;
	 m&=x;
	 b=(~y)&(b);
	 b&=x;
	 z|=b;
	 x&=~m;
 }
 _assign(d_x,x);
 _assign(d_z,z);
};

void HIT(Table *T, Table *F, Slice *X, Slice *Z)
{unsigned long long int *d_x,*d_z;
LongPointer *d_t, *d_f;
int NN=X->NN;
int size=T->size;
d_x=X->get_device_pointer();
d_z=Z->get_device_pointer();
d_t=T->get_device_pointer();
d_f=F->get_device_pointer();
 hit_kernel<<<NN,1>>>(d_t,d_f,d_x,d_z,size);
};
__global__ void hit_kernel(LongPointer *d_t, LongPointer *d_f,unsigned long long int *d_x,unsigned long long int *d_z,int size)
{
   hit(d_t,d_f,d_x,d_z,size);
};
__device__ void hit(LongPointer *d_t, LongPointer *d_f,unsigned long long int *d_x,unsigned long long int *d_z, int size )
{    unsigned long long int *col_t,*col_f,b,y,z;
     z=_assign(d_x);
	 for (int i=0;i<size;i++)
	 {
		 col_t=_col(d_t,i);
		 col_f=_col(d_f,i);
		 b=_assign(col_t);
		 y=_assign(col_f);
		 y^=b;
		 z&=~y;
	 }
	 _assign(d_z,z);
};

void TMERGE(Table *T,  Slice *X, Table *F,int k)
{
	unsigned long long int *d_x;
	LongPointer *d_t, *d_f;
	int NN=X->NN;
	int size=T->size;
	d_x=X->get_device_pointer();
	d_t=T->get_device_pointer();
	d_f=F->get_device_pointer();
	 tmerge_kernel<<<NN,k>>>(d_t,d_x,d_f,size);

};
__global__ void tmerge_kernel(LongPointer *d_t,unsigned long long int *d_x, LongPointer *d_f,int size)//<<<NN,1...M>>>
{   if (gridDim.y==1)
	{
	    tmerge(d_t,d_x,d_f,size);
//	    printf("1D parallization");
	}
    else
    {
    	tmerge_par(d_t,d_x,d_f);
//    	printf("2D parallization\n");
    }
//	printf("<<(),(%i,%i)>>",blockDim.x,blockDim.y);
};
__device__ void tmerge(LongPointer *d_t,unsigned long long int *d_x, LongPointer *d_f,int size)
{   unsigned long long int *col_t,*col_f,y,z,a,x;
     x=_assign(d_x);
     a=~x;
	 for (int i=0;i<size;i++)
//     int i=blockIdx.y;
		 {
		 col_t=_col(d_t,i);
		 col_f=_col(d_f,i);
		 y=_assign(col_t)&x;
		 z=_assign(col_f)&a;
		 z|=y;
		 _assign(col_f,z);
		 }
//		 printf("b_x=%i,b_y=%i,thr_x=%i,thr_y=%i ,i=%i\n",blockIdx.x,blockIdx.y, threadIdx.x,threadIdx.y,i);
};

__device__ void tmerge_par(LongPointer *d_t,unsigned long long int *d_x, LongPointer *d_f)
{   unsigned long long int *col_t,*col_f,y,z,a,x;
     x=_assign(d_x);
     a=~x;
//	 for (int i=0;i<M;i++)
     int i=blockIdx.y;
		 {
		 col_t=_col(d_t,i);
		 col_f=_col(d_f,i);
		 y=_assign(col_t)&x;
		 z=_assign(col_f)&a;
		 z|=y;
		 _assign(col_f,z);
		 }
//		 printf("b_x=%i,b_y=%i,thr_x=%i,thr_y=%i ,i=%i\n",blockIdx.x,blockIdx.y, threadIdx.x,threadIdx.y,i);
};

void WMERGE(Slice *v,  Slice *X, Table *F,int k)
{
	unsigned long long int *d_x,*d_v;
	LongPointer *d_f;
	int NN=X->NN;
	d_v=v->get_device_pointer();
	d_x=X->get_device_pointer();
	d_f=F->get_device_pointer();
    int MM=F->size;
	 wmerge_kernel<<<NN,k>>>(d_v,d_x,d_f,MM);
};
__global__ void wmerge_kernel(unsigned long long int *d_v,unsigned long long int *d_x, LongPointer *d_f,int MM)//<<<NN,1...M>>>
{
	wmerge(d_v,d_x,d_f,size,MM);
};
__device__ void wmerge(unsigned long long int *d_v,unsigned long long int *d_x, LongPointer *d_f, int MM)
{ unsigned long long int *col_f,y,z,x;
  int pos;
   x=_assign(d_x);
   y=~x;
   for (int i=threadIdx.x;i<MM;i=i+blockDim.x)
  		 {
          col_f=_col(d_f,i);
          z=_assign(col_f)&y;
          pos=_get_bit(d_v,i+1);
          if (pos==1)z|=x;
 	   	  _assign(col_f,z);
  		 }
};

void WCOPY(Slice *v,  Slice *X, Table *F,int k)
{
	unsigned long long int *d_x,*d_v;
	LongPointer *d_f;
	int NN=X->NN;
    int size=F->size;
	d_v=v->get_device_pointer();
	d_x=X->get_device_pointer();
	d_f=F->get_device_pointer();
	 wcopy_kernel<<<NN,k>>>(d_v,d_x,d_f,size);
};
__global__ void wcopy_kernel(unsigned long long int *d_v,unsigned long long int *d_x, LongPointer *d_f,int k)//<<<NN,1...M>>>
{
	wcopy(d_v,d_x,d_f,k);
};
__device__ void wcopy(unsigned long long int *d_v,unsigned long long int *d_x, LongPointer *d_f,int k)
{ unsigned long long int *col_f,y;
     int pos;
     y=0;
     for (int i=threadIdx.x;i<k;i=i+blockDim.x)
		 {
    	  col_f=_col(d_f,i);
    	  pos=_get_bit(d_v,i+1);
    	  if (pos==1)
    	  {   col_f[blockIdx.x]=d_x[blockIdx.x];
    		  //_assign(col_f,d_x);
    	  }
    	  else
    	  { // _assign(col_f,y);
    	    col_f[blockIdx.x]=y;
    	  }
		 }
};

void TCOPY(Table *T, Table *F,int k)
{
	LongPointer *d_t, *d_f;
	int NN=(T->length-1)/SIZE_OF_LONG_INT+1;
    int r=T->size;
 //   printf("NN=%i, r=%i", NN, r);
	d_t=T->get_device_pointer();
	d_f=F->get_device_pointer();
	if (k>r)k=r;
	 tcopy_kernel<<<NN,k>>>(d_t,d_f,r);

};
__global__ void tcopy_kernel(LongPointer *d_t, LongPointer *d_f,int r)
{
	tcopy(d_t,d_f,r);
};
__device__ void tcopy(LongPointer *d_t, LongPointer *d_f,int r)
{ unsigned long long int *col_f, *col_t,y;
     for (int i=threadIdx.x;i<r;i=i+blockDim.x)
		 {
    	    col_t=_col(d_t,i);
	        col_f=_col(d_f,i);
//	        y=_assign(col_t);
//	        _assign(col_f,y);
	        _assign(col_f,col_t);
//	        printf("copy %i %llu=%llu \n",i,d_t[i],d_f[i]);
		 }
};

void TCOPY1(Table *T,int j, int h, Table *F,int k)
{
	LongPointer *d_t, *d_f;
	int NN=(T->length-1)/SIZE_OF_LONG_INT+1;

//	 hipError_t err = hipGetLastError();
//		 printf("error befor TCOPY1 %d \n",err);

	d_t=T->get_device_pointer();
	d_f=F->get_device_pointer();
	if ((k==1) || (k>h))k=h;
//	printf("=============================TCOPY1=%i<%i,%i>======================\n",j,NN,k);
	 tcopy1_kernel<<<NN,k>>>(d_t,j,h,d_f);
//	  err = hipGetLastError();
//	 printf("error after TCOPY1 %d \n",err);
};
__global__ void tcopy1_kernel(LongPointer *d_t, int j,int h, LongPointer *d_f)//<<<NN,k>>> k=1,...,h
{
//	printf("TCOPY1 %i %i\n", j, h);
	tcopy1(d_t,j,h,d_f);
};
__device__ void tcopy1(LongPointer *d_t, int j, int h, LongPointer *d_f)
{    unsigned long long int *col_f, *col_t,y;
//printf("tcopy1===============i=%i[%i]   \n", threadIdx.x,blockIdx.x);
     int k=(j-1)*h;
     for (int i=threadIdx.x;i<h;i=i+blockDim.x)
	 {
	    col_t=_col(d_t,k+i);
        col_f=_col(d_f,i);

//        y=col_t[blockIdx.x];
//       col_f[blockIdx.x]=y;
        col_f[blockIdx.x]=col_t[blockIdx.x];

//   if((threadIdx.x==0)||(threadIdx.x==(blockDim.x-1))) printf("strip %i: <%i,%i> i=%i k=%i \n",j,threadIdx.x,blockIdx.x, i,k+i);
	 }
};

void TCOPY2(Table *T,int j, int h, Table *F,int k)
{
	LongPointer *d_t, *d_f;
	int NN=(T->length-1)/SIZE_OF_LONG_INT+1;
	d_t=T->get_device_pointer();
	d_f=F->get_device_pointer();
	if ((k==1) || (k>h))k=h;
//printf(" TCOPY2 NN=%i \t k=%i \t",NN,k);
	 tcopy2_kernel<<<NN,k>>>(d_t,j,h,d_f);
//puts("TCOPY2 done");
};
__global__ void tcopy2_kernel(LongPointer *d_t, int j,int h, LongPointer *d_f)//<<<NN,k>>> k=1,...,h
{
//printf("tcopy2_kernel \n");
	tcopy2(d_t,j,h,d_f);
};
__device__ void tcopy2(LongPointer *d_t, int j, int h, LongPointer *d_f)
{    unsigned long long int *col_f, *col_t,y;
     int k=(j-1)*h+threadIdx.x;
     for (int i=threadIdx.x;i<h;i=i+blockDim.x)
	 {
	    col_t=_col(d_t,i);
        col_f=_col(d_f,k);
 //printf("T \t %i -> F \t %i \n",i,k);
  //     y=_assign(col_t);
  //     _assign(col_f,y);
       _assign(col_f,col_t);
       k=k+blockDim.x;
	 }
	;
};

void ADDV(Table *T, Table *R, Slice *X, Table *S)
{
	LongPointer *d_t, *d_r,*d_s;
	unsigned long long int *d_x, *d_b;
	int h=T->size;
	int NN=X->NN;
	Slice *B;
	B=new Slice;
    B->Init(LENGTH1);

	d_t=T->get_device_pointer();
	d_r=R->get_device_pointer();
	d_s=S->get_device_pointer();
	d_x=X->get_device_pointer();
	d_b=B->get_device_pointer();

	 addv_kernel<<<NN,1>>>(d_t,d_r,h,d_x,d_s,d_b);

//	 if (Y->SOME()) S->With(Y);
};
__global__ void addv_kernel(LongPointer *d_t,LongPointer *d_r,int h,unsigned long long int *d_x,LongPointer *d_s,unsigned long long int *d_b)
{
	addv(d_t,d_r,h,d_x,d_s,d_b);
}

__device__ void addv(LongPointer *d_t,LongPointer *d_r,int h,unsigned long long int *d_x,LongPointer *d_s,unsigned long long int *d_b)
{ unsigned long long int x,b,y,z,m,*col_t, *col_r,*col_s;
  m=0;
  x=_assign(d_x);
  for (int i=h-1;i>=0;i--)
  {
	  col_t=_col(d_t,i);
      y=_assign(col_t)&x;
      col_r=_col(d_r,i);
      z=_assign(col_r)&x;
      b=y&z;
      z^=y;
      y=z^m;
      col_s=_col(d_s,i); _assign(col_s,y);
      y=z & m;
      b|=y;
      m=b;
  }
  _assign(d_b,b);
};


void ADDC(Table *T, Slice *w, Slice *X, Table *S)
{
	LongPointer *d_t,*d_s;
	unsigned long long int *d_x, *d_b,*d_w;
	int h=T->size;
	int NN=X->NN;
	Slice *B;
	B=new Slice;
    B->Init(LENGTH1);

	d_t=T->get_device_pointer();
	d_w=w->get_device_pointer();
	d_s=S->get_device_pointer();
	d_x=X->get_device_pointer();
	d_b=B->get_device_pointer();

	 addc_kernel<<<NN,1>>>(d_t,d_w,h,d_x,d_s,d_b);

	 if (B->SOME()) puts("AddC is incorrect.");// S->With(Y);
};
__global__ void addc_kernel(LongPointer *d_t,unsigned long long int *d_w,int h,unsigned long long int *d_x,LongPointer *d_s,unsigned long long int *d_b)
{
	addc(d_t,d_w,h,d_x,d_s,d_b);
};
__device__ void addc(LongPointer *d_t,unsigned long long int *d_w,int h,unsigned long long int *d_x,LongPointer *d_s,unsigned long long int *d_b)
{ unsigned long long int y,m,*col_t,*col_s,b,x;
 int pos;
  b=0;
 // x=_assign(d_x);
  x=d_x[blockIdx.x];
  for (int i=h-1;i>=0;i--)
  {
	  col_t=_col(d_t,i);
//      y= _assign(col_t);
	  y=col_t[blockIdx.x];
      m=b^y;
      pos=_get_bit(d_w,i+1);
      if (pos==0)
      {	  b&=y;
//      printf("x:%llu \t %llu 0",x,y, m);
      }
      else
      {
    	  m=~m;
    	  b|=y;
//    	printf("x:%llu \t %llu 1",x,y, m);
      }
      col_s=_col(d_s,i);
      m&=x;
      //_assign(col_s,m);
      col_s[blockIdx.x]=m;
//    printf("%llu\n" m);
      b&=x;

   }
 // if (b!=0) printf("addc is incorrect\n");
  _assign(d_b,b);
};

int ADDC1( Slice *X, Slice *w, Table *S)
{	LongPointer *d_s;
unsigned long long int *d_x, *d_b,*d_w;
int h=S->size;
int k,i,j;
int NN=X->NN;
Slice *B;
B=new Slice;
B->Init(LENGTH1);

d_w=w->get_device_pointer();
d_s=S->get_device_pointer();
d_x=X->get_device_pointer();
d_b=B->get_device_pointer();

 addc1_kernel<<<NN,1>>>(d_x,d_w,h,d_s,d_b);

	 if (B->SOME())
	 {	// puts("AddC1 is incorrect.");//S->With(Y) and error;
	 //    B->print("ADDC1_error",0);
//		k=B->FND();
//		 printf("B(%i)=1 \n",k);
	     return k;
	 }
	 else return 0;
};
__global__ void addc1_kernel(unsigned long long int *d_x,unsigned long long int *d_w,int h,LongPointer *d_s,unsigned long long int *d_b)
{
	addc1(d_x,d_w,h,d_s,d_b);
};
__device__ void addc1(unsigned long long int *d_x, unsigned long long int *d_w,int h,LongPointer *d_s,unsigned long long int *d_b)
{ unsigned long long int y,y1,z,m,x,nx,*col_s,b;
 int pos;
   b=0;
   x=d_x[blockIdx.x];
   nx=~x;
   m=0;
   for (int i=h;i>0;i--)
   {
//printf("addc1 i=%i\n",i);
 	  col_s=_col(d_s,i-1);
       y= col_s[blockIdx.x];
       m=b^y;
       pos=_get_bit(d_w,i);
       if (pos==0) b&=y;
       else
       {
     	  m=~m;
     	  b|=y;
       }
       m&=x;
       m|=(y&nx);//
       col_s[blockIdx.x]=m;
       b&=x;
/*/
	   col_s=_col(d_s,i-1);
	   y= col_s[blockIdx.x];
	   y1=y&nx;//неизменяемая часть;
	   y&=x;
	   m=b^y;
	   pos=_get_bit(d_w,i);
	   if (pos==0) z=0;
	   else z=x;
	   b=y&z;
	   z^=y;
	   col_s[blockIdx.x]=(z^m)|y1;
	   b|=y;
	   m=b;
	  // */
    }
//   if (b==0) puts("add1c is incorrect");
   d_b[blockIdx.x]=b&x;
   };

void SUBTV(Table *T, Table *R, Slice *X,Table *S)
{
	LongPointer *d_t, *d_r,*d_s;
		unsigned long long int *d_x, *d_b;
		int h=T->size;
		int NN=X->NN;

		Slice *B;
		B=new Slice;
	    B->Init(LENGTH1);

		d_t=T->get_device_pointer();
		d_r=R->get_device_pointer();
		d_s=S->get_device_pointer();
		d_x=X->get_device_pointer();
		d_b=B->get_device_pointer();
//         printf("length=%i, NN=%i, blocks=%i, threads=%i \n",LENGTH1, NN1, blocks1,threads1);
		 subtv_kernel<<<blocks1,threads1>>>(d_t,d_r,h,d_x,d_s,d_b);
 //        B->print("subtv_error.dat",0);
//		 if (B->SOME()) error;
};
__global__ void subtv_kernel(LongPointer *d_t, LongPointer *d_r,int k, unsigned long long int *d_x, LongPointer *d_s,unsigned long long int *d_m)
{
	subtv(d_t,d_r,k,d_x,d_s,d_m);
};
__device__ void subtv(LongPointer *d_t, LongPointer *d_r,int k, unsigned long long int *d_x, LongPointer *d_s,unsigned long long int *d_m)
{ unsigned long long int *col_t, *col_r, *col_s;
  unsigned long long int a,b,p,y,z,m,x;

  m=0;
  x=_assign(d_x);
//  int ind= index;
//  printf("_%i_",ind);
  for (int i=k-1;i>-1;i--)
  {
	  col_t=_col(d_t,i);
	  col_r=_col(d_r,i);
      col_s=_col(d_s,i);
	  z=_assign(col_t);
	  z&=x;
	  y=_assign(col_r);
	  y&=x;
	  p=y^m;
	  a=(p^z)&x;
	  _assign(col_s,a);
	  b=p&(~z);
	  b|=y&m;
	  m=b;
// if((index>=NN))
//	 printf("i=%i ,ind=%i,m=%llu, a==%llu \n",i, ind,m,a);
  }
  _assign(d_m,m);
};

void SUBTC(Table *T, Slice *X, Slice *w, Table *S)
{	LongPointer *d_t,*d_s;
unsigned long long int *d_x, *d_b, *d_w;
int h=T->size;
int NN=X->NN;
Slice *B;
B=new Slice;
B->Init(LENGTH1);

d_t=T->get_device_pointer();
d_w=w->get_device_pointer();
d_s=S->get_device_pointer();
d_x=X->get_device_pointer();
d_b=B->get_device_pointer();

 subtc_kernel<<<NN,1>>>(d_t,d_x,d_w,d_s,h,d_b);

//	 if (B->SOME()) error;
};
__global__ void subtc_kernel(LongPointer *d_t, unsigned long long int *d_x, unsigned long long int *d_w, LongPointer *d_s,int k,unsigned long long int *d_m)
{
	subtc(d_t,d_x,d_w,d_s,k,d_m);
};
__device__ void subtc(LongPointer *d_t, unsigned long long int *d_x, unsigned long long int *d_w, LongPointer *d_s,int k,unsigned long long int *d_m)
{   unsigned long long int *col_t, *col_s;
    unsigned long long int a,b,t,m,s,x;
    int pos;
    x=_assign(d_x);
    m=0;
    for (int i=k-1;i>-1;i--)
    { col_t=_col(d_t,i);
	  col_s=_col(d_s,i);
	  t=_assign(col_t);
	  a=m^t;
	  pos=_get_bit(d_w,i+1);
	       if (pos==0)
	       {
	    	b=m&(~t);
	    	s=a;
	       }
	       else
	       {
	       s=~a;
	       b=s|m;
	       }
	       s&=x;
	       _assign(col_s,s);
	       b&=x;
	       m=b;

    }
    _assign(d_m,m);
};

void SUBTC1(Table *T, Slice *X, Slice *w, Table *S)
{	LongPointer *d_t,*d_s;
unsigned long long int *d_x, *d_b, *d_w;
int h=T->size;
int NN=X->NN;
Slice *B;
B=new Slice;
B->Init(LENGTH1);

d_t=T->get_device_pointer();
d_w=w->get_device_pointer();
d_s=S->get_device_pointer();
d_x=X->get_device_pointer();
d_b=B->get_device_pointer();

 subtc1_kernel<<<NN,1>>>(d_t,d_x,d_w,d_s,h,d_b);

//	 if (B->SOME()) error;
};
__global__ void subtc1_kernel(LongPointer *d_t, unsigned long long int *d_x, unsigned long long int *d_w, LongPointer *d_s,int k,unsigned long long int *d_m)
{
	subtc1(d_t,d_x,d_w,d_s,k,d_m);
};
__device__ void subtc1(LongPointer *d_t, unsigned long long int *d_x, unsigned long long int *d_w, LongPointer *d_s,int k,unsigned long long int *d_m)
{      unsigned long long int *col_t, *col_s;
unsigned long long int a,b,t,m,s,x;
int pos;
x=_assign(d_x);
m=0;
for (int i=k-1;i>-1;i--)
{ col_t=_col(d_t,i);
  col_s=_col(d_s,i);
  t=_assign(col_t);
  a=m^t;
  pos=_get_bit(d_w,i+1);
       if (pos==0)
       {
    	b=m&(~t);
    	s=a;
       }
       else
       {
       s=~a;
       b=s|m;
       }
       s&=x;
       s|=(~x)&t;
       _assign(col_s,s);
       b&=x;
       m=b;
}
_assign(d_m,m);
};

void CLEAR(Table *T)
{};

__device__ void clear(LongPointer *d_tab, int h)
{   unsigned long long int *col_tab,y=0;

    for (int i=threadIdx.x;i<h;i=i+blockDim.x)
    {
      col_tab=_col(d_tab,i);
     _assign(col_tab,y);
    }
};


void WTRANS(Slice *w, int h, Table *R)
{
  wtrans_kernel<<<NN1,SIZE_OF_LONG_INT>>>(w->get_device_pointer(),h,R->length,R->get_device_pointer());
};

__global__ void wtrans_kernel(unsigned long long int *d_w, int h, int length, LongPointer *d_r)
{
  wtrans(d_w,h,length,d_r);
};

__device__ void wtrans(unsigned long long int *d_w, int h, int length, LongPointer *d_r)
{
	unsigned int bid=blockIdx.x;
	unsigned int tid=threadIdx.x;
	unsigned int M1=(length>SIZE_OF_LONG_INT*(bid+1))?SIZE_OF_LONG_INT:length%SIZE_OF_LONG_INT;
	__shared__ unsigned long long  int tmp[SIZE_OF_LONG_INT];
	unsigned int i,r_i;//, M1=SIZE_OF_LONG_INT;
	unsigned long long int *d_n;
	r_i=h*tid+SIZE_OF_LONG_INT*bid;
	for (i=0; i<h;i++)
	{
		tmp[tid]=0;
		d_n=d_r[i];
		if(get_position_bit(d_w,r_i)==1)
		{
//если в позиции 1
		 tmp[tid]=1<<tid;

		}
		if(tid<M1) printf("col %i,%i,%i, <%i, %i>, pos %i =%i \n",i,length,M1,bid,tid,r_i,get_position_bit(d_w,r_i));
		d_n[bid]=  get_array(tmp,0,M1)
		  	    	    	|  get_array(tmp,1,M1)
		  	    	    	|  get_array(tmp,2,M1)
		  	    	    	|  get_array(tmp,3,M1)
		  	    	        |  get_array(tmp,4,M1)
		  	    	        |  get_array(tmp,5,M1)
		  	    	        |  get_array(tmp,6,M1)
		  			        |  get_array(tmp,7,M1)
		  	         		|  get_array(tmp,8,M1)
		  			        |  get_array(tmp,9,M1)
		  	                |  get_array(tmp,10,M1)
		  	                |  get_array(tmp,11,M1)
		  	                |  get_array(tmp,12,M1)
			                |  get_array(tmp,13,M1)
			                |  get_array(tmp,14,M1)
			                |  get_array(tmp,15,M1)
		                    |  get_array(tmp,16,M1)
		                    |  get_array(tmp,17,M1)
		                    |  get_array(tmp,18,M1)
		          			|  get_array(tmp,19,M1)
					        |  get_array(tmp,20,M1)
				            |  get_array(tmp,21,M1)
		                    |  get_array(tmp,22,M1)
		                    |  get_array(tmp,23,M1)
		                    |  get_array(tmp,24,M1)
		   	                |  get_array(tmp,25,M1)
			                |  get_array(tmp,26,M1)
			                |  get_array(tmp,27,M1)
		                    |  get_array(tmp,28,M1)
		                    |  get_array(tmp,29,M1)
		                    |  get_array(tmp,30,M1)
			                |  get_array(tmp,31,M1)
				            |  get_array(tmp,32,M1)
				            |  get_array(tmp,33,M1)
		                    |  get_array(tmp,34,M1)
		                    |  get_array(tmp,35,M1)
		                    |  get_array(tmp,36,M1)
		   	                |  get_array(tmp,37,M1)
			                |  get_array(tmp,38,M1)
			                |  get_array(tmp,39,M1)
		                    |  get_array(tmp,40,M1)
		                    |  get_array(tmp,41,M1)
		                    |  get_array(tmp,42,M1)
					        |  get_array(tmp,43,M1)
			            	|  get_array(tmp,44,M1)
			             	|  get_array(tmp,45,M1)
		                    |  get_array(tmp,46,M1)
		                    |  get_array(tmp,47,M1)
		                    |  get_array(tmp,48,M1)
		    	            |  get_array(tmp,49,M1)
			                |  get_array(tmp,50,M1)
			                |  get_array(tmp,51,M1)
		                    |  get_array(tmp,52,M1)
		                    |  get_array(tmp,53,M1)
		                    |  get_array(tmp,54,M1)
				            |  get_array(tmp,55,M1)
			         	    |  get_array(tmp,56,M1)
				            |  get_array(tmp,57,M1)
		                    |  get_array(tmp,58,M1)
		                    |  get_array(tmp,59,M1)
		                    |  get_array(tmp,60,M1)
		                    |  get_array(tmp,61,M1)
		                    |  get_array(tmp,62,M1)
		                    |  get_array(tmp,63,M1);

		r_i++;
	}

};

// d_x - строка, по которой вставляют
// k - номер столбца, который вставляют
__global__ void x_w_table_or_kernel(LongPointer *d_tab, unsigned long long int *d_w, unsigned long long int *d_x)
{	 unsigned long long int *d_v;
	        int s;

		//    d_w=d_tab[k-1];//tab->GetCol(w,k);
		    s=_get_bit(d_x,threadIdx.x+1);
	    	//s=get_position_bit(d_x,threadIdx.x+1);
	    	if(s==1)
	    	{
	    		d_v=d_tab[threadIdx.x];//tab->GetCol(w,k);
	    		_or(d_v,d_w);

	    	}
}

__global__ void x_w_table_and_kernel(LongPointer *d_tab, unsigned long long int *d_w, unsigned long long int *d_x)
{	 unsigned long long int *d_v;
     int s;
//    d_w=d_tab[k-1];//tab->GetCol(w,k);
    s=_get_bit(d_x,threadIdx.x+1);
//s=get_position_bit(d_x,threadIdx.x+1);
    if(s==1)
	{
		d_v=d_tab[threadIdx.x];//tab->GetCol(w,k);
	   _and(d_v,d_w);
   	}
}


void TCOPY3(Table *T,int j, int h, Table *F,int k)
{
	LongPointer *d_t, *d_f;
	int NN=(F->length-1)/SIZE_OF_LONG_INT+1;
	d_t=T->get_device_pointer();
	d_f=F->get_device_pointer();
	int h1=F->size;
	if ((k==1) || (k>h1))k=h1;
//printf(" TCOPY2 NN=%i \t k=%i \t",NN,k);
	 tcopy3_kernel<<<NN,k>>>(d_t,j,h,d_f);
//puts("TCOPY2 done");
}

__global__ void tcopy3_kernel(LongPointer *d_t, int j,int h, LongPointer *d_f)//!<<<NN,k>>> k=1,...,f.size
{
	tcopy3(d_t, j,h,d_f);
}

//<<<NN,h>>>
__device__ void tcopy3(LongPointer *d_t, int j, int h, LongPointer *d_f)
{
	unsigned long long int *d_v,*d_v_in;
// проверка, если не все столбцы обрабатываются своей нитью
	d_v=d_f[threadIdx.x];
	d_v_in=d_t[threadIdx.x];
	trim_(d_v,d_v_in,j,h);
}


void TCOPY4(Table *T,int j, int h, Table *F,int k)// Копирует T как горизонтальную полосу в F
{
	LongPointer *d_t, *d_f;
	int NN=(F->length-1)/SIZE_OF_LONG_INT+1;
	d_t=T->get_device_pointer();
	d_f=F->get_device_pointer();
	int h1=F->size;
	if ((k==1) || (k>h1))k=h1;
//printf(" TCOPY2 NN=%i \t k=%i \t",NN,k);
	 tcopy4_kernel<<<NN,k>>>(d_t,j,h,d_f);
//puts("TCOPY4 done");
}
__global__ void tcopy4_kernel(LongPointer *d_t, int j,int h, LongPointer *d_f)//!<<<NN,k>>> k=1,...,h
{
	tcopy4(d_t, j,h,d_f);
}
__device__ void tcopy4(LongPointer *d_t, int j, int h, LongPointer *d_f)
{
	unsigned long long int *d_v,*d_v_in, head, teal;
// проверка, если не все столбцы обрабатываются своей нитью
	d_v=d_f[threadIdx.x];//маленький
	d_v_in=d_t[threadIdx.x];// большой
	int num_el=j>>6;//номер первого элемента в результирующем слайсе
	int num_el1=h>>6; // номер последнего элемента в маленьком
	int num_bit_first= j % SIZE_OF_LONG_INT -1; // номер бита в элементе, в который копируется первый маленького слайса
	int num_bit_last = (j+h-1) % SIZE_OF_LONG_INT;// число элементов
	   char prb[65];
	//   printf("num_els %i and %i (%i) bits from %i to %i \n",num_el,num_el1,num_el2, num_bit_first, num_bit_last);
//	   unsigned long long int teal,head;
	   if (blockIdx.x >num_el)//?????????
	   {      head =d_v_in[blockIdx.x]<<(num_bit_first);
	   	   teal =d_v_in[blockIdx.x-1]>>(SIZE_OF_LONG_INT-num_bit_first);
	   	   d_v[blockIdx.x+num_el]=head | teal;
//	   	   printf("num_els %i (%i)%llu head=%llu (%i)%llu teal=%llu  \n",blockIdx.x,blockIdx.x+num_el,d_v_in[blockIdx.x],head,blockIdx.x-1,d_v_in[blockIdx.x-1],teal);
	   }

}


__device__ void tcopy_hor(LongPointer *d_t, int i,int j, int h, LongPointer *d_f)
// d_t - откуда
// i - первый бит из d_t
// j - первый бит из d_f
// h - сколько бит/строк копируется
// d_f -куда
{
	unsigned long long int *d_v,*d_v_in, head, teal;
// проверка, если не все столбцы обрабатываются своей нитью
	d_v=d_f[threadIdx.x];//маленький
	d_v_in=d_t[threadIdx.x];// большой
	int n1m_el=i>>6;//номер первого элемента в исходном слайсе
	int n2m_el=j>>6;//номер первого элемента в исходном слайсе
	int n1m_el1=(i-1+h)>>6; // номер последнего элемента в маленьком
	int n2m_el1=(j-1+h)>>6; // номер последнего элемента в маленьком
//	int num_bit_first= i % SIZE_OF_LONG_INT -1; // номер бита в элементе, в который копируется первый маленького слайса
//	int num_bit_last = h % SIZE_OF_LONG_INT;
	   char prb[65];
	//   printf("num_els %i and %i (%i) bits from %i to %i \n",num_el,num_el1,num_el2, num_bit_first, num_bit_last);
//	   unsigned long long int head, teal;
     if (i==j) //without shift
     {
    	 if((blockIdx.x>0)&&(blockIdx.x<10))
    	 { d_v_in[blockIdx.x+n2m_el]=d_v[blockIdx.x+n1m_el];}
    	 else {}
     }
     else
    	 if(i<j) //shift_down(j-i)
    	 {

    	 }
    	 else //shift_up(i-j)
    	 {

    	 }
}
